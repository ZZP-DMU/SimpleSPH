#include "hip/hip_runtime.h"
#include "Lib.cuh"
using namespace std;

void ptc_nnps_direct(SPH *sph)
//void ptc_nnps_direct(SPH_PARTICLE *particle,SPH_PAIR *pair)
{   
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    particle = sph->particle;
    pair = sph->pair;

    pair->total = 0;
    for(int i=0;i<particle->total;i++)
    {   
        for(int j=i+1;j<particle->total;j++)
        {
        /*  if the distance between the particles i and j is less or equ to PTC_RADIUS,then they are a pair*/
            if(PTC_DISTANCE(i,j) <= PTC_REGION_RADIUS) 
            {
                if(particle->type[i] == 0)
                {
                    pair->i[pair->total] = i;
                    pair->j[pair->total] = j;
                    pair->total = pair->total+1;
                }
                else if (particle->type[j] == 0)
                {
                    pair->i[pair->total] = j;
                    pair->j[pair->total] = i;
                    pair->total = pair->total+1;
                }
            }
        }
    }
}

void ptc_nnps_mesh(SPH *sph)
//void ptc_nnps_mesh(SPH_PARTICLE *particle,SPH_PAIR *pair,unsigned int ***mesh)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_MESH mesh;
    particle = sph->particle;
    pair = sph->pair;
    mesh = sph->mesh;

    pair->total = 0;

    omp_lock_t lock;
    omp_init_lock(&lock);

    for(int j=0;j<MESH_LENGTH_NUM;j++)
    {
        for(int i=0;i<MESH_DEEPTH_NUM;i++)
        {
            for(unsigned int k=0;k<mesh[i][j][MESH_PTC_NUM-1];k++)
            {
                //mesh[i][j]-->mesh[i][j]
                for(unsigned int m=k+1;m<mesh[i][j][MESH_PTC_NUM-1];m++)
                {
                    if(PTC_DISTANCE(mesh[i][j][k],mesh[i][j][m])<=PTC_REGION_RADIUS)
                    {
                        if(particle->type[mesh[i][j][k]]==0)
			            {
                            pair->i[pair->total] = mesh[i][j][k];
                            pair->j[pair->total] = mesh[i][j][m];
                            pair->total++;
                        }
                        else if (particle->type[mesh[i][j][m]]==0)
                        {
                            pair->i[pair->total] = mesh[i][j][m];
                            pair->j[pair->total] = mesh[i][j][k];
                            pair->total++;
                        }
                    }
                }
                //mesh[i][j]-->mesh[i][j+1]
                if(j<(MESH_LENGTH_NUM-1))
                {
                    for(unsigned int m=0;m<mesh[i][j+1][MESH_PTC_NUM-1];m++)
                    {
                        if(PTC_DISTANCE(mesh[i][j][k],mesh[i][j+1][m])<=PTC_REGION_RADIUS)
                        {
                            if(particle->type[mesh[i][j][k]]==0)
			                {
                                pair->i[pair->total] = mesh[i][j][k];
                                pair->j[pair->total] = mesh[i][j+1][m];
                                pair->total++;
                            }
                            else if (particle->type[mesh[i][j+1][m]]==0)
                            {
                                pair->i[pair->total] = mesh[i][j+1][m];
                                pair->j[pair->total] = mesh[i][j][k];
                                pair->total++;
                            }
                        }
                    }
                }
                //mesh[i][j]-->mesh[i+1][j]
                if(i<(MESH_DEEPTH_NUM-1))
                {
                    for(unsigned int m=0;m<mesh[i+1][j][MESH_PTC_NUM-1];m++)
                    {
                        if(PTC_DISTANCE(mesh[i][j][k],mesh[i+1][j][m])<=PTC_REGION_RADIUS)
                        {
                            if(particle->type[mesh[i][j][k]]==0)
			                {
                                pair->i[pair->total] = mesh[i][j][k];
                                pair->j[pair->total] = mesh[i+1][j][m];
                                pair->total++;
                            }
                            else if (particle->type[mesh[i+1][j][m]]==0)
                            {
                                pair->i[pair->total] = mesh[i+1][j][m];
                                pair->j[pair->total] = mesh[i][j][k];
                                pair->total++;
                            }
                        }
                    }
                }
                //mesh[i][j]-->mesh[i+1][j+1]
                if(i < (MESH_DEEPTH_NUM-1) && j < (MESH_LENGTH_NUM-1))
                {
                    for(unsigned int m=0;m<mesh[i+1][j+1][MESH_PTC_NUM-1];m++)
                    {
                        if(PTC_DISTANCE(mesh[i][j][k],mesh[i+1][j+1][m])<=PTC_REGION_RADIUS)
                        {
                            if(particle->type[mesh[i][j][k]]==0)
			                {
                                pair->i[pair->total] = mesh[i][j][k];
                                pair->j[pair->total] = mesh[i+1][j+1][m];
                                pair->total++;
                            }
                            else if (particle->type[mesh[i+1][j+1][m]]==0)
                            {
                                pair->i[pair->total] = mesh[i+1][j+1][m];
                                pair->j[pair->total] = mesh[i][j][k];
                                pair->total++;
                            }
                        }
                    }

                }
                //mesh[i][j]-->mesh[i-1][j+1]
                if(i > 0 && j<(MESH_LENGTH_NUM-1))
                {
                    for(unsigned int m=0;m<mesh[i-1][j+1][MESH_PTC_NUM-1];m++)
                    {
                        if(PTC_DISTANCE(mesh[i][j][k],mesh[i-1][j+1][m])<=PTC_REGION_RADIUS)
                        {
                            if(particle->type[mesh[i][j][k]]==0)
			                {
                                pair->i[pair->total] = mesh[i][j][k];
                                pair->j[pair->total] = mesh[i-1][j+1][m];
                                pair->total++;
                            }
                            else if (particle->type[mesh[i-1][j+1][m]]==0)
                            {
                                pair->i[pair->total] = mesh[i-1][j+1][m];
                                pair->j[pair->total] = mesh[i][j][k];
                                pair->total++;
                            }
                        }
                    }
                }
            }
        }
    }
}

void ptc_nnps_check(SPH_PAIR *pair,SPH_PAIR *pair_direct,unsigned int *total)
{
    *total = 0;
    if(pair->total == pair_direct->total)
    {
        for(int i=0;i<pair->total;i++)
        {   
            for(int j=0;j<pair_direct->total;j++)
            {
                if(pair->i[i]==pair_direct->i[j] && pair->j[i] == pair_direct->j[j])
                {
                    (*total)++;
                }
                else if(pair->i[i] == pair_direct->j[j] && pair->j[i] == pair_direct->i[j])
                {
                    (*total)++;
                }
            }
        }
        }
}
__global__ void sph_nnps_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{

    //n---> (x)length direction
    //m ---> (y)depth direction

    double dx = 0.0;
    double dy = 0.0;
    double q = 0.0;
    int index_i = 0;
    int index_j = 0;
    //int arg->pair_num=0;
    int mesh_id = 0;
    //if( blockIdx.x >= arg->mesh_ynum) return;
    //if( threadIdx.x >= arg->mesh_xnum) return;
    //const int mesh_id = threadIdx.x + blockIdx.x * blockDim.x;
    for(int m =0;m<arg->mesh_ynum;m++)
    {
        for(int n=0;n<arg->mesh_xnum;n++)
        {
            mesh_id = n+m*arg->mesh_xnum;   
            for(int i=0;i<cuda->mesh_count[mesh_id];i++)
            {
                index_i = mesh_id + i*arg->mesh_num;
                //(x,y)->(x,y)
                for(int j=i+1;j<cuda->mesh_count[mesh_id];j++)
                {
                    index_j = mesh_id + j*arg->mesh_num;
                    dx = cuda->x[cuda->mesh[index_i]] - cuda->x[cuda->mesh[index_j]];
                    dy = cuda->y[cuda->mesh[index_i]] - cuda->y[cuda->mesh[index_j]];
                    q = sqrt(dx*dx+dy*dy)/arg->h;
                    if(q<2.0)
                    {
                        if(cuda->type[cuda->mesh[index_i]] == 0)
                        {
                            //arg->pair_num = atomicAdd(&(arg->pair_num),1);
                            cuda->pair_i[arg->pair_num] = cuda->mesh[index_i];
                            cuda->pair_j[arg->pair_num] = cuda->mesh[index_j];
                            arg->pair_num +=1;
                        }
                        else if(cuda->type[cuda->mesh[index_j]] == 0)
                        {
                            //arg->pair_num = atomicAdd(&(arg->pair_num),1);
                            cuda->pair_i[arg->pair_num] = cuda->mesh[index_j];
                            cuda->pair_j[arg->pair_num] = cuda->mesh[index_i];
                            arg->pair_num +=1;
                        }
                    }
                }
                //(x,y)->(x+1,y)
                if( n<(arg->mesh_xnum-1))
                {
                    for(int j=0;j<cuda->mesh_count[mesh_id+1];j++)
                    {
                        index_j = mesh_id + 1 + j*arg->mesh_num;
                        dx = cuda->x[cuda->mesh[index_i]] - cuda->x[cuda->mesh[index_j]];
                        dy = cuda->y[cuda->mesh[index_i]] - cuda->y[cuda->mesh[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(cuda->type[cuda->mesh[index_i]] == 0)
                            {
                                //arg->pair_num = atomicAdd(&(arg->pair_num),1);
                                cuda->pair_i[arg->pair_num] = cuda->mesh[index_i];
                                cuda->pair_j[arg->pair_num] = cuda->mesh[index_j];
                                arg->pair_num +=1;
                            }
                            else if(cuda->type[cuda->mesh[index_j]] == 0)
                            {
                                //arg->pair_num = atomicAdd(&(arg->pair_num),1);
                                cuda->pair_i[arg->pair_num] = cuda->mesh[index_j];
                                cuda->pair_j[arg->pair_num] = cuda->mesh[index_i];
                                arg->pair_num +=1;
                            }
                        }
                    }   
                }

                //(x,y)->(x,y+1)
                if( m<(arg->mesh_ynum-1))
                {
                    for(int j=0;j<cuda->mesh_count[mesh_id+arg->mesh_xnum];j++)
                    {
                        index_j = mesh_id + arg->mesh_xnum + j*arg->mesh_num;
                        dx = cuda->x[cuda->mesh[index_i]] - cuda->x[cuda->mesh[index_j]];
                        dy = cuda->y[cuda->mesh[index_i]] - cuda->y[cuda->mesh[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(cuda->type[cuda->mesh[index_i]] == 0)
                            {
                                //arg->pair_num = atomicAdd(&(arg->pair_num),1);
                                cuda->pair_i[arg->pair_num] = cuda->mesh[index_i];
                                cuda->pair_j[arg->pair_num] = cuda->mesh[index_j];
                                arg->pair_num +=1;
                            }
                            else if(cuda->type[cuda->mesh[index_j]] == 0)
                            {
                                //arg->pair_num = atomicAdd(&(arg->pair_num),1);
                                cuda->pair_i[arg->pair_num] = cuda->mesh[index_j];
                                cuda->pair_j[arg->pair_num] = cuda->mesh[index_i];
                                arg->pair_num +=1;
                            }
                        }
                    }
                }

                //(x,y)->(x+1,y+1)
                if( n<(arg->mesh_xnum-1) && m<(arg->mesh_ynum-1))
                {
                    for(int j=0;j<cuda->mesh_count[mesh_id+1+arg->mesh_xnum];j++)
                    {
                        index_j = mesh_id + 1 + arg->mesh_xnum + j*arg->mesh_num;
                        dx = cuda->x[cuda->mesh[index_i]] - cuda->x[cuda->mesh[index_j]];
                        dy = cuda->y[cuda->mesh[index_i]] - cuda->y[cuda->mesh[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(cuda->type[cuda->mesh[index_i]] == 0)
                            {
                                //arg->pair_num = atomicAdd(&(arg->pair_num),1);
                                cuda->pair_i[arg->pair_num] = cuda->mesh[index_i];
                                cuda->pair_j[arg->pair_num] = cuda->mesh[index_j];
                                arg->pair_num +=1;
                            }
                            else if(cuda->type[cuda->mesh[index_j]] == 0)
                            {
                                //arg->pair_num = atomicAdd(&(arg->pair_num),1);
                                cuda->pair_i[arg->pair_num] = cuda->mesh[index_j];
                                cuda->pair_j[arg->pair_num] = cuda->mesh[index_i];
                                arg->pair_num +=1;
                            }
                        }
                    }
                }

                //(x,y)->(x-1,y+1)
                if( n>0 && m<(arg->mesh_ynum-1))
                {
                    for(int j=0;j<cuda->mesh_count[mesh_id-1+arg->mesh_xnum];j++)
                    {
                        index_j = mesh_id - 1 + arg->mesh_xnum + j*arg->mesh_num;
                        dx = cuda->x[cuda->mesh[index_i]] - cuda->x[cuda->mesh[index_j]];
                        dy = cuda->y[cuda->mesh[index_i]] - cuda->y[cuda->mesh[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(cuda->type[cuda->mesh[index_i]] == 0)
                            {
                                //arg->pair_num = atomicAdd(&(arg->pair_num),1);
                                cuda->pair_i[arg->pair_num] = cuda->mesh[index_i];
                                cuda->pair_j[arg->pair_num] = cuda->mesh[index_j];
                                arg->pair_num +=1;
                            }
                            else if(cuda->type[cuda->mesh[index_j]] == 0)
                            {
                                //arg->pair_num = atomicAdd(&(arg->pair_num),1);
                                cuda->pair_i[arg->pair_num] = cuda->mesh[index_j];
                                cuda->pair_j[arg->pair_num] = cuda->mesh[index_i];
                                arg->pair_num +=1;
                            }
                        }
                    }
                }
            }
            cuda->mesh_count[mesh_id] = 0;
        }
    }
}
