#include "hip/hip_runtime.h"
#include "Lib.cuh"

void ptc_density_correct(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    particle = sph->particle;
    pair= sph->pair;
    kernel = sph->kernel;

    double a = ALPHA;
    //double m = PTC_MASS;


    for(unsigned int i=0;i<particle->total;i++)
    {
        if(particle->type[i] == 0)
        {
            particle->w[i] = (a*2.0*particle->mass[i])/(3.0*particle->density[i]);
        }
    }

    for(unsigned int i=0;i<pair->total;i++)
    {
        particle->w[pair->i[i]] += kernel->w[i]*particle->mass[pair->j[i]]/particle->density[pair->j[i]];
        if(particle->type[pair->j[i]]==0)
        {
            particle->w[pair->j[i]] += kernel->w[i]*particle->mass[pair->i[i]]/particle->density[pair->i[i]];
        }
    }

    for(unsigned int i=0;i<particle->total;i++)
    {
        if(particle->type[i] == 0)
        {
            particle->density[i] = (a*2.0*particle->mass[i])/(3.0*particle->w[i]);
        }
    }

    for(unsigned int i=0;i<pair->total;i++)
    {
        particle->density[pair->i[i]] += particle->mass[pair->j[i]]*kernel->w[i]/particle->w[pair->i[i]];
        if(particle->type[pair->j[i]] == 0)
        {
            particle->density[pair->j[i]] += particle->mass[pair->i[i]]*kernel->w[i]/particle->w[pair->j[i]];
        }
    }
}

void ptc_dummy(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    SPH_RIGID *wedge;
    particle = sph->particle;
    pair = sph->pair;
    kernel = sph->kernel;
    wedge = sph->rigid;


    //rigid body(wall & wedge)vx,vy,accx,accy,pressure init
    for(unsigned int i=0;i<particle->total;i++)
    {
        if(particle->type[i] != 0)
        {
            particle->w[i] = 0;
            particle->vx[i] = 0;
            particle->vy[i] = 0;
            particle->pressure[i] = 0;
            particle->density[i] = 0;
        }
    }
    
    //the not fluid weight term 
    for(unsigned int i=0;i<pair->total;i++)
    {
        if(particle->type[pair->j[i]] != 0) 
        {
            particle->w[pair->j[i]] += kernel->w[i];
        }
    }
    
    //rigid body(wall & wedege) pressure and velocity
    for(unsigned int i=0;i<pair->total;i++)
    {
        double dx = 0.0;
        double dy = 0.0;
        double rigid_acc_x = 0.0;
        double rigid_acc_y = 0.0;
        if(particle->type[pair->j[i]] != 0 && particle->w[pair->j[i]] != 0.0)
        {
            if(particle->type[pair->j[i]] == -1)
            {
                rigid_acc_x = 0.0;
                rigid_acc_y = 0.0;
            }
            else if (particle->type[pair->j[i]] == 1)
            {
                rigid_acc_x = wedge->accx - pow(wedge->omega,2)*(particle->x[pair->j[i]]-wedge->cogx)- \
                              wedge->alpha*(particle->y[pair->j[i]]-wedge->cogy);
                rigid_acc_y = wedge->accy - pow(wedge->omega,2)*(particle->y[pair->j[i]]-wedge->cogy)+ \
                              wedge->alpha*(particle->x[pair->j[i]]-wedge->cogx);
            }
            dx = particle->x[pair->i[i]] - particle->x[pair->j[i]];
            dy = particle->y[pair->i[i]] - particle->y[pair->j[i]];
            particle->pressure[pair->j[i]] += (particle->pressure[pair->i[i]]+particle->density[pair->i[i]]*\
                        (rigid_acc_x*dx+(rigid_acc_y+GRAVITY_ACC)*dy))*kernel->w[i]/particle->w[pair->j[i]];
            particle->vx[pair->j[i]] += particle->vx[pair->i[i]]*kernel->w[i]/particle->w[pair->j[i]];
            particle->vy[pair->j[i]] += particle->vy[pair->i[i]]*kernel->w[i]/particle->w[pair->j[i]];
        }
    }

    //rigid body(wall & wedege) densiy
    for(unsigned int i=0;i<particle->total;i++)
    {
        if(particle->type[i] != 0)
        {
            particle->density[i] = particle->pressure[i]/pow(sph->c,2)+REF_DENSITY;
        }
    }
}

void sph_avg_time(SPH *sph)
{
    static unsigned int step = 0;
    static double start;
    static double end;
    if(step == 0)
    {
        start = (double)time(nullptr);
    }
    else
    {
        end = (double)time(nullptr);
        sph->avg_time = (end-start)/(double)step;
    }
    step++;
}
void sph_read_info(SPH *sph)
{
    string filename = sph->arg->case_dir;
    filename += "/case_info"
    ifstream case_info;
    case_info.open(filename.c_str());
    string line;
    //case_info << sph->arg->c << endl;
    getline(case_info,line);
    sph->arg->c = stod(line.c_str());
    //case_info << sph->arg->g <<endl;   //gravity acceleration
    getline(case_info,line);
    sph->arg->g = stod(line.c_str());
    //case_info << sph->arg->ref_rho << endl;     //reference density
    getline(case_info,line);
    sph->arg->ref_rho = stod(line.c_str());
    //case_info << sph->arg->ptc_dx << endl;      //ptc delta spacing
    getline(case_info,line);
    sph->arg->ptc_dx = stod(line.c_str());
    //case_info << sph-arg->r << endl;       //ptc radius
    getline(case_info,line);
    sph->arg->r = stod(line.c_str());
    //case_info << sph->arg->h << endl;   //smoothed length
    getline(case_info,line);
    sph->arg->h = stod(line.c_str());
    //case_info << sph->arg->m << endl;   //ptc mass
    getline(case_info,line);
    sph->arg->m = stod(line.c_str());
    //case_info << sph->arg->alpha << endl;   //kernel function's para
    getline(case_info,line);
    sph->arg->alpha = stod(line.c_str());
    //case_info << sph->arg->sst << endl;    //single step time
    getline(case_info,line);
    sph->arg->sst = stod(line.c_str());
    //case_info << sph->arg->dt << endl;  //delta t
    getline(case_info,line);
    sph->arg->dt = stod(line.c_str());

    //case_info << sph->arg->fluid_x << endl;     //fluid length
    getline(case_info,line);
    sph->arg->fluid_x = stod(line.c_str());
    //case_info << sph->arg->fluid_y << endl;     //fluid depth
    getline(case_info,line);
    sph->arg->fluid_y = stod(line.c_str());
    //case_info << sph->arg->fluid_xnum << endl;     //fluid length direction ptc num
    getline(case_info,line);
    sph->arg->fluid_xnum = stoi(line.c_str());
    //case_info << sph->arg->fluid_ynum << endl;     //fluid depth direction ptc num
    getline(case_info,line);
    sph->arg->fluid_ynum = stoi(line.c_str());
    
    //case_info << sph->arg->domain_x << endl;    //total domain length
    getline(case_info,line);
    sph->arg->domain_x = stod(line.c_str());
    //case_info << sph->arg->domain_y << endl;    //total domain depth
    getline(case_info,line);
    sph->arg->domain_y = stod(line.c_str());

    //case_info << sph->arg->mesh_dx << endl;     //mesh delta spacing
    getline(case_info,line);
    sph->arg->mesh_dx = stod(line.c_str());
    //case_info << sph->arg->mesh_xnum << endl;      //mesh length direction num
    getline(case_info,line);
    sph->arg->mesh_xnum = stoi(line.c_str());
    //case_info << sph->arg->mesh_ynum << endl;      //mesh depth direction num
    getline(case_info,line);
    sph->arg->mesh_ynum = stoi(line.c_str());
    //case_info << sph->arg->mesh_num << endl;       //total mesh num
    getline(case_info,line);
    sph->arg->mesh_num = stoi(line.c_str());
    //case_info << sph->arg->mesh_volume << endl;    //single mesh volume
    getline(case_info,line);
    sph->arg->mesh_volume = stoi(line.c_str());

    //case_info << sph->arg->init_step << endl;  //inital time step
    getline(case_info,line);
    sph->arg->init_setp = stoi(line.c_str());
    //case_info << sph->arg->total_step << endl; //total time step
    getline(case_info,line);
    sph->arg->total_step = stoi(line.c_str());

    //current process flags
    //case_info << sph->arg->new_case_flag << endl;  // if 1 then creat a new case,or continue to run the old case
    getline(case_info,line);
    sph->arg->new_case_flag = stoi(line.c_str());
    //case_info << sph->arg->init_impac_flag << endl; //if 1 then run the init step,or run the impac step
    getline(case_info,line);
    sph->arg->init_impac_flag = stoi(line.c_str());
    //case_info << sph->arg->save_last_flag << endl; //if 1 then save the last step,or donnot save it
    getline(case_info,line);
    sph->arg->save_last_flag = stoi(line.c_str());

    //rigid info
    //case_info << sph->rigid->vx << endl;  //rigid body x-direction velocity
    getline(case_info,line);
    sph->rigid->vx = stod(line.c_str());
    //case_info << sph->rigid->vy << endl;  //rigid body y-direction velocity
    getline(case_info,line);
    sph->rigid->vy = stod(line.c_str());
    //case_info << sph->rigid->omega << endl;   //rigid body angular velocity
    getline(case_info,line);
    sph->rigid->omega = stod(line.c_str());
    //case_info << sph->rigid->accx << endl;    //rigid body x-direciton acceleration
    getline(case_info,line);
    sph->rigid->accx = stod(line.c_str());
    //case_info << sph->rigid->accy << endl;    //rigid body y-direction acceleration
    getline(case_info,line);
    sph->rigid->accy = stod(line.c_str());
    //case_info << sph->rigid->alpha << endl;   //rigid body angular acceleration
    getline(case_info,line);
    sph->rigid->alpha = stod(line.c_str());
    //case_info << sph->rigid->cogx << endl;    //x-direction center of gravity coordinate
    getline(case_info,line);
    sph->rigid->cogx = stod(line.c_str());
    //case_info << sph->rigid->cogy << endl;    //y-direction center of gravity coordinate 
    getline(case_info,line);
    sph->rigid->cogy = stod(line.c_str());
    //case_info << sph->rigid->mass << endl;    //rigid body mass 
    getline(case_info,line);
    sph->rigid->mass = stod(line.c_str());
    //case_info << sph->rigid->moi << endl;     //rigid body moment of inertia
    getline(case_info,line);
    sph->rigid->moi = stod(line.c_str());
    //case_info << sph->rigid->total << endl;   //rigid body ptc num
    getline(case_info,line);
    sph->rigid->total = stoi(line.c_str());
    
    case_info.close();

}

void sph_write_info(SPH *sph)
{
    string filename = sph->arg->case_dir;
    filename += "/case_info"
    ofstream case_info;
    case_info.open(filename.c_str());

    //sph paraments
    case_info << sph->arg->c << endl;
    case_info << sph->arg->g <<endl;   //gravity acceleration
    case_info << sph->arg->ref_rho << endl;     //reference density
    case_info << sph->arg->ptc_dx << endl;      //ptc delta spacing
    case_info << sph-arg->r << endl;       //ptc radius
    case_info << sph->arg->h << endl;   //smoothed length
    case_info << sph->arg->m << endl;   //ptc mass
    case_info << sph->arg->alpha << endl;   //kernel function's para
    case_info << sph->arg->sst << endl;    //single step time
    case_info << sph->arg->dt << endl;  //delta t

    case_info << sph->arg->fluid_x << endl;     //fluid length
    case_info << sph->arg->fluid_y << endl;     //fluid depth
    case_info << sph->arg->fluid_xnum << endl;     //fluid length direction ptc num
    case_info << sph->arg->fluid_ynum << endl;     //fluid depth direction ptc num
    
    case_info << sph->arg->domain_x << endl;    //total domain length
    case_info << sph->arg->domain_y << endl;    //total domain depth

    case_info << sph->arg->mesh_dx << endl;     //mesh delta spacing
    case_info << sph->arg->mesh_xnum << endl;      //mesh length direction num
    case_info << sph->arg->mesh_ynum << endl;      //mesh depth direction num
    case_info << sph->arg->mesh_num << endl;       //total mesh num
    case_info << sph->arg->mesh_volume << endl;    //single mesh volume

    case_info << sph->arg->init_step << endl;  //inital time step
    case_info << sph->arg->total_step << endl; //total time step

    //current process flags
    case_info << sph->arg->new_case_flag << endl;  // if 1 then creat a new case,or continue to run the old case
    case_info << sph->arg->init_impac_flag << endl; //if 1 then run the init step,or run the impac step
    case_info << sph->arg->save_last_flag << endl; //if 1 then save the last step,or donnot save it

    //rigid info
    case_info << sph->rigid->vx << endl;  //rigid body x-direction velocity
    case_info << sph->rigid->vy << endl;  //rigid body y-direction velocity
    case_info << sph->rigid->omega << endl;   //rigid body angular velocity
    case_info << sph->rigid->accx << endl;    //rigid body x-direciton acceleration
    case_info << sph->rigid->accy << endl;    //rigid body y-direction acceleration
    case_info << sph->rigid->alpha << endl;   //rigid body angular acceleration
    case_info << sph->rigid->cogx << endl;    //x-direction center of gravity coordinate
    case_info << sph->rigid->cogy << endl;    //y-direction center of gravity coordinate 
    case_info << sph->rigid->mass << endl;    //rigid body mass 
    case_info << sph->rigid->moi << endl;     //rigid body moment of inertia
    case_info << sph->rigid->total << endl;   //rigid body ptc num

    case_info.close();
}

void sph_generate_case(SPH *sph)
{
    cout << "ptc spacing is:";
    cin >> sph->arg->ptc_dx;

    cout << "fluid domain length is:";
    cin >> sph->arg->fluid_x;

    cout << "fluid domain depth is:";
    cin >> sph->arg->fluid_y;

    sph->arg->fluid_xnum = int(sph->arg->fluid_x/sph->arg->ptc_dx)+1;
    sph->arg->fluid_ynum = int(sph->arg->fluid_y/sph->arg->ptc_dx)+1;

    sph->arg->domain_x = sph->arg->fluid_x;
    sph->arg->domain_y = 1.5*sph->arg->fluid.y;
    
    sph->arg->h = 1.005*sph->arg->dx;
    sph->arg->r = 2.0*sph->arg->h;
    sph->arg->mesh_dx = sph->arg->r;
    sph->arg->mesh_xnum = int(sph->arg->domain_x/sph->arg->mesh_dx)+1;
    sph->arg->mesh_ynum = int(sph->arg->domain_y/sph->arg->mesh_dx)+1;
    sph->arg->mesh_num = sph->arg->mesh_xnum*sph->arg->mesh_ynum;
    sph->arg->mesh_volume = 32;

    sph->arg->g = 9.8;
    sph->arg->c = 10*sqrt(sph->arg->g*sph->arg->fluid_y);
    sph->arg->ref_rho = 1000.0;
    sph->arg->m = sph->arg->ref_rho*sph->arg->ptc_dx*sph->arg->ptc_dx;
    sph->arg->alpha = 15.0/(7*3.14159265358*sph->arg->h*sph->arg->h);

    cout << "the init step:";
    cin >> sph->arg->init_step;
    cout << "the total step:";
    cin >> sph->arg->total_step;

    cout << "new case flag,if 1 then creat a new case:";
    cin >> sph->arg->new_case_flag;
    cout << "init or impac simulation,if 1 then run the init step:";
    cin >> sph->arg->init_impac_flag;
    cout << "save the last time step info,if 1 to save:";
    cin >> sph->arg->save_last_flag;
}

void sph_change_case(SPH *sph)
{

}
