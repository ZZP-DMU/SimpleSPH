#include "hip/hip_runtime.h"
#include "SPH.cuh"
#include <fstream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>
using namespace std;

__constant__  int c = ART_SOUND_VEL;
__constant__  int rho_0 = REF_DENSITY;
__constant__  int mesh_lnum = MESH_LENGTH_NUM;
__constant__  int mesh_dnum = MESH_DEEPTH_NUM;
__constant__  int mesh_pnum = MESH_PTC_NUM;
__constant__  int mesh_spacing = MESH_SPACING;

int main(void)
{ 
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh = NULL;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.rigid = &wedge;
    sph.mesh = mesh;

    sph_init(&sph); 
    hipSetDevice(0);

    double *dev_x;
    double *dev_y;
    double *dev_vx;
    double *dev_vy;
    double *dev_rho;
    double *dev_p;
    /*
    dev_pair_i,dev_pair_j,dev_pair_accx,dev_pair_accy,dev_pair_drho = NULL;
    */
    int *dev_mesh =NULL;

    int temp = 0;

    CUDA_CHECK(hipMalloc((double**)&dev_x,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_y,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_vx,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_vy,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_rho,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_p,particle.total*sizeof(double)));
/*
    CUDA_CHECK(hipMalloc((double**)&dev_pair_i,size*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_j,size*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_accx,size*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_accy,size*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_drho,size*sizeof(double)));
    */

    CUDA_CHECK(hipMalloc((int**)&dev_mesh,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int)));

       
    sph_avg_time(&sph);
    for(sph.current_step;sph.current_step<sph.total_step;sph.current_step++)
    {
        hipMemcpy((void *)dev_x, (void *)particle.x, particle.total*sizeof(double), hipMemcpyHostToDevice); 
        hipMemcpy((void *)dev_y, (void *)particle.y, particle.total*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy((void *)dev_vx, (void *)particle.vx, particle.total*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy((void *)dev_vy, (void *)particle.vy, particle.total*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy((void *)dev_rho, (void *)particle.density, particle.total*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy((void *)dev_p, (void *)particle.pressure, particle.total*sizeof(double), hipMemcpyHostToDevice);

        ptc_mesh_cuda<<<384,160>>>(dev_x,dev_y,dev_mesh,particle.total);
        hipMemcpy(mesh, dev_mesh, MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM,hipMemcpyDeviceToHost);


    string filename = "../data/postprocess/vtk/sph"; 
    filename += to_string(sph.current_step/PRINT_TIME_STEP);
    filename += ".vtk";

    ofstream vtkfile;
    vtkfile.open(filename.c_str());

    vtkfile << "# vtk DataFile Version 3.0" << endl;
    vtkfile << "sph data" << endl;
    vtkfile << "ASCII" << endl;
    vtkfile << "DATASET UNSTRUCTURED_GRID" << endl;
    vtkfile << "POINTS " << particle.total << " " << "double" << endl;

    for(unsigned int i=0;i<MESH_DEEPTH_NUM;i++)
    {
        for(unsigned int j=0;j<MESH_LENGTH_NUM;j++)
        {
            for(unsigned int k=0;k<MESH_PTC_NUM;k++)
            temp = mesh[i*MESH_LENGTH_NUM+j+k];
            vtkfile << setiosflags(ios::scientific) << particle.x[temp] << " " \
            << particle.y[temp] << " " << 0.0 << endl;
        }
    }
    vtkfile.close();



        /*
        if(sph.current_step%PRINT_TIME_STEP == 0)
        {
            sph_save_single(&sph);
        }
        //calculate and integration
        sph_time_integral(&sph); 
        sph_save_rigid(&sph);
        ptc_info(&sph);
        sph_avg_time(&sph);
        */
    }
    sph_save_last(&sph);
    sph_free(&sph);
    return 0;
}
/*__global__ void sph_predict_cuda(double *x,double *y,double *temp_x,double *temp_y,double *vx,double *vy,double *temp_vx,double *temp_vy,double *accx,double *accy,double *rho,double *temp_rho,double *drho,double *p,int *type,int ptc_num)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= ptc_num )return;

    if(type[id] == 0)
    {
        temp_x[id] = x[id];
        temp_y[id] = y[id];
        temp_vx[id] = vx[id];
        temp_vy[id] = vy[id];
        temp_rho[id] = rho[id];

        x[id] += vx[id]*dev_dt*0.5;
        y[id] += vy[id]*dev_dt*0.5;
        vx[id] += accx[id]*dev_dt*0.5;
        vy[id] += accy[id]*dev_dt*0.5;
        rho[id] += drho[id]*dev_dt*0.5;
        if(rho[id] < REF_DENSITY) rho[id]=REF_DENSITY;
    }
 
    else
    {
        vx[id] = 0.0;
        vy[id] = 0.0;
        p[id] = 0.0;
    }

}


__global__ void sph_correct_cuda(double *x,double *y,double *temp_x,double *temp_y,double *vx,double *vy,double *temp_vx,double *temp_vy,double *accx,double *accy,double *rho,double *temp_rho,double *drho,double *p,int *type,int ptc_num)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= ptc_num )return; 

    if(type[id] == 0)
    {
        x[id] = temp_x[id] + vx[id]*dev_dt;
        y[id] = temp_y[id] + vy[id]*dev_dt;
        vx[id] = temp_vx[id] + accx[id]*dev_dt;
        vy[id] = temp_vy[id] + accy[id]*dev_dt;
        rho[id] = temp_rho[id] + drho[id]*dev_dt;
        if(rho[id] < REF_DENSITY) rho[id]=REF_DENSITY;
    }
    else
    {
        vx[id] = 0.0;
        vy[id] = 0.0;
        p[id] = 0.0;
    }
}*/
/*
        CUDA_CHECK(hipMemcpy(sph.mesh,dev_mesh,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int),hipMemcpyDeviceToHost));
    string filename = "../data/postprocess/vtk/sph"; 
    filename += to_string(sph.current_step/PRINT_TIME_STEP);
    filename += ".vtk";

    ofstream vtkfile;
    vtkfile.open(filename.c_str());

    vtkfile << "# vtk DataFile Version 3.0" << endl;
    vtkfile << "sph data" << endl;
    vtkfile << "ASCII" << endl;
    vtkfile << "DATASET UNSTRUCTURED_GRID" << endl;
    vtkfile << "POINTS " << sph.particle->total << " " << "double" << endl;

    for(unsigned int i=0;i<MESH_DEEPTH_NUM;i++)
    {
        for(unsigned int j=0;j<MESH_LENGTH_NUM;j++)
        {
            temp = sph.mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH_NUM*(MESH_PTC_NUM-1)];
            for(unsigned int k=0;k<temp;k++)
            {
                temp_1 = sph.mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH_NUM*k];
                vtkfile << setiosflags(ios::scientific) << sph.particle->x[temp_1] << " " \
                << sph.particle->y[temp_1] << " " << 0.0 << endl;
            }
        }
    }
    vtkfile.close();*/