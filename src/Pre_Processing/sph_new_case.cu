#include "hip/hip_runtime.h"
#include "SPH.cuh"
#include <iostream>
using namespace std;

void get_input(SPH *);
void fluid_ptc_generate(SPH *);
void rigid_ptc_generate(SPH *);

int main(int argc,char *argv[])
{
    SPH_ARG arg;
    SPH_RIGID rigid;
    SPH_PARTICLE particle;
    SPH sph;
    sph.host_arg = &arg;
    sph.host_rigid = &rigid;
    sph.particle = &particle;
    if(argc != 2) printf("\033[0;32;31m Error in %s:%d\n",__FILE__,__LINE__);
    arg.case_dir = argv[1];
    get_input(&sph);
    fluid_ptc_generate(&sph);
    rigid_ptc_generate(&sph);
    sph_write_info(&sph);

    return 0;
}

void get_input(SPH *sph)
{
    SPH_ARG *arg;
    arg = sph->host_arg;
    SPH_RIGID *rigid;
    rigid = sph->host_rigid;
    SPH_PARTICLE *particle;
    particle = sph->particle;

    cout << "fluid length (fluid_x) is:" << endl;
    cin >> arg->fluid_x;
    cout << "fluid depth (fluid_y) is:" << endl;
    cin >> arg->fluid_y;
    cout << "particle spacing (ptc_dx) is:" << endl;
    cin >> arg->ptc_dx;

    arg->h = 1.0005*arg->ptc_dx;
    arg->r = 2.0*arg->h;
    arg->wall_layer = 2;
    arg->fluid_xnum = (int)(arg->fluid_x/arg->ptc_dx)+1-2*arg->wall_layer;
    arg->fluid_ynum = (int)(arg->fluid_y/arg->ptc_dx)+1-arg->wall_layer;
    particle->rigid_ptc_num = 0;
    particle->fluid_ptc_num = arg->fluid_xnum*arg->fluid_ynum;
    particle->wall_ptc_num = ((int)(arg->fluid_x/arg->ptc_dx)+1)*((int)(1.1*arg->fluid_y/arg->ptc_dx)+1)- \
                             ((int)(arg->fluid_x/arg->ptc_dx)+1-2*arg->wall_layer)*((int)(1.1*arg->fluid_y/arg->ptc_dx)+1-arg->wall_layer);
    particle->total = particle->fluid_ptc_num + particle->wall_ptc_num;
    arg->ptc_num = particle->total;

    arg->mesh_dx = arg->r;
    arg->mesh_x = arg->fluid_x;
    arg->mesh_y = arg->fluid_y * 1.5;
    arg->mesh_xnum = (int)(arg->mesh_x/arg->mesh_dx)+1;
    arg->mesh_ynum = (int)(arg->mesh_y/arg->mesh_dx)+1;
    arg->mesh_num = arg->mesh_xnum*arg->mesh_ynum;
    arg->mesh_volume = 33;

    arg->g = 9.8;
    arg->c = 10.0*sqrt(arg->g*arg->fluid_y);
    arg->ref_rho = 1000.0;
    arg->m = arg->ref_rho*arg->ptc_dx*arg->ptc_dx;
    arg->dt = 0.00002;
    arg->sst = 0.0;
    arg->alpha = 15.0/(7.0*3.14159265358*pow(arg->h,2));

    arg->init_step = 0;
    arg->total_step = 80000;
    arg->print_step = 400;

    arg->new_case_flag =1;
    arg->init_impac_flag = 1;
    arg->save_last_flag = 1;

    rigid->vx = 0.0;
    rigid->vy = 0.0;
    rigid->omega = 0.0;
    rigid->accx = 0.0;
    rigid->accy = 0.0;
    rigid->alpha = 0.0;
    rigid->mass = 12.8;
    rigid->offset_x = 0.0;
    rigid->offset_y = 0.0;
    rigid->offset_angl = 0.0;
    rigid->cog_ptc_id = 0.0;
    rigid->cogx = 0.0;
    rigid->cogy = 0.0; 
    rigid->total = 0;

    //return 0;
}

void fluid_ptc_generate(SPH *sph)
{
    //return 0;
}

void rigid_ptc_generate(SPH *sph)
{
    //return 0;
}

