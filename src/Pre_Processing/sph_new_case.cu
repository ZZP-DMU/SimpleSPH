#include "hip/hip_runtime.h"
#include "SPH.cuh"
#include <iostream>
#include <iomanip>
#include <string>
#include <assert.h>
#include <vtkUnstructuredGridReader.h>
#include <vtkSmartPointer.h>
#include <vtkType.h>
#include <vtkUnstructuredGrid.h>
#include <vtkPointSet.h>
#include <vtkDataSetReader.h>
#include <vtkPointData.h>
#include <vtkDataArray.h>
#include <vtkDoubleArray.h>

using namespace std;

void get_input(SPH *);
void get_rigid_num(SPH *);
void fluid_ptc_generate(SPH *);
void rigid_ptc_generate(SPH *);
void write_vtk(SPH *);
void rigid_init(SPH *);

int main(int argc,char *argv[])
{
    SPH_ARG arg;
    SPH_RIGID rigid;
    SPH_PARTICLE particle;
    SPH sph;
    sph.host_arg = &arg;
    sph.host_rigid = &rigid;
    sph.particle = &particle;
    //if(argc != 2) printf("\033[0;32;31m Error in %s:%d\033[m\n",__FILE__,__LINE__);
    assert(argc == 2);
    arg.case_dir = argv[1];
    get_input(&sph);
    get_rigid_num(&sph);
    arg.pair_volume = (int)(64*arg.ptc_num/arg.mesh_num);
    particle.x = (double *)calloc(arg.ptc_num,sizeof(double));
    particle.y = (double *)calloc(arg.ptc_num,sizeof(double));
    particle.type = (int *)calloc(arg.ptc_num,sizeof(int));
    fluid_ptc_generate(&sph);
    rigid_ptc_generate(&sph);

    rigid_init(&sph);

    sph_write_info(&sph);
    write_vtk(&sph);

    return 0;
}

void get_input(SPH *sph)
{
    SPH_ARG *arg;
    arg = sph->host_arg;
    SPH_RIGID *rigid;
    rigid = sph->host_rigid;
    //SPH_PARTICLE *particle;
    //particle = sph->particle;

    cout << "fluid length (fluid_x) is:" << endl;
    cin >> arg->fluid_x;
    cout << "fluid depth (fluid_y) is:" << endl;
    cin >> arg->fluid_y;
    cout << "particle spacing (ptc_dx) is:" << endl;
    cin >> arg->ptc_dx;

    arg->h = 2.0*arg->ptc_dx;
    arg->r = 2.0*arg->h;
    arg->wall_layer = 4;
    arg->fluid_xnum = (int)(arg->fluid_x/arg->ptc_dx)+1-2*arg->wall_layer;
    arg->fluid_ynum = (int)(arg->fluid_y/arg->ptc_dx)+1-arg->wall_layer;

    arg->rigid_ptc_num = 0;
    arg->fluid_ptc_num = arg->fluid_xnum*arg->fluid_ynum;
    arg->wall_ptc_num = ((int)(arg->fluid_x/arg->ptc_dx)+1)*((int)(1.1*arg->fluid_y/arg->ptc_dx)+1)- \
                             ((int)(arg->fluid_x/arg->ptc_dx)+1-2*arg->wall_layer)*((int)(1.1*arg->fluid_y/arg->ptc_dx)+1-arg->wall_layer); 
    arg->ptc_num = arg->rigid_ptc_num + arg->fluid_ptc_num + arg->wall_ptc_num;

    arg->mesh_dx = arg->r;
    arg->mesh_x = arg->fluid_x;
    arg->mesh_y = arg->fluid_y * 1.5;
    arg->mesh_xnum = (int)(arg->mesh_x/arg->mesh_dx)+1;
    arg->mesh_ynum = (int)(arg->mesh_y/arg->mesh_dx)+1;
    arg->mesh_num = arg->mesh_xnum*arg->mesh_ynum;
    arg->mesh_volume = 33;

    arg->g = 9.8;
    arg->c = 10.0*sqrt(arg->g*arg->fluid_y);
    arg->ref_rho = 1000.0;
    arg->m = arg->ref_rho*arg->ptc_dx*arg->ptc_dx;
    arg->dt = 0.00002;
    arg->sst = 0.0;
    arg->alpha = 7.0/(4.0*3.14159265358*pow(arg->h,2));

    arg->init_step = 0;
    arg->total_step = 80000;
    arg->print_step = 400;

    arg->new_case_flag =1;
    arg->init_impac_flag = 1;
    arg->save_last_flag = 1;

    rigid->vx = 0.0;
    rigid->vy = 0.0;
    rigid->omega = 0.0;
    rigid->accx = 0.0;
    rigid->accy = 0.0;
    rigid->alpha = 0.0;
    rigid->mass = 12.8;
    rigid->moi = 0.0;
    rigid->offset_x = 0.0;
    rigid->offset_y = 0.0;
    rigid->offset_angl = 0.0;
    rigid->cog_ptc_id = 0.0;
    rigid->cogx = 0.0;
    rigid->cogy = 0.0; 
}

void get_rigid_num(SPH *sph)
{
    SPH_ARG *arg;
    //SPH_PARTICLE *particle;
    //SPH_RIGID *rigid;
    arg = sph->host_arg;
    //particle = sph->particle;
    //rigid = sph->host_rigid;

    std::string filename = arg->case_dir;
    filename += "/wedge.vtk";

    arg->rigid_ptc_num = 0;
    double x[3];

    vtkSmartPointer<vtkUnstructuredGridReader> reader = vtkSmartPointer<vtkUnstructuredGridReader>::New();
    reader->SetFileName(filename.c_str());
    reader->Update();
    
    vtkUnstructuredGrid *vtkdata;
    vtkdata = reader->GetOutput();
    for(vtkIdType i=0;i<vtkdata->GetNumberOfPoints();i++)
    {
        vtkdata->GetPoint(i,x);
        if(x[2]==0)
        {
            arg->rigid_ptc_num++;
        }
    }
    arg->ptc_num += arg->rigid_ptc_num;
}

void fluid_ptc_generate(SPH *sph)
{
    SPH_ARG *arg;
    SPH_PARTICLE *particle;
    //SPH_RIGID *rigid;
    arg = sph->host_arg;
    particle = sph->particle;
    //rigid = sph->host_rigid;
    int index = 0;
    for(int x=0;x<arg->fluid_xnum;x++)
    {
        for(int y=0;y<arg->fluid_ynum;y++)
        {
            particle->x[index] = (x+arg->wall_layer)*arg->ptc_dx;
            particle->y[index] = (y+arg->wall_layer)*arg->ptc_dx;
            particle->type[index] = 0;
            index++;
        }
    }
    for(int x=0;x<(arg->fluid_xnum+2*arg->wall_layer);x++)
    {
        for(int y=0;y<((int)(1.1*arg->fluid_y/arg->ptc_dx)+1);y++)
        {
            if(x < arg->wall_layer || x > (arg->fluid_xnum+arg->wall_layer-1))
            {
                particle->x[index] = x*arg->ptc_dx;
                particle->y[index] = y*arg->ptc_dx;
                particle->type[index] = -1;
                index++;
            }
            else if (y < arg->wall_layer)
            {
                particle->x[index] = x*arg->ptc_dx;
                particle->y[index] = y*arg->ptc_dx;
                particle->type[index] = -1;
                index++;
            }
        }
    }
    //if(index != (particle->fluid_ptc_num+particle->wall_ptc_num)) printf("\033[0;32;31m Error in %s:%d\033[m\n",__FILE__,__LINE__);
    assert(index == (arg->fluid_ptc_num + arg->wall_ptc_num));
}

void rigid_ptc_generate(SPH *sph)
{
    SPH_ARG *arg;
    SPH_PARTICLE *particle;
    //SPH_RIGID *rigid;
    arg = sph->host_arg;
    particle = sph->particle;
    //rigid = sph->host_rigid;

    std::string filename = arg->case_dir;
    filename += "/wedge.vtk"; 

    double x[3];
    int index = arg->fluid_ptc_num + arg->wall_ptc_num;

    vtkSmartPointer<vtkUnstructuredGridReader> reader = vtkSmartPointer<vtkUnstructuredGridReader>::New();
    reader->SetFileName(filename.c_str());
    reader->Update();
    
    vtkUnstructuredGrid *vtkdata;
    vtkdata = reader->GetOutput();
    for(vtkIdType i=0;i<vtkdata->GetNumberOfPoints();i++)
    {
        vtkdata->GetPoint(i,x);
        if(x[2]==0)
        {
            particle->x[index] = x[0] + arg->fluid_x/2.0;
            particle->y[index] = x[1] + arg->fluid_y*1.1;
            particle->type[index] = 1;
            index++;
        }
    }
    assert(index == arg->ptc_num);
}

void write_vtk(SPH *sph)
{
    SPH_ARG *arg;
    SPH_PARTICLE *particle;
    //SPH_RIGID *rigid;
    arg = sph->host_arg;
    particle = sph->particle;
    //rigid = sph->host_rigid;
    
    string filename = arg->case_dir; 
    filename += "/init.vtk";

    ofstream vtkfile;
    vtkfile.open(filename.c_str());

    vtkfile << "# vtk DataFile Version 3.0" << endl;
    vtkfile << "sph data" << endl;
    vtkfile << "ASCII" << endl;
    vtkfile << "DATASET UNSTRUCTURED_GRID" << endl;
    vtkfile << "POINTS " << arg->ptc_num << " " << "double" << endl;

    for(unsigned int i=0;i<arg->ptc_num;i++)
    {
        vtkfile << setiosflags(ios::scientific) << particle->x[i] << " " \
        << particle->y[i] << " " << 0.0 << endl;
    }

    vtkfile << "POINT_DATA" << " " << arg->ptc_num << endl;

    vtkfile << "SCALARS "<< "density double 1" << endl;
    vtkfile << "LOOKUP_TABLE DEFAULT" << endl;
    for(unsigned int i=0;i<arg->ptc_num;i++)
    {
        vtkfile << setiosflags(ios::scientific) << arg->ref_rho << endl;
    }
    vtkfile << "SCALARS "<< "pressure double 1" << endl;
    vtkfile << "LOOKUP_TABLE DEFAULT" << endl;
    for(unsigned int i=0;i<arg->ptc_num;i++)
    {
        vtkfile << setiosflags(ios::scientific) << 0.0 << endl;
    }
    vtkfile << "SCALARS " << "type int 1" << endl;
    vtkfile << "LOOKUP_TABLE DEFAULT" << endl;
    for(unsigned int i=0;i<arg->ptc_num;i++)
    {
        vtkfile << particle->type[i] << endl; 
    }
    vtkfile << "VECTORS "<< "velocity double" << endl;
    for(unsigned int i=0;i<arg->ptc_num;i++)
    {
        vtkfile << setiosflags(ios::scientific) << 0.0 <<" " << 0.0 << " " \
        << 0.0 << endl;

    }
    vtkfile << "VECTORS "<< "acceleration double" << endl;
    for(unsigned int i=0;i<arg->ptc_num;i++)
    {
        vtkfile << setiosflags(ios::scientific) << 0.0 <<" " << 0.0 << " " \
        << 0.0 << endl;
    }

    vtkfile.close();

}

void rigid_init(SPH *sph)
{
    SPH_ARG *arg;
    SPH_PARTICLE *particle;
    SPH_RIGID *rigid;
    arg = sph->host_arg;
    particle = sph->particle;
    rigid = sph->host_rigid;

    double tmp_cogx = 0.0;
    double tmp_cogy = 0.0;
    double tmp_r = 10000.0;

    for(int i=0;i<arg->ptc_num;i++)
    {
        if(particle->type[i] == 1)
        {
            tmp_cogx += particle->x[i];
            tmp_cogy += particle->y[i];
        }
    }
    tmp_cogx /= (double)arg->rigid_ptc_num;
    tmp_cogy /= (double)arg->rigid_ptc_num;

    for(int i=0;i<arg->ptc_num;i++)
    {
        if(particle->type[i] == 1)
        {
            if(tmp_r >= (pow((particle->x[i]-tmp_cogx),2)+pow((particle->y[i]-tmp_cogy),2)) )
            {
                tmp_r = pow((particle->x[i]-tmp_cogx),2)+pow((particle->y[i]-tmp_cogy),2);
                rigid->cog_ptc_id = i;
            }
        }
    }
    rigid->cogx = particle->x[rigid->cog_ptc_id];
    rigid->cogy = particle->y[rigid->cog_ptc_id];

    for(int i=0;i<arg->ptc_num;i++)
    {
        if(particle->type[i] == 1)
        {
            rigid->moi += (rigid->mass)*(pow((particle->x[i]-rigid->cogx),2)+pow((particle->y[i]-rigid->cogy),2));
        }
    }
    rigid->moi /= arg->rigid_ptc_num;
}