#include "hip/hip_runtime.h"
#include "PreProcess.cuh"

void ptc_info_init(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    particle = sph->particle;
    pair = sph->pair;

    double free_surf = PTC_SPACING*(FLUID_DEEPTH_NUM-1+4);

    if(sph->new_case_flag == 1)
    {
        for(int i=0;i<particle->total;i++)
        {
            particle->vx[i] = particle->vy[i] = 0;
            #ifndef ANALYSIS
                particle->pressure[i] = 0.0;
                particle->density[i] = REF_DENSITY;
                particle->mass[i] = PTC_MASS;
            #else
                if(particle->y[i] < free_surf)
                {
                    particle->pressure[i] = REF_DENSITY*GRAVITY_ACC*(free_surf-particle->y[i]);
                    particle->density[i] = particle->pressure[i]/pow(sph->c,2)+REF_DENSITY;
                    particle->mass[i] = particle->density[i]*pow(PTC_SPACING,2);
                }
                else
                {
                    particle->pressure[i] = 0.0;
                    particle->density[i] = REF_DENSITY;
                    particle->mass[i] = PTC_MASS;
                }
            #endif
        }
    }
}

void ptc_init(SPH *sph)
{
    ptc_info_init(sph);
    ptc_rigid_init(sph);
}