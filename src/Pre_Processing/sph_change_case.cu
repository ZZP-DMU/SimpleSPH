#include "hip/hip_runtime.h"
#include "SPH.cuh"
#include <assert.h>
#include <string>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <vtkUnstructuredGridReader.h>
#include <vtkSmartPointer.h>
#include <vtkType.h>
#include <vtkUnstructuredGrid.h>
#include <vtkPointSet.h>
#include <vtkDataSetReader.h>
#include <vtkPointData.h>
#include <vtkDataArray.h>
#include <vtkDoubleArray.h>

using namespace std;

int new_rigid_num(SPH *);
void rigid_ptc_generate(SPH *);
void rigid_init(SPH *);


int main(int argc,char *argv[])
{
    SPH_ARG arg;
    SPH_RIGID rigid;
    SPH_PARTICLE particle;
    SPH sph;
    sph.host_arg = &arg;
    sph.host_rigid = &rigid;
    sph.particle = &particle;

    SPH_ARG tmp_arg;
    SPH_RIGID tmp_rigid;
    SPH_PARTICLE tmp_particle;
    SPH tmp_sph;
    tmp_sph.host_arg = &tmp_arg;
    tmp_sph.host_rigid = &tmp_rigid;
    tmp_sph.particle = &tmp_particle;

    assert(argc == 3);
    sph.host_arg->case_dir = argv[1];
    //tmp_sph.host_arg->case_dir = argv[2];

    sph_read_info(&sph);
    particle.x = (double *)calloc(arg.ptc_num,sizeof(double));
    particle.y = (double *)calloc(arg.ptc_num,sizeof(double));
    particle.vx = (double *)calloc(arg.ptc_num,sizeof(double));
    particle.vy = (double *)calloc(arg.ptc_num,sizeof(double));
    particle.accx = (double *)calloc(arg.ptc_num,sizeof(double));
    particle.accy = (double *)calloc(arg.ptc_num,sizeof(double));
    particle.density = (double *)calloc(arg.ptc_num,sizeof(double));
    particle.pressure = (double *)calloc(arg.ptc_num,sizeof(double));
    particle.type = (int *)calloc(arg.ptc_num,sizeof(int));
    sph_read_vtk(&sph);

    memcpy(&tmp_arg,&arg,sizeof(SPH_ARG));
    memcpy(&tmp_rigid,&rigid,sizeof(SPH_RIGID));

    tmp_sph.host_arg->case_dir = argv[2];
    tmp_arg.rigid_ptc_num = new_rigid_num(&tmp_sph);
    tmp_arg.pair_volume = (int)(tmp_arg.ptc_num*32/tmp_arg.mesh_num);
    tmp_arg.ptc_num = tmp_arg.fluid_ptc_num + tmp_arg.wall_ptc_num + tmp_arg.rigid_ptc_num;
    tmp_particle.x = (double *)calloc(tmp_arg.ptc_num,sizeof(double));
    tmp_particle.y = (double *)calloc(tmp_arg.ptc_num,sizeof(double));
    tmp_particle.vx = (double *)calloc(tmp_arg.ptc_num,sizeof(double));
    tmp_particle.vy = (double *)calloc(tmp_arg.ptc_num,sizeof(double));
    tmp_particle.accx = (double *)calloc(tmp_arg.ptc_num,sizeof(double));
    tmp_particle.accy = (double *)calloc(tmp_arg.ptc_num,sizeof(double));
    tmp_particle.density = (double *)calloc(tmp_arg.ptc_num,sizeof(double));
    tmp_particle.pressure = (double *)calloc(tmp_arg.ptc_num,sizeof(double));
    tmp_particle.type = (int *)calloc(tmp_arg.ptc_num,sizeof(int)); 

    for(int i=0;i<arg.ptc_num;i++)
    {
        if(particle.type[i] != 1)
        {
            tmp_particle.x[i] = particle.x[i];
            tmp_particle.y[i] = particle.y[i];
            tmp_particle.vx[i] = particle.vx[i];
            tmp_particle.vy[i] = particle.vy[i];
            tmp_particle.accx[i] = particle.accx[i];
            tmp_particle.accy[i] = particle.accy[i];
            tmp_particle.density[i] = particle.density[i];
            tmp_particle.pressure[i] = particle.pressure[i];
            tmp_particle.type[i] = particle.type[i];
        }
    }
    rigid_ptc_generate(&tmp_sph);
    rigid_init(&tmp_sph);
    tmp_sph.host_arg->case_dir = argv[1];
    sph_save_last(&tmp_sph);
    
    return 0;
}

int new_rigid_num(SPH *sph)
{
    string filename = sph->host_arg->case_dir; 
    unsigned int tol=0;

    vtkSmartPointer<vtkUnstructuredGridReader> reader = vtkSmartPointer<vtkUnstructuredGridReader>::New();
    reader->SetFileName(filename.c_str());
    reader->Update();
    double x[3];
    
    vtkUnstructuredGrid *vtkdata;
    vtkdata = reader->GetOutput();
    for(vtkIdType i=0;i<vtkdata->GetNumberOfPoints();i++)
    {
        vtkdata->GetPoint(i,x);
        if(x[2]==0)
        {
            tol++;
        }
    }
    return tol;
}

void rigid_ptc_generate(SPH *sph)
{
    SPH_ARG *arg;
    SPH_PARTICLE *particle;
    SPH_RIGID *rigid;
    arg = sph->host_arg;
    particle = sph->particle;
    rigid = sph->host_rigid;

    std::string filename = arg->case_dir;

    double x[3];
    int index = arg->fluid_ptc_num + arg->wall_ptc_num;

    vtkSmartPointer<vtkUnstructuredGridReader> reader = vtkSmartPointer<vtkUnstructuredGridReader>::New();
    reader->SetFileName(filename.c_str());
    reader->Update();
    
    vtkUnstructuredGrid *vtkdata;
    vtkdata = reader->GetOutput();
    for(vtkIdType i=0;i<vtkdata->GetNumberOfPoints();i++)
    {
        vtkdata->GetPoint(i,x);
        if(x[2]==0)
        {
            particle->x[index] = x[0] + arg->fluid_x/2.0;
            particle->y[index] = x[1] + arg->fluid_y*1.1;
            particle->type[index] = 1;
            index++;
        }
    }
    assert(index == arg->ptc_num);
}

void rigid_init(SPH *sph)
{
    SPH_ARG *arg;
    SPH_PARTICLE *particle;
    SPH_RIGID *rigid;
    arg = sph->host_arg;
    particle = sph->particle;
    rigid = sph->host_rigid;

    double tmp_cogx = 0.0;
    double tmp_cogy = 0.0;
    double tmp_r = 10000.0;

    for(int i=0;i<arg->ptc_num;i++)
    {
        if(particle->type[i] == 1)
        {
            tmp_cogx += particle->x[i];
            tmp_cogy += particle->y[i];
        }
    }
    tmp_cogx /= (double)arg->rigid_ptc_num;
    tmp_cogy /= (double)arg->rigid_ptc_num;

    for(int i=0;i<arg->ptc_num;i++)
    {
        if(particle->type[i] == 1)
        {
            if(tmp_r >= (pow((particle->x[i]-tmp_cogx),2)+pow((particle->y[i]-tmp_cogy),2)) )
            {
                tmp_r = pow((particle->x[i]-tmp_cogx),2)+pow((particle->y[i]-tmp_cogy),2);
                rigid->cog_ptc_id = i;
            }
        }
    }
    rigid->cogx = particle->x[rigid->cog_ptc_id];
    rigid->cogy = particle->y[rigid->cog_ptc_id];

    for(int i=0;i<arg->ptc_num;i++)
    {
        if(particle->type[i] == 1)
        {
            rigid->moi += (arg->m/rigid->mass)*(pow((particle->x[i]-rigid->cogx),2)+pow((particle->y[i]-rigid->cogy),2));
        }
    }
}