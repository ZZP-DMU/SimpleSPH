#include "hip/hip_runtime.h"
#include "PreProcess.cuh"
using namespace std;

void ptc_rigid_generate(SPH *sph)
//void solid_ptc_generate(SPH_PARTICLE *particle)
{
    SPH_PARTICLE *particle;
    particle = sph->particle;
    
    std::string filename = "../data/preprocess/wedge.vtk";

    double x[3] = {0};
    unsigned int tol=0;
    tol = particle->fulid_ptc_num+particle->wall_ptc_num;

    vtkSmartPointer<vtkUnstructuredGridReader> reader = vtkSmartPointer<vtkUnstructuredGridReader>::New();
    reader->SetFileName(filename.c_str());
    reader->Update();
    
    vtkUnstructuredGrid *vtkdata;
    vtkdata = reader->GetOutput();
    for(vtkIdType i=0;i<vtkdata->GetNumberOfPoints();i++)
    {
        vtkdata->GetPoint(i,x);
        if(x[2]==0)
        {
            particle->x[tol] = x[0]+FLUID_DOMAIN_LENGTH/2.0;
            particle->y[tol] = x[1]+FLUID_DOMAIN_DEEPTH-8.0*PTC_SPACING;
            particle->type[tol] = 1;
            tol++;
        }
    }
}

unsigned int ptc_rigid_num(void)
{
    std::string filename = "../data/preprocess/wedge.vtk";

    double x[3];
    unsigned int tol=0;

    vtkSmartPointer<vtkUnstructuredGridReader> reader = vtkSmartPointer<vtkUnstructuredGridReader>::New();
    reader->SetFileName(filename.c_str());
    reader->Update();
    
    vtkUnstructuredGrid *vtkdata;
    vtkdata = reader->GetOutput();
    for(vtkIdType i=0;i<vtkdata->GetNumberOfPoints();i++)
    {
        vtkdata->GetPoint(i,x);
        if(x[2]==0)
        {
            tol++;
        }
    }
    return tol;
}

void ptc_rigid_init(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_RIGID *wedge;
    particle = sph->particle;
    wedge = sph->host_rigid;

    wedge->accx = wedge->accy = wedge->alpha = 0.0;
    wedge->mass = 12.8;
    //wedge->cog_ptc_id = 245939;

    if(sph->host_arg->new_case_flag == 1 || sph->host_arg->init_impac_flag == 1)
    {
        wedge->vx = wedge->vy = wedge->omega = 0.0;
        //wedge->cogx = FLUID_DOMAIN_LENGTH/2.0;
        //wedge->cogy = FLUID_DOMAIN_DEEPTH+4.0*PTC_SPACING+0.032;
        //wedge->cogx = particle->x[wedge->cog_ptc_id];
        //wedge->cogy = particle->y[wedge->cog_ptc_id];
        double tmp_cogx = 0;
        double tmp_cogy = 0;
        double r = 10000;
        //calculate the center of gravity of wedge
        for(unsigned int i=0;i<particle->total;i++)
        {
            if(particle->type[i] == 1)
            {
                tmp_cogx += particle->x[i]/(double)(particle->rigid_ptc_num);
                tmp_cogy += particle->y[i]/(double)(particle->rigid_ptc_num);
            }
        }
        for(unsigned int i=0;i<particle->total;i++)
        {
            if (particle->type[i] == 1)
            {
                if(r > (pow((tmp_cogx-particle->x[i]),2)+pow((tmp_cogy-particle->y[i]),2)))
                {
                    r = sqrt(pow((tmp_cogx-particle->x[i]),2)+pow((tmp_cogy-particle->y[i]),2));
                    wedge->cog_ptc_id = i;
                    wedge->cogx = particle->x[i];
                    wedge->cogy = particle->y[i];
                }
            }
        }
        printf("the center of gravity of rigid is:%d\n",wedge->cog_ptc_id);

        //calculate the moi of the wedge
        for(unsigned int i=0;i<particle->total;i++)
        {
            if(particle->type[i] == 1)
            {
                wedge->moi = (wedge->mass/particle->rigid_ptc_num)*(pow((particle->x[i]-wedge->cogx),2)+pow((particle->y[i]-wedge->cogy),2));
            }
        }
    }
    else if(sph->host_arg->new_case_flag == 0 && sph->host_arg->init_impac_flag == 0)
    {
        ifstream infofile;
        infofile.open("../data/preprocess/info.txt");

        std::string line;

        getline(infofile,line);//vx
        wedge->vx = stod(line.c_str());
        getline(infofile,line);//vy
        wedge->vy = stod(line.c_str());
        getline(infofile,line);//omega
        wedge->omega = stod(line.c_str());
        getline(infofile,line);//cogx
        wedge->cogx = stod(line.c_str());
        getline(infofile,line);//cogy
        wedge->cogy = stod(line.c_str());
        getline(infofile,line);//moi
        wedge->moi = stod(line.c_str());
        getline(infofile,line);//cog_ptc_id
        wedge->cog_ptc_id = stoi(line.c_str());

        infofile.close();
    }   
}

void ptc_read_vtk(SPH *sph)
{
    SPH_PARTICLE *particle;
    particle = sph->particle;

    std::string filename = "../data/preprocess/init.vtk";

    vtkSmartPointer<vtkUnstructuredGridReader> reader = vtkSmartPointer<vtkUnstructuredGridReader>::New();
    reader->SetFileName(filename.c_str());
    reader->ReadAllScalarsOn();
    reader->ReadAllVectorsOn();
    reader->Update();

    vtkUnstructuredGrid *vtkdata;
    vtkdata = reader->GetOutput();

    if(vtkdata->GetNumberOfPoints() != particle->total)
    {
        while(true)
        {
            std::cout << "here is ptc_read_vtk,the num of vtk file not equal to the defined ptc num" << endl;
        }
    }

    vtkPointData* pointdata = vtkdata->GetPointData();  

    vtkDataArray* pressure_array = pointdata->GetScalars("pressure");
    vtkDataArray* density_array = pointdata->GetScalars("density");
    vtkDataArray* mass_array = pointdata->GetScalars("mass");
    vtkDataArray* type_array = pointdata->GetScalars("type");
    vtkDataArray* velocity_array = pointdata->GetVectors("velocity");
    vtkDataArray* acc_array = pointdata->GetVectors("acceleration");

    vtkDoubleArray* pressure_data = vtkDoubleArray::SafeDownCast(pressure_array);
    vtkDoubleArray* density_data = vtkDoubleArray::SafeDownCast(density_array);
    vtkDoubleArray* mass_data = vtkDoubleArray::SafeDownCast(mass_array);
    vtkIntArray* type_data = vtkIntArray::SafeDownCast(type_array);
    vtkDoubleArray* velocity_data = vtkDoubleArray::SafeDownCast(velocity_array);
    vtkDoubleArray* acc_data = vtkDoubleArray::SafeDownCast(acc_array);

    if(pressure_data != nullptr && density_data != nullptr && mass_data != nullptr \
        && type_data != nullptr && velocity_data != nullptr && acc_data != nullptr)
    {
	    for(vtkIdType i=0;i<vtkdata->GetNumberOfPoints();i++)
	     {
            double p = 0.0;
            double d = 0.0;
            double m = 0.0;
            double v[3] = {0.0};
            double x[3] = {0.0};
            double a[3] = {0.0};
            int t = 0;
	    
            pressure_data->GetTuple(i,&p);
            density_data->GetTuple(i,&d);
            mass_data->GetTuple(i,&m);
            t=type_data->GetValue(i);
            velocity_data->GetTuple(i,v);
            vtkdata->GetPoint(i,x);
            acc_data->GetTuple(i,a);
            
            particle->x[i] = x[0];
            particle->y[i] = x[1];
            particle->pressure[i] = p;
            particle->density[i] = d;
            particle->mass[i] = m;
            particle->type[i] = t;
            particle->vx[i] = v[0];
            particle->vy[i] = v[1];
            particle->accx[i] = a[0];
            particle->accy[i] = a[1];
	    }
    }
    else
    {
        while (true)
        {
            cout << "some case are null" << endl;
        }
        
    }
}