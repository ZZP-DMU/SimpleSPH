#include "SPH.cuh"
#include <fstream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>

using namespace std;



void sph_init(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    SPH_CUDA temp_cuda;
    SPH_MESH mesh;
    particle = sph->particle;
    pair = sph->pair;
    kernel = sph->kernel;

    particle->fulid_ptc_num = FLUID_PTC_NUM;  //fluid ptc num
    particle->wall_ptc_num = WALL_PTC_NUM;    //wall ptc num
    particle->rigid_ptc_num = ptc_rigid_num(); //rigid ptc num
    //get all of the particle number
    particle->total = particle->fulid_ptc_num+particle->wall_ptc_num+particle->rigid_ptc_num; 

    /************stack is too small,so init data in heap***************/
    //particle data init
    particle->x = (double *)(calloc(particle->total,sizeof(double)));
    particle->y = (double *)(calloc(particle->total,sizeof(double)));
    particle->vx = (double *)(calloc(particle->total,sizeof(double)));
    particle->vy = (double *)(calloc(particle->total,sizeof(double)));
    particle->accx = (double *)(calloc(particle->total,sizeof(double)));
    particle->accy = (double *)(calloc(particle->total,sizeof(double)));
    particle->dif_density = (double *)(calloc(particle->total,sizeof(double)));
    particle->density = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_x = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_y = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_vx = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_vy = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_density = (double *)(calloc(particle->total,sizeof(double))); 
    particle->mass = (double *)(calloc(particle->total,sizeof(double))); 
    particle->w = (double *)(calloc(particle->total,sizeof(double)));
    particle->pressure = (double *)(calloc(particle->total,sizeof(double)));
    particle->type = (int *)(calloc(particle->total,sizeof(int)));  

    //kernel data init
    kernel->w = (double *)(calloc(30*particle->total,sizeof(double)));  //this code donnot use kernel value
    kernel->dwdx = (double *)(calloc(30*particle->total,sizeof(double)));
    kernel->dwdy = (double *)(calloc(30*particle->total,sizeof(double)));
   
    //pair data init
    pair->total = 0; 
    pair->i = (unsigned int *)(calloc(30*particle->total,sizeof(unsigned int)));
    pair->j = (unsigned int *)(calloc(30*particle->total,sizeof(unsigned int)));

    //mesh data init
    mesh = (SPH_MESH)(calloc(MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM,sizeof(int)));
    /*
    for(int i=0;i<MESH_DEEPTH_NUM;i++)
    {
        mesh[i] = (unsigned int **)(calloc(MESH_LENGTH_NUM,sizeof(unsigned int *)));
        for(int j=0;j<MESH_LENGTH_NUM;j++)
        {
            mesh[i][j] = (unsigned int *)(calloc(MESH_PTC_NUM,sizeof(unsigned int)));
        }
    }
    */
    
    sph->mesh = mesh;
    sph->d_time = DELTA_TIME;
    sph->c = ART_SOUND_VEL;
    sph->g = 0.0;
    sph->avg_time = 0.0;


    /*cuda mem alloc*/
    hipMalloc(&(sph->cuda),sizeof(SPH_CUDA));
    hipMalloc(&(temp_cuda.x),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.y),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.temp_x),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.temp_y),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.vx),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.vy),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.temp_vx),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.temp_vy),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.accx),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.accy),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.rho),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.drho),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.temp_rho),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.p),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.type),particle->total*sizeof(int));
    hipMalloc(&(temp_cuda.ptc_w),particle->total*sizeof(double));

    hipMalloc(&(temp_cuda.pair_w),32*particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.dwdx),32*particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.dwdy),32*particle->total*sizeof(double));
    hipMalloc(&(temp_cuda.pair_i),32*particle->total*sizeof(int));
    hipMalloc(&(temp_cuda.pair_j),32*particle->total*sizeof(int));
    hipMalloc(&(temp_cuda.mesh),MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int));
    hipMalloc(&(temp_cuda.mesh_count),MESH_DEEPTH_NUM*MESH_LENGTH_NUM*sizeof(int));

    hipMemcpy(sph->cuda,&temp_cuda,sizeof(SPH_CUDA),hipMemcpyHostToDevice);

    hipMemcpy(temp_cuda.x, particle->x, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda.y, particle->y, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda.vx, particle->vx, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda.vy, particle->vy, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda.type, particle->type, particle->total*sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda.rho, particle->density, particle->total*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda.accx, particle->accx, particle->total*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda.accy, particle->accx, particle->total*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda.drho, particle->dif_density, particle->total*sizeof(double), hipMemcpyHostToDevice);
    
    hipMemset(temp_cuda.p,0,particle->total*sizeof(double));
    hipMemset(temp_cuda.temp_x,0,particle->total*sizeof(double));
    hipMemset(temp_cuda.temp_y,0,particle->total*sizeof(double));
    hipMemset(temp_cuda.temp_vx,0,particle->total*sizeof(double));
    hipMemset(temp_cuda.temp_vy,0,particle->total*sizeof(double));
    hipMemset(temp_cuda.temp_rho,0,particle->total*sizeof(double));
    hipMemset(temp_cuda.ptc_w,0,particle->total*sizeof(double));

    hipMemset(temp_cuda.pair_w,0,32*particle->total*sizeof(double));
    hipMemset(temp_cuda.dwdx,0,32*particle->total*sizeof(double));
    hipMemset(temp_cuda.dwdy,0,32*particle->total*sizeof(double));
    hipMemset(temp_cuda.pair_i,0,32*particle->total*sizeof(int));
    hipMemset(temp_cuda.pair_j,0,32*particle->total*sizeof(int));
    hipMemset(temp_cuda.mesh,0,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int));
    hipMemset(temp_cuda.mesh_count,0,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*sizeof(int));


    cout << "run a new case or an old case(press 1 for new,0 for old)" << endl;
    cin >> sph->new_case_flag;

    if(sph->new_case_flag == 1)
    {
        sph->current_step = 0;
        sph->total_step = INIT_TIME_STEP;
    }
    else if(sph->new_case_flag == 0)
    {
        cout << "the sph current time step is: " << endl;
        cin >> sph->current_step;
        cout << "the total sph time step is: " << endl;
        cin >> sph->total_step;
    }

    cout << "run a init case or a dynamic case(press 1 for init,0 for dynamic)" << endl;
    cin >> sph->init_impac_flag;
    
    cout << "save the last step or not(press 1 ta save,0 for not)" << endl;
    cin >> sph->save_last_flag;

    ptc_generate(sph);
    ptc_init(sph);
}