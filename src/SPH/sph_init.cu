#include "SPH.cuh"
#include <fstream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>

using namespace std;

void sph_init(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    SPH_MESH *mesh;
    SPH_CUDA *temp_cuda;
    particle = sph->particle;
    pair = sph->pair;
    kernel = sph->kernel;
    mesh = sph->mesh;
    temp_cuda = sph->tmp_cuda;

    sph_read_info(sph);

    /************stack is too small,so init data in heap***************/
    //particle data init
    particle->x = (double *)(calloc(particle->total,sizeof(double)));
    particle->y = (double *)(calloc(particle->total,sizeof(double)));
    particle->vx = (double *)(calloc(particle->total,sizeof(double)));
    particle->vy = (double *)(calloc(particle->total,sizeof(double)));
    particle->accx = (double *)(calloc(particle->total,sizeof(double)));
    particle->accy = (double *)(calloc(particle->total,sizeof(double)));
    particle->dif_density = (double *)(calloc(particle->total,sizeof(double)));
    particle->density = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_x = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_y = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_vx = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_vy = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_density = (double *)(calloc(particle->total,sizeof(double))); 
    //particle->mass = (double *)(calloc(particle->total,sizeof(double))); 
    particle->w = (double *)(calloc(particle->total,sizeof(double)));
    particle->pressure = (double *)(calloc(particle->total,sizeof(double)));
    particle->type = (int *)(calloc(particle->total,sizeof(int)));  

    //kernel data init
    kernel->w = (double *)(calloc(32*particle->total,sizeof(double)));  //this code donnot use kernel value
    kernel->dwdx = (double *)(calloc(32*particle->total,sizeof(double)));
    kernel->dwdy = (double *)(calloc(32*particle->total,sizeof(double)));
   
    //pair data init
    pair->total = 0; 
    pair->i = (unsigned int *)(calloc(32*particle->total,sizeof(unsigned int)));
    pair->j = (unsigned int *)(calloc(32*particle->total,sizeof(unsigned int)));

    //mesh data init
    mesh->ptc = (int *)calloc(MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM,sizeof(int));
    mesh->count = (int *)calloc(MESH_DEEPTH_NUM*MESH_LENGTH_NUM,sizeof(int));
    sph->mesh = mesh;

    sph_read_vtk(sph);

    hipMalloc(&(sph->dev_arg),sizeof(SPH_ARG));
    hipMemcpy(sph->dev_arg,sph->host_arg,sizeof(SPH_ARG),hipMemcpyHostToDevice);

    hipMalloc(&(sph->dev_rigid),sizeof(SPH_RIGID));
    hipMemcpy(sph->dev_rigid,sph->host_rigid,sizeof(SPH_RIGID),hipMemcpyHostToDevice);

    /*cuda mem alloc*/
    hipMalloc(&(sph->cuda),sizeof(SPH_CUDA));
    hipMalloc(&(temp_cuda->x),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->y),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->temp_x),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->temp_y),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->vx),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->vy),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->temp_vx),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->temp_vy),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->accx),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->accy),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->rho),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->drho),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->temp_rho),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->p),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->type),particle->total*sizeof(int));
    hipMalloc(&(temp_cuda->ptc_w),particle->total*sizeof(double));

    hipMalloc(&(temp_cuda->pair_w),32*particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->dwdx),32*particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->dwdy),32*particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->pair_i),32*particle->total*sizeof(int));
    hipMalloc(&(temp_cuda->pair_j),32*particle->total*sizeof(int));
    hipMalloc(&(temp_cuda->pair_count),MESH_DEEPTH_NUM*MESH_LENGTH_NUM*sizeof(int));
    hipMalloc(&(temp_cuda->mesh),MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int));
    hipMalloc(&(temp_cuda->mesh_count),MESH_DEEPTH_NUM*MESH_LENGTH_NUM*sizeof(int));

    hipMemcpy(temp_cuda->x, particle->x, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->y, particle->y, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->vx, particle->vx, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->vy, particle->vy, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->type, particle->type, particle->total*sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->rho, particle->density, particle->total*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda->accx, particle->accx, particle->total*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda->accy, particle->accx, particle->total*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda->drho, particle->dif_density, particle->total*sizeof(double), hipMemcpyHostToDevice);
    
    hipMemset(temp_cuda->p,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->temp_x,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->temp_y,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->temp_vx,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->temp_vy,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->temp_rho,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->ptc_w,0,particle->total*sizeof(double));

    hipMemset(temp_cuda->pair_w,0,32*particle->total*sizeof(double));
    hipMemset(temp_cuda->dwdx,0,32*particle->total*sizeof(double));
    hipMemset(temp_cuda->dwdy,0,32*particle->total*sizeof(double));
    hipMemset(temp_cuda->pair_i,0,32*particle->total*sizeof(int));
    hipMemset(temp_cuda->pair_j,0,32*particle->total*sizeof(int));
    hipMemset(temp_cuda->pair_count,0,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*sizeof(int));
    hipMemset(temp_cuda->mesh,0,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int));
    hipMemset(temp_cuda->mesh_count,0,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*sizeof(int));

    hipMemcpy(sph->cuda,temp_cuda,sizeof(SPH_CUDA),hipMemcpyHostToDevice);
}
