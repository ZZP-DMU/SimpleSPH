#include "SPH.cuh"
#include <fstream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>

using namespace std;

void sph_init(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    SPH_MESH *mesh;
    SPH_CUDA *temp_cuda;
    SPH_ARG *arg;
    particle = sph->particle;
    pair = sph->pair;
    kernel = sph->kernel;
    mesh = sph->mesh;
    temp_cuda = sph->tmp_cuda;
    arg = sph->host_arg;

    sph_read_info(sph);

    /************stack is too small,so init data in heap***************/
    //particle data init
    particle->x = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->y = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->vx = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->vy = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->accx = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->accy = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->dif_density = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->density = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->temp_x = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->temp_y = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->temp_vx = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->temp_vy = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->temp_density = (double *)(calloc(arg->ptc_num,sizeof(double))); 
    //particle->mass = (double *)(calloc(arg->ptc_num,sizeof(double))); 
    particle->w = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->pressure = (double *)(calloc(arg->ptc_num,sizeof(double)));
    particle->type = (int *)(calloc(arg->ptc_num,sizeof(int)));  

    //kernel data init
    kernel->w = (double *)(calloc(arg->pair_list_num,sizeof(double)));  //this code donnot use kernel value
    kernel->dwdx = (double *)(calloc(arg->pair_list_num,sizeof(double)));
    kernel->dwdy = (double *)(calloc(arg->pair_list_num,sizeof(double)));
   
    //pair data init
    pair->total = 0; 
    pair->i = (unsigned int *)(calloc(arg->pair_list_num,sizeof(unsigned int)));
    pair->j = (unsigned int *)(calloc(arg->pair_list_num,sizeof(unsigned int)));

    //mesh data init
    mesh->ptc = (int *)calloc(sph->host_arg->mesh_num*sph->host_arg->mesh_volume,sizeof(int));
    mesh->count = (int *)calloc(sph->host_arg->mesh_num,sizeof(int));
    sph->mesh = mesh;

    sph_read_vtk(sph);
    if(arg->init_impac_flag == 0)
    {
        for(int i=0;i<arg->ptc_num;i++)
        {
            if(particle->y[i] <= arg->fluid_x)
            {
                particle->pressure[i] = arg->ref_rho*arg->g*(arg->fluid_x - particle->y[i]);
                particle->density[i] = arg->ref_rho + particle->pressure/(arg->c * arg->c);
            }
        }
    }

    hipMalloc(&(sph->dev_arg),sizeof(SPH_ARG));
    hipMemcpy(sph->dev_arg,sph->host_arg,sizeof(SPH_ARG),hipMemcpyHostToDevice);

    hipMalloc(&(sph->dev_rigid),sizeof(SPH_RIGID));
    hipMemcpy(sph->dev_rigid,sph->host_rigid,sizeof(SPH_RIGID),hipMemcpyHostToDevice);

    /*cuda mem alloc*/
    hipMalloc(&(sph->cuda),sizeof(SPH_CUDA));
    hipMalloc(&(temp_cuda->x),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->y),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->temp_x),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->temp_y),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->vx),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->vy),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->temp_vx),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->temp_vy),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->accx),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->accy),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->rho),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->drho),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->temp_rho),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->p),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->type),arg->ptc_num*sizeof(int));
    hipMalloc(&(temp_cuda->ptc_w),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->Lxx),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->Lxy),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->Lyx),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->Lyy),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->Lrho_x),arg->ptc_num*sizeof(double));
    hipMalloc(&(temp_cuda->Lrho_y),arg->ptc_num*sizeof(double));

    hipMalloc(&(temp_cuda->pair_w),arg->pair_list_num*sizeof(double));
    hipMalloc(&(temp_cuda->dwdx),arg->pair_list_num*sizeof(double));
    hipMalloc(&(temp_cuda->dwdy),arg->pair_list_num*sizeof(double));
    hipMalloc(&(temp_cuda->pair_i),arg->pair_list_num*sizeof(int));
    hipMalloc(&(temp_cuda->pair_j),arg->pair_list_num*sizeof(int));
    hipMalloc(&(temp_cuda->pair_count),sph->host_arg->pair_mesh_num*sizeof(int));
    hipMalloc(&(temp_cuda->mesh),sph->host_arg->mesh_num*sph->host_arg->mesh_volume*sizeof(int));
    hipMalloc(&(temp_cuda->mesh_count),sph->host_arg->mesh_num*sizeof(int));

    hipMemcpy(temp_cuda->x, particle->x, arg->ptc_num*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->y, particle->y, arg->ptc_num*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->vx, particle->vx, arg->ptc_num*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->vy, particle->vy, arg->ptc_num*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->type, particle->type, arg->ptc_num*sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->rho, particle->density, arg->ptc_num*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda->accx, particle->accx, arg->ptc_num*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda->accy, particle->accx, arg->ptc_num*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda->drho, particle->dif_density, arg->ptc_num*sizeof(double), hipMemcpyHostToDevice);
    
    hipMemset(temp_cuda->p,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->temp_x,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->temp_y,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->temp_vx,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->temp_vy,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->temp_rho,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->ptc_w,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->Lxx,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->Lxy,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->Lyx,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->Lyy,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->Lrho_x,0,arg->ptc_num*sizeof(double));
    hipMemset(temp_cuda->Lrho_y,0,arg->ptc_num*sizeof(double));

    hipMemset(temp_cuda->pair_w,0,arg->pair_list_num*sizeof(double));
    hipMemset(temp_cuda->dwdx,0,arg->pair_list_num*sizeof(double));
    hipMemset(temp_cuda->dwdy,0,arg->pair_list_num*sizeof(double));
    hipMemset(temp_cuda->pair_i,0,arg->pair_list_num*sizeof(int));
    hipMemset(temp_cuda->pair_j,0,arg->pair_list_num*sizeof(int));
    hipMemset(temp_cuda->pair_count,0,sph->host_arg->pair_mesh_num*sizeof(int));
    hipMemset(temp_cuda->mesh,0,sph->host_arg->mesh_num*sph->host_arg->mesh_volume*sizeof(int));
    hipMemset(temp_cuda->mesh_count,0,sph->host_arg->mesh_num*sizeof(int));

    hipMemcpy(sph->cuda,temp_cuda,sizeof(SPH_CUDA),hipMemcpyHostToDevice);
}
