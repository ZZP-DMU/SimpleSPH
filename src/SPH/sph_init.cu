#include "SPH.cuh"
#include <fstream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>

using namespace std;

void sph_init(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    SPH_MESH *mesh;
    SPH_CUDA *temp_cuda;
    particle = sph->particle;
    pair = sph->pair;
    kernel = sph->kernel;
    mesh = sph->mesh;
    temp_cuda = sph->tmp_cuda;

    

    particle->fulid_ptc_num = FLUID_PTC_NUM;  //fluid ptc num
    particle->wall_ptc_num = WALL_PTC_NUM;    //wall ptc num
    particle->rigid_ptc_num = ptc_rigid_num(); //rigid ptc num
    //get all of the particle number
    particle->total = particle->fulid_ptc_num+particle->wall_ptc_num+particle->rigid_ptc_num; 

    /************stack is too small,so init data in heap***************/
    //particle data init
    particle->x = (double *)(calloc(particle->total,sizeof(double)));
    particle->y = (double *)(calloc(particle->total,sizeof(double)));
    particle->vx = (double *)(calloc(particle->total,sizeof(double)));
    particle->vy = (double *)(calloc(particle->total,sizeof(double)));
    particle->accx = (double *)(calloc(particle->total,sizeof(double)));
    particle->accy = (double *)(calloc(particle->total,sizeof(double)));
    particle->dif_density = (double *)(calloc(particle->total,sizeof(double)));
    particle->density = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_x = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_y = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_vx = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_vy = (double *)(calloc(particle->total,sizeof(double)));
    particle->temp_density = (double *)(calloc(particle->total,sizeof(double))); 
    particle->mass = (double *)(calloc(particle->total,sizeof(double))); 
    particle->w = (double *)(calloc(particle->total,sizeof(double)));
    particle->pressure = (double *)(calloc(particle->total,sizeof(double)));
    particle->type = (int *)(calloc(particle->total,sizeof(int)));  

    //kernel data init
    kernel->w = (double *)(calloc(32*particle->total,sizeof(double)));  //this code donnot use kernel value
    kernel->dwdx = (double *)(calloc(32*particle->total,sizeof(double)));
    kernel->dwdy = (double *)(calloc(32*particle->total,sizeof(double)));
   
    //pair data init
    pair->total = 0; 
    pair->i = (unsigned int *)(calloc(32*particle->total,sizeof(unsigned int)));
    pair->j = (unsigned int *)(calloc(32*particle->total,sizeof(unsigned int)));

    //mesh data init
    mesh->ptc = (int *)calloc(MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM,sizeof(int));
    mesh->count = (int *)calloc(MESH_DEEPTH_NUM*MESH_LENGTH_NUM,sizeof(int));
    //mesh = (SPH_MESH)(calloc(MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM,sizeof(int)));
    /*
    for(int i=0;i<MESH_DEEPTH_NUM;i++)
    {
        mesh[i] = (unsigned int **)(calloc(MESH_LENGTH_NUM,sizeof(unsigned int *)));
        for(int j=0;j<MESH_LENGTH_NUM;j++)
        {
            mesh[i][j] = (unsigned int *)(calloc(MESH_PTC_NUM,sizeof(unsigned int)));
        }
    }
    */
    
    sph->mesh = mesh;
    
    /*sph->d_time = DELTA_TIME;
    sph->c = ART_SOUND_VEL;
    sph->g = 0.0;
    sph->avg_time = 0.0;*/

    //cuda arg mem alloc and cpy
    sph->host_arg->ptc_dx = PTC_SPACING;
    sph->host_arg->fluid_x = FLUID_DOMAIN_LENGTH;
    sph->host_arg->fluid_y = FLUID_DOMAIN_DEEPTH;
    sph->host_arg->mesh_x = TOL_DOMAIN_LENGTH;
    sph->host_arg->mesh_y = TOL_DOMAIN_DEEPTH;
    sph->host_arg->fluid_xnum = FLUID_LENGTH_NUM;
    sph->host_arg->fluid_ynum = FLUID_DEEPTH_NUM;
    sph->host_arg->mesh_dx = MESH_SPACING;
    sph->host_arg->mesh_xnum = MESH_LENGTH_NUM;
    sph->host_arg->mesh_ynum = MESH_DEEPTH_NUM;
    sph->host_arg->mesh_num = sph->host_arg->mesh_xnum*sph->host_arg->mesh_ynum;
    sph->host_arg->mesh_volume = MESH_PTC_NUM;
    sph->host_arg->m = PTC_MASS;
    sph->host_arg->g = GRAVITY_ACC;
    sph->host_arg->c = ART_SOUND_VEL;
    sph->host_arg->h = PTC_SML;
    sph->host_arg->alpha = ALPHA;
    sph->host_arg->dt = DELTA_TIME;
    sph->host_arg->ref_rho = REF_DENSITY;
    sph->host_arg->ptc_num = particle->total;
    sph->host_arg->pair_volume = 32*sph->host_arg->ptc_num/sph->host_arg->mesh_num;
    cout << "run a new case or an old case(press 1 for new,0 for old)" << endl;
    cin >> sph->host_arg->new_case_flag;
    if(sph->host_arg->new_case_flag == 1)
    {
        sph->host_arg->init_step = 0;
        sph->host_arg->total_step = INIT_TIME_STEP;
    }
    else if(sph->host_arg->new_case_flag == 0)
    {
        cout << "the sph current time step is: " << endl;
        cin >> sph->host_arg->init_step;
        cout << "the total sph time step is: " << endl;
        cin >> sph->host_arg->total_step;
    }
    cout << "run a init case or a dynamic case(press 1 for init,0 for dynamic)" << endl;
    cin >> sph->host_arg->init_impac_flag;
    cout << "save the last step or not(press 1 ta save,0 for not)" << endl;
    cin >> sph->host_arg->save_last_flag;
    //sph->host_arg->ptc_num = particle->total;
    sph->host_arg->pair_num = 0;
    sph->host_arg->lock = 1;
    sph->host_arg->tmp = 0;
    hipMalloc(&(sph->dev_arg),sizeof(SPH_ARG));
    hipMemcpy(sph->dev_arg,sph->host_arg,sizeof(SPH_ARG),hipMemcpyHostToDevice);

    ptc_generate(sph);
    ptc_init(sph);
    hipMalloc(&(sph->dev_rigid),sizeof(SPH_RIGID));
    hipMemcpy(sph->dev_rigid,sph->host_rigid,sizeof(SPH_RIGID),hipMemcpyHostToDevice);

    /*cuda mem alloc*/
    hipMalloc(&(sph->cuda),sizeof(SPH_CUDA));
    hipMalloc(&(temp_cuda->x),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->y),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->temp_x),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->temp_y),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->vx),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->vy),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->temp_vx),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->temp_vy),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->accx),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->accy),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->rho),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->drho),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->temp_rho),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->p),particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->type),particle->total*sizeof(int));
    hipMalloc(&(temp_cuda->ptc_w),particle->total*sizeof(double));

    hipMalloc(&(temp_cuda->pair_w),32*particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->dwdx),32*particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->dwdy),32*particle->total*sizeof(double));
    hipMalloc(&(temp_cuda->pair_i),32*particle->total*sizeof(int));
    hipMalloc(&(temp_cuda->pair_j),32*particle->total*sizeof(int));
    hipMalloc(&(temp_cuda->pair_count),MESH_DEEPTH_NUM*MESH_LENGTH_NUM*sizeof(int));
    hipMalloc(&(temp_cuda->mesh),MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int));
    hipMalloc(&(temp_cuda->mesh_count),MESH_DEEPTH_NUM*MESH_LENGTH_NUM*sizeof(int));

    hipMemcpy(temp_cuda->x, particle->x, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->y, particle->y, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->vx, particle->vx, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->vy, particle->vy, particle->total*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->type, particle->type, particle->total*sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(temp_cuda->rho, particle->density, particle->total*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda->accx, particle->accx, particle->total*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda->accy, particle->accx, particle->total*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_cuda->drho, particle->dif_density, particle->total*sizeof(double), hipMemcpyHostToDevice);
    
    hipMemset(temp_cuda->p,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->temp_x,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->temp_y,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->temp_vx,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->temp_vy,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->temp_rho,0,particle->total*sizeof(double));
    hipMemset(temp_cuda->ptc_w,0,particle->total*sizeof(double));

    hipMemset(temp_cuda->pair_w,0,32*particle->total*sizeof(double));
    hipMemset(temp_cuda->dwdx,0,32*particle->total*sizeof(double));
    hipMemset(temp_cuda->dwdy,0,32*particle->total*sizeof(double));
    hipMemset(temp_cuda->pair_i,0,32*particle->total*sizeof(int));
    hipMemset(temp_cuda->pair_j,0,32*particle->total*sizeof(int));
    hipMemset(temp_cuda->pair_count,0,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*sizeof(int));
    hipMemset(temp_cuda->mesh,0,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int));
    hipMemset(temp_cuda->mesh_count,0,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*sizeof(int));

    hipMemcpy(sph->cuda,temp_cuda,sizeof(SPH_CUDA),hipMemcpyHostToDevice);
}
