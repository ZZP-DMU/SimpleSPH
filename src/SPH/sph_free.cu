#include "SPH.cuh"

void sph_free(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    SPH_RIGID *wedge;
    particle = sph->particle;
    pair = sph->pair;
    kernel = sph->kernel;
    wedge = sph->host_rigid;

    hipFree(sph->cuda);
    free(particle->x);
    free(particle->y);
    free(particle->vx);
    free(particle->vy);
    free(particle->accx);
    free(particle->accy);
    free(particle->density);
    free(particle->dif_density);
    free(particle->pressure);
    free(particle->type);
    free(particle->mass);

    free(kernel->w);
    free(kernel->dwdx);
    free(kernel->dwdy);
    
    free(pair->i);
    free(pair->j);
    //free(mesh);
}