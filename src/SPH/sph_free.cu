#include "SPH.cuh"

void sph_free(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    SPH_CUDA *cuda;
    particle = sph->particle;
    pair = sph->pair;
    kernel = sph->kernel;
    cuda = sph->tmp_cuda;
    //hipMemcpy(&cuda,sph->cuda,sizeof(SPH_CUDA),hipMemcpyDeviceToHost);
    
    hipFree(sph->dev_arg);
    hipFree(sph->dev_rigid);
    hipFree(cuda->x);
    hipFree(cuda->y);
    hipFree(cuda->vx);
    hipFree(cuda->vy);
    hipFree(cuda->Lxx);
    hipFree(cuda->Lxy);
    hipFree(cuda->Lyx);
    hipFree(cuda->Lyy);
    hipFree(cuda->Lrho_x);
    hipFree(cuda->Lrho_y);
    hipFree(cuda->pair_count);
    hipFree(cuda->temp_x);
    hipFree(cuda->temp_y);
    hipFree(cuda->temp_vx);
    hipFree(cuda->temp_vy);
    hipFree(cuda->rho);
    hipFree(cuda->drho);
    hipFree(cuda->temp_rho);
    hipFree(cuda->accx);
    hipFree(cuda->accy);
    hipFree(cuda->p);
    hipFree(cuda->type);
    hipFree(cuda->ptc_w);
    hipFree(cuda->pair_i);
    hipFree(cuda->pair_j);
    hipFree(cuda->pair_w);
    hipFree(cuda->dwdx);
    hipFree(cuda->dwdy);
    hipFree(cuda->mesh);
    hipFree(cuda->mesh_count);
    hipFree(sph->cuda);

    free(particle->x);
    free(particle->y);
    free(particle->vx);
    free(particle->vy);
    free(particle->accx);
    free(particle->accy);
    free(particle->density);
    free(particle->dif_density);
    free(particle->pressure);
    free(particle->type);
    //free(particle->mass);

    free(kernel->w);
    free(kernel->dwdx);
    free(kernel->dwdy);
    
    free(pair->i);
    free(pair->j);
    //free(mesh);
}