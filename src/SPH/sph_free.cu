#include "SPH.cuh"

void sph_free(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    SPH_RIGID *wedge;
    SPH_CUDA cuda;
    particle = sph->particle;
    pair = sph->pair;
    kernel = sph->kernel;
    wedge = sph->host_rigid;
    hipMemcpy(&cuda,sph->cuda,sizeof(SPH_CUDA),hipMemcpyDeviceToHost);

    hipFree(cuda->x);
    hipFree(cuda->y);
    hipFree(cuda->vx);
    hipFree(cuda->vy);
    hipFree(cuda->temp_x);
    hipFree(cuda->temp_y);
    hipFree(cuda->temp_vx);
    hipFree(cuda->temp_vy);
    hipFree(cuda->rho);
    hipFree(cuda->drho);
    hipFree(cuda->temp_rho);
    hipFree(cuda->accx);
    hipFree(cuda->accy);
    hipFree(cuda->p);
    hipFree(cuda->type);
    hipFree(cuda->ptc_w);
    hipFree(cuda->pair_i);
    hipFree(cuda->pair_j);
    hipFree(cuda->pair_w);
    hipFree(cuda->dwdx);
    hipFree(cuda->dwdy);
    hipFree(cuda->mesh);
    hipFree(cuda->mesh_count);
    hipFree(sph->cuda);

    free(particle->x);
    free(particle->y);
    free(particle->vx);
    free(particle->vy);
    free(particle->accx);
    free(particle->accy);
    free(particle->density);
    free(particle->dif_density);
    free(particle->pressure);
    free(particle->type);
    free(particle->mass);

    free(kernel->w);
    free(kernel->dwdx);
    free(kernel->dwdy);
    
    free(pair->i);
    free(pair->j);
    //free(mesh);
}