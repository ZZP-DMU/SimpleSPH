#include "hip/hip_runtime.h"
#include "SPH.cuh"
#include <fstream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>
using namespace std;

__device__ int count;

int main(void)
{ 
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh = NULL;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.rigid = &wedge;
    sph.mesh = mesh;

    sph_init(&sph); 
    hipSetDevice(0);

    double *dev_x;
    double *dev_y;
    double *dev_vx;
    double *dev_vy;
    double *dev_rho;
    double *dev_p;
    double *dev_pair_i;
    double *dev_pair_j;
    int *dev_type;
    /*
    dev_pair_i,dev_pair_j,dev_pair_accx,dev_pair_accy,dev_pair_drho = NULL;
    */
    int *dev_mesh =NULL;

    int temp = 0;
    int temp_1 = 0;
    int host_count = 0;


    CUDA_CHECK(hipMalloc((double**)&dev_x,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_y,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_vx,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_vy,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_rho,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_p,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((int**)&dev_type,particle.total*sizeof(double)));

    CUDA_CHECK(hipMalloc((double**)&dev_pair_i,32*particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_j,32*particle.total*sizeof(double)));
    /*
    CUDA_CHECK(hipMalloc((double**)&dev_pair_accx,size*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_accy,size*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_drho,size*sizeof(double)));
    */

    CUDA_CHECK(hipMalloc((int**)&dev_mesh,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int)));

       
   // sph_avg_time(&sph);
        CUDA_CHECK(cudaMencpy(&count,&host_count,sizeof(int),hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dev_x, particle.x, particle.total*sizeof(double), hipMemcpyHostToDevice)); 
        CUDA_CHECK(hipMemcpy(dev_y, particle.y, particle.total*sizeof(double), hipMemcpyHostToDevice)); 
        CUDA_CHECK(hipMemcpy(dev_vx, particle.vx, particle.total*sizeof(double), hipMemcpyHostToDevice)); 
        CUDA_CHECK(hipMemcpy(dev_vy, particle.vy, particle.total*sizeof(double), hipMemcpyHostToDevice)); 
        CUDA_CHECK(hipMemcpy(dev_type, particle.type, particle.total*sizeof(double), hipMemcpyHostToDevice)); 
        CUDA_CHECK(hipMemcpy(dev_rho, particle.density, particle.total*sizeof(double), hipMemcpyHostToDevice)); 
        dim3 block(64,64);
        dim3 grid(MESH_LENGTH_NUM,MESH_DEEPTH_NUM);
        sph_mesh_cuda<<<384,160>>>(dev_x,dev_y,dev_mesh,particle.total);
        hipDeviceSynchronize();
        sph_nnps_cuda<<<grid,block>>>(dev_mesh,dev_x,dev_y,dev_type,dev_pair_i,dev_pair_j);
        hipMemcpy(&host_count,&count,sizeof(int),hipMemcpyDeviceToHost);
        
        //__global__ void sph_nnps_cuda(int *mesh,double *x,double *y,int *type,int *pair_i,int *pair_j)
        //CUDA_CHECK(hipMemcpy(mesh, dev_mesh, sizeof(int)*MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM,hipMemcpyDeviceToHost));
        printf("gpu find :%d \n",host_count);
/*
    string filename = "../data/postprocess/vtk/sph"; 
    filename += to_string(sph.current_step/PRINT_TIME_STEP);
    filename += ".vtk";

    ofstream vtkfile;
    vtkfile.open(filename.c_str());

    vtkfile << "# vtk DataFile Version 3.0" << endl;
    vtkfile << "sph data" << endl;
    vtkfile << "ASCII" << endl;
    vtkfile << "DATASET UNSTRUCTURED_GRID" << endl;
    vtkfile << "POINTS " << particle.total << " " << "double" << endl;

    for(unsigned int i=0;i<MESH_DEEPTH_NUM;i++)
    {
        for(unsigned int j=0;j<MESH_LENGTH_NUM;j++)
        {
            temp = sph.mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH_NUM*(MESH_PTC_NUM-1)];
            for(unsigned int k=0;k<temp;k++)
            {
                temp_1 = sph.mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH_NUM*k];
                vtkfile << setiosflags(ios::scientific) << particle.x[temp_1] << " " \
                << particle.y[temp_1] << " " << 0.0 << endl;
            }
        }
    }
    vtkfile.close();

*/
    sph_free(&sph);
    return 0;
}
