#include "SPH.cuh"
#include <fstream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>
using namespace std;

__constant__  int c = ART_SOUND_VEL;
__constant__  int rho_0 = REF_DENSITY;
__constant__  int mesh_lnum = MESH_LENGTH_NUM;
__constant__  int mesh_dnum = MESH_DEEPTH_NUM;
__constant__  int mesh_pnum = MESH_PTC_NUM;
__constant__  int mesh_spacing = MESH_SPACING;

int main(void)
{ 
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh = NULL;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.rigid = &wedge;
    sph.mesh = mesh;

    sph_init(&sph); 
    hipSetDevice(0);

    double *dev_x;
    double *dev_y;
    double *dev_vx;
    double *dev_vy;
    double *dev_rho;
    double *dev_p;
    /*
    dev_pair_i,dev_pair_j,dev_pair_accx,dev_pair_accy,dev_pair_drho = NULL;
    */
    int *dev_mesh =NULL;

    int temp = 0;

    CUDA_CHECK(hipMalloc((double**)&dev_x,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_y,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_vx,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_vy,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_rho,particle.total*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_p,particle.total*sizeof(double)));
/*
    CUDA_CHECK(hipMalloc((double**)&dev_pair_i,size*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_j,size*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_accx,size*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_accy,size*sizeof(double)));
    CUDA_CHECK(hipMalloc((double**)&dev_pair_drho,size*sizeof(double)));
    */

    CUDA_CHECK(hipMalloc((int**)&dev_mesh,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int)));

       
   // sph_avg_time(&sph);
    for(sph.current_step;sph.current_step<sph.total_step;sph.current_step++)
    {
        hipMemcpy((void *)dev_x, (void *)particle.x, particle.total*sizeof(double), hipMemcpyHostToDevice); 
        hipMemcpy((void *)dev_y, (void *)particle.y, particle.total*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy((void *)dev_vx, (void *)particle.vx, particle.total*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy((void *)dev_vy, (void *)particle.vy, particle.total*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy((void *)dev_rho, (void *)particle.density, particle.total*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy((void *)dev_p, (void *)particle.pressure, particle.total*sizeof(double), hipMemcpyHostToDevice);

        ptc_mesh_cuda<<<384,160>>>(dev_x,dev_y,dev_mesh,particle.total);
        hipMemcpy(mesh, dev_mesh, MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM,hipMemcpyDeviceToHost);


    string filename = "../data/postprocess/vtk/sph"; 
    filename += to_string(sph.current_step/PRINT_TIME_STEP);
    filename += ".vtk";

    ofstream vtkfile;
    vtkfile.open(filename.c_str());

    vtkfile << "# vtk DataFile Version 3.0" << endl;
    vtkfile << "sph data" << endl;
    vtkfile << "ASCII" << endl;
    vtkfile << "DATASET UNSTRUCTURED_GRID" << endl;
    vtkfile << "POINTS " << particle.total << " " << "double" << endl;

    for(unsigned int i=0;i<MESH_DEEPTH_NUM;i++)
    {
        for(unsigned int j=0;j<MESH_LENGTH_NUM;j++)
        {
            for(unsigned int k=0;k<MESH_PTC_NUM;k++)
            temp = mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH*k];
            vtkfile << setiosflags(ios::scientific) << particle.x[temp] << " " \
            << particle.y[temp] << " " << 0.0 << endl;
        }
    }
    vtkfile.close();



        /*
        if(sph.current_step%PRINT_TIME_STEP == 0)
        {
            sph_save_single(&sph);
        }
        //calculate and integration
        sph_time_integral(&sph); 
        sph_save_rigid(&sph);
        ptc_info(&sph);
        sph_avg_time(&sph);
        */
    }
    //sph_save_last(&sph);
    sph_free(&sph);
    return 0;
}
