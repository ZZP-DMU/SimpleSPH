#include "hip/hip_runtime.h"
#include "SPH.cuh"

int main(void)
{
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh;
    SPH_ARG arg;
    SPH_CUDA tmp_cuda;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.host_rigid = &wedge;
    sph.host_arg = &arg;
    sph.mesh = &mesh;
    sph.tmp_cuda = &tmp_cuda;

    hipSetDevice(0);
    sph_init(&sph); 

    //define the seed for ptc data structure
    dim3 ptc_block(256);
    dim3 ptc_grid((int)(sph.particle->total/256)+1);
    //define the seed for mesh data structure
    dim3 mesh_block(32,32);
    dim3 mesh_grid(MESH_LENGTH_NUM,MESH_DEEPTH_NUM);
    //define the seed for pair data structre
    dim3 pair_block(128);
    dim3 pair_grid(MESH_LENGTH_NUM,MESH_DEEPTH_NUM);


    //SPH_CUDA cuda;
    //SPH_ARG tmp_arg;
    //hipMemcpy(&cuda,sph.cuda,sizeof(SPH_CUDA),hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    //int *host_pair_count = (int *)calloc(sph.host_arg->mesh_num,sizeof(int));
    //int id = 0;
    //int *cpu_pair_i;
    //int *cpu_pair_j;
    //hipMalloc(&cpu_pair_i,sizeof(int)*32*sph.particle->total);
    //hipMalloc(&cpu_pair_j,sizeof(int)*32*sph.particle->total);

    for(sph.host_arg->init_step;sph.host_arg->init_step<sph.host_arg->total_step;sph.host_arg->init_step++)
    {
        printf("current step is:%d ",sph.host_arg->init_step);
        
        sph_mesh_cuda<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
        hipDeviceSynchronize();
        sph_nnps_cuda<<<mesh_grid,mesh_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();
        sph_kernel_cuda<<<pair_grid,pair_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();
        sph_governing_cuda<<<pair_grid,pair_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();
        sph_predict_cuda<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();
        sph_dummy_cuda<<<pair_grid,pair_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();
        
        sph_mesh_cuda<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
        hipDeviceSynchronize();
        sph_nnps_cuda<<<mesh_grid,mesh_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();
        sph_kernel_cuda<<<pair_grid,pair_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();
        sph_governing_cuda<<<pair_grid,pair_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();
        sph_correct_cuda<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();
        sph_dummy_cuda<<<pair_grid,pair_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();

        if(sph.host_arg->init_step%400 == 0)
        {
            hipMemcpy(sph.particle->x,sph.tmp_cuda->x,sizeof(double)*sph.particle->total,hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->y,sph.tmp_cuda->y,sizeof(double)*sph.particle->total,hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->vx,sph.tmp_cuda->vx,sizeof(double)*sph.particle->total,hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->vy,sph.tmp_cuda->vy,sizeof(double)*sph.particle->total,hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->accx,sph.tmp_cuda->accx,sizeof(double)*sph.particle->total,hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->accy,sph.tmp_cuda->accy,sizeof(double)*sph.particle->total,hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->density,sph.tmp_cuda->rho,sizeof(double)*sph.particle->total,hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->pressure,sph.tmp_cuda->p,sizeof(double)*sph.particle->total,hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

            sph_save_single(&sph);
        }

    
        hipError_t sph_error = hipGetLastError();
        printf("%s\n",hipGetErrorName(sph_error));
    }

    sph_free(&sph);
    hipDeviceReset();
    return 0;
}

__global__ void sph_predict_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id < arg->ptc_num)
    {
        cuda->temp_x[id] = cuda->x[id];
        cuda->temp_y[id] = cuda->y[id];
        cuda->temp_vx[id] = cuda->vx[id];
        cuda->temp_vy[id] = cuda->vy[id];
        cuda->temp_rho[id] = cuda->rho[id];
        if(cuda->type[id] == 0)
        {
            cuda->x[id] += cuda->vx[id]*arg->dt*0.5;
            cuda->y[id] += cuda->vy[id]*arg->dt*0.5;
            cuda->vx[id] += cuda->accx[id]*arg->dt*0.5;
            cuda->vy[id] += cuda->accy[id]*arg->dt*0.5;
            cuda->rho[id] += cuda->drho[id]*arg->dt*0.5;
            if(cuda->rho[id] < arg->ref_rho) cuda->rho[id] = arg->ref_rho;
            cuda->p[id] = arg->c*arg->c*(cuda->rho[id] - arg->ref_rho);
        }
        else
        {
            cuda->p[id] = 0.0;
            cuda->rho[id] = 0.0;
            cuda->vx[id] = 0.0;
            cuda->vy[id] = 0.0;
            cuda->rho[id] = arg->ref_rho;
        }
    }
}

__global__ void sph_correct_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id < arg->ptc_num)
    {
        if(cuda->type[id] == 0)
        {
            cuda->x[id] = cuda->temp_x[id] + cuda->vx[id]*arg->dt;
            cuda->y[id] = cuda->temp_y[id] + cuda->vy[id]*arg->dt;
            cuda->vx[id] = cuda->temp_vx[id] + cuda->accx[id]*arg->dt;
            cuda->vy[id] = cuda->temp_vy[id] + cuda->accy[id]*arg->dt;
            cuda->rho[id] = cuda->temp_rho[id] + cuda->drho[id]*arg->dt;
            if(cuda->rho[id] < arg->ref_rho) cuda->rho[id] = arg->ref_rho;
            cuda->p[id] = arg->c*arg->c*(cuda->rho[id] - arg->ref_rho);
        }
        else
        {
            cuda->p[id] = 0.0;
            cuda->rho[id] = 0.0;
            cuda->vx[id] = 0.0;
            cuda->vy[id] = 0.0;
            cuda->rho[id] = arg->ref_rho;
        }
    }
}
