#include "hip/hip_runtime.h"
#include "SPH.cuh"
#include <assert.h>

int main(int argc,char *argv[])
{
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh;
    SPH_ARG arg;
    SPH_CUDA tmp_cuda;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.host_rigid = &wedge;
    sph.host_arg = &arg;
    sph.mesh = &mesh;
    sph.tmp_cuda = &tmp_cuda;
    assert("argc == 2");
    arg.case_dir = argv[1];

    hipSetDevice(0);
    sph_init(&sph);

    // define the seed for ptc data structure
    dim3 ptc_block(256);
    dim3 ptc_grid((int)(sph.host_arg->ptc_num / 256) + 1);
    // define the seed for mesh data structure
    dim3 mesh_block(32, 32);
    dim3 mesh_grid(sph.host_arg->mesh_xnum, sph.host_arg->mesh_ynum);
    // define the seed for pair data structre
    dim3 pair_block(128);
    dim3 pair_grid(sph.host_arg->mesh_xnum, sph.host_arg->mesh_ynum);

    // SPH_CUDA cuda;
    // SPH_ARG tmp_arg;
    // hipMemcpy(&cuda,sph.cuda,sizeof(SPH_CUDA),hipMemcpyDeviceToHost);
    // hipDeviceSynchronize();
    // int *host_pair_count = (int *)calloc(sph.host_arg->mesh_num,sizeof(int));
    // int id = 0;
    // int *cpu_pair_i;
    // int *cpu_pair_j;
    // hipMalloc(&cpu_pair_i,sizeof(int)*32*sph.host_arg->ptc_num);
    // hipMalloc(&cpu_pair_j,sizeof(int)*32*sph.host_arg->ptc_num);

    for (sph.host_arg->init_step; sph.host_arg->init_step < sph.host_arg->total_step; sph.host_arg->init_step++)
    {
        printf("current step is:%d ", sph.host_arg->init_step);
        sph_mesh_cuda<<<ptc_grid, ptc_block>>>(sph.cuda, sph.dev_arg);
        if (sph.host_arg->init_step % sph.host_arg->print_step == 2 && sph.host_arg->init_impac_flag == 0)
        {
            sph_write_csv(&sph);
        }
        hipDeviceSynchronize();
        sph_nnps_cuda<<<mesh_grid, mesh_block>>>(sph.cuda, sph.dev_arg, sph.dev_rigid);
        if (sph.host_arg->init_step % sph.host_arg->print_step == 1)
        {
            sph_save_single(&sph);
        }
        hipDeviceSynchronize();
        sph_kernel_cuda<<<pair_grid, pair_block>>>(sph.cuda, sph.dev_arg, sph.dev_rigid);
        hipDeviceSynchronize();
        sph_governing_cuda<<<pair_grid, pair_block>>>(sph.cuda, sph.dev_arg, sph.dev_rigid);
        hipDeviceSynchronize();
        sph_predict_cuda<<<ptc_grid, ptc_block>>>(sph.cuda, sph.dev_arg, sph.dev_rigid);
        hipDeviceSynchronize();
        sph_dummy_cuda<<<pair_grid, pair_block>>>(sph.cuda, sph.dev_arg, sph.dev_rigid);
        hipDeviceSynchronize();

        sph_mesh_cuda<<<ptc_grid, ptc_block>>>(sph.cuda, sph.dev_arg);
        hipDeviceSynchronize();
        sph_nnps_cuda<<<mesh_grid, mesh_block>>>(sph.cuda, sph.dev_arg, sph.dev_rigid);
        hipDeviceSynchronize();
        sph_kernel_cuda<<<pair_grid, pair_block>>>(sph.cuda, sph.dev_arg, sph.dev_rigid);
        hipDeviceSynchronize();
        sph_governing_cuda<<<pair_grid, pair_block>>>(sph.cuda, sph.dev_arg, sph.dev_rigid);
        hipDeviceSynchronize();
        sph_correct_cuda<<<ptc_grid, ptc_block>>>(sph.cuda, sph.dev_arg, sph.dev_rigid);
        hipDeviceSynchronize();
        sph_dummy_cuda<<<pair_grid, pair_block>>>(sph.cuda, sph.dev_arg, sph.dev_rigid);
        hipDeviceSynchronize();
        if(sph.host_arg->init_impac_flag == 0)
        {
            sph_rigid_cuda<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
            hipDeviceSynchronize();
        }

        if (sph.host_arg->init_step % sph.host_arg->print_step == 0)
        {
            hipMemcpy(sph.particle->x, sph.tmp_cuda->x, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->y, sph.tmp_cuda->y, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->vx, sph.tmp_cuda->vx, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->vy, sph.tmp_cuda->vy, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->accx, sph.tmp_cuda->accx, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->accy, sph.tmp_cuda->accy, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->density, sph.tmp_cuda->rho, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            hipMemcpy(sph.particle->pressure, sph.tmp_cuda->p, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            if(sph.host_arg->init_impac_flag == 0)
            {
                hipMemcpy(sph.host_rigid,sph.dev_rigid,sizeof(SPH_RIGID),hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
            }
            
        }
        hipError_t sph_error = hipGetLastError();
        printf("%s\n", hipGetErrorName(sph_error));
    }

    //save the last frame
    hipMemcpy(sph.particle->x, sph.tmp_cuda->x, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(sph.particle->y, sph.tmp_cuda->y, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(sph.particle->vx, sph.tmp_cuda->vx, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(sph.particle->vy, sph.tmp_cuda->vy, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(sph.particle->accx, sph.tmp_cuda->accx, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(sph.particle->accy, sph.tmp_cuda->accy, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(sph.particle->density, sph.tmp_cuda->rho, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(sph.particle->pressure, sph.tmp_cuda->p, sizeof(double) * sph.host_arg->ptc_num, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(sph.particle->type,sph.tmp_cuda->type,sizeof(int)*sph.host_arg->ptc_num,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(sph.host_rigid,sph.dev_rigid,sizeof(SPH_RIGID),hipMemcpyDeviceToHost);
    sph_save_last(&sph);

    sph_free(&sph);
    hipDeviceReset();
    return 0;
}

__global__ void sph_predict_cuda(SPH_CUDA *cuda, SPH_ARG *arg, SPH_RIGID *rigid)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < arg->ptc_num)
    {
        cuda->temp_x[id] = cuda->x[id];
        cuda->temp_y[id] = cuda->y[id];
        cuda->temp_vx[id] = cuda->vx[id];
        cuda->temp_vy[id] = cuda->vy[id];
        cuda->temp_rho[id] = cuda->rho[id];
        if (cuda->type[id] == 0)
        {
            cuda->x[id] += cuda->vx[id] * arg->dt * 0.5;
            cuda->y[id] += cuda->vy[id] * arg->dt * 0.5;
            cuda->vx[id] += cuda->accx[id] * arg->dt * 0.5;
            cuda->vy[id] += cuda->accy[id] * arg->dt * 0.5;
            cuda->rho[id] += cuda->drho[id] * arg->dt * 0.5;
            if (cuda->rho[id] < arg->ref_rho)
                cuda->rho[id] = arg->ref_rho;
            cuda->p[id] = arg->c * arg->c * (cuda->rho[id] - arg->ref_rho);
        }
        else
        {
            cuda->p[id] = 0.0;
            cuda->rho[id] = 0.0;
            cuda->vx[id] = 0.0;
            cuda->vy[id] = 0.0;
            cuda->rho[id] = arg->ref_rho;
        }
    }
}

__global__ void sph_correct_cuda(SPH_CUDA *cuda, SPH_ARG *arg, SPH_RIGID *rigid)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < arg->ptc_num)
    {
        if (cuda->type[id] == 0)
        {
            cuda->x[id] = cuda->temp_x[id] + cuda->vx[id] * arg->dt;
            cuda->y[id] = cuda->temp_y[id] + cuda->vy[id] * arg->dt;
            cuda->vx[id] = cuda->temp_vx[id] + cuda->accx[id] * arg->dt;
            cuda->vy[id] = cuda->temp_vy[id] + cuda->accy[id] * arg->dt;
            cuda->rho[id] = cuda->temp_rho[id] + cuda->drho[id] * arg->dt;
            if (cuda->rho[id] < arg->ref_rho)
                cuda->rho[id] = arg->ref_rho;
            cuda->p[id] = arg->c * arg->c * (cuda->rho[id] - arg->ref_rho);
        }
        else if(cuda->type[id] == 1)
        {
            cuda->p[id] = 0.0;
            cuda->rho[id] = 0.0;
            cuda->vx[id] = 0.0;
            cuda->vy[id] = 0.0;
            cuda->rho[id] = arg->ref_rho;
            cuda->x[id] = cuda->temp_x[id] + arg->dt*(rigid->vx - rigid->omega*(cuda->y[id]-rigid->cogy));
            cuda->y[id] = cuda->temp_y[id] + arg->dt*(rigid->vy + rigid->omega*(cuda->x[id]-rigid->cogx));
        }
        else if(cuda->type[id] == -1)
        {
            cuda->p[id] = 0.0;
            cuda->rho[id] = 0.0;
            cuda->vx[id] = 0.0;
            cuda->vy[id] = 0.0;
            cuda->rho[id] = arg->ref_rho;
        }
    }
}

__global__ void sph_rigid_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    __shared__ double accx;
    __shared__ double accy;
    __shared__ double alpha;
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id == 0)
    {
        rigid->accx = 0.0;
        rigid->accy = -arg->g;
        rigid->alpha = 0.0;
        rigid->cogx = cuda->x[rigid->cog_ptc_id];
        rigid->cogy = cuda->y[rigid->cog_ptc_id];
    }
    if(threadIdx.x == 0)
    {
        accx = 0.0;
        accy = 0.0;
        alpha = 0.0;
    }
    __syncthreads();
    if(id < arg->ptc_num)
    {
        if(cuda->type[id] == 1)
        {
            accx += cuda->accx[id]*arg->m/rigid->mass;
            accy += cuda->accy[id]*arg->m/rigid->mass;
            alpha += (cuda->accy[id]*(cuda->x[id]-rigid->cogx)-cuda->accx[id]*(cuda->y[id]-rigid->cogy))*arg->m/rigid->moi;
        }   
    }
    __syncthreads();
    if( threadIdx.x == 0)
    {
        atomicAdd(&(rigid->accx),accx);
        atomicAdd(&(rigid->accy),accy);
        atomicAdd(&(rigid->alpha),alpha);
        atomicAdd(&(rigid->vx),accx*arg->dt);
        atomicAdd(&(rigid->vy),accy*arg->dt);
        atomicAdd(&(rigid->omega),alpha*arg->dt);
    }
    __syncthreads();
}