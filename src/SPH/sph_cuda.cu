#include "hip/hip_runtime.h"
#include "SPH.cuh"
#include <fstream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>
using namespace std;


int main(void)
{
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh = NULL;
    SPH_CUDA *cuda;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.rigid = &wedge;
    sph.cuda = cuda;
    sph.mesh = mesh;
    
    hipSetDevice(0);
    sph_init(&sph); 
    
/*
    double *dev_rigid = NULL;
    double host_rigid[10];

    host_rigid[VX] = sph.rigid->vx;
    host_rigid[VY] = sph.rigid->vy;
    host_rigid[ACCX] = sph.rigid->accx;
    host_rigid[ACCY] = sph.rigid->accy;
    host_rigid[OMEGA] = sph.rigid->omega;
    host_rigid[R_ALPHA] = sph.rigid->alpha;
    host_rigid[MASS] = sph.rigid->mass;
    host_rigid[MOI] = sph.rigid->moi;
    host_rigid[COGX] = sph.rigid->cogx;
    host_rigid[COGY] = sph.rigid->cogy;

    int host_count;
    int *dev_count;

    
    CUDA_CHECK(hipMalloc((int**)&dev_mesh,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int)));

    CUDA_CHECK(hipMalloc((int**)&dev_count,sizeof(int)));
    CUDA_CHECK(hipMemset(dev_count,0,sizeof(int)));

    CUDA_CHECK(hipMalloc((double**)&dev_rigid,sizeof(double)*10));
*/
/*
    //define the seed for ptc data structure
    dim3 ptc_block(256);
    dim3 ptc_grid((int)(sph.particle->total/256)+1);
    //define the seed for mesh data structure
    dim3 mesh_block(32);
    dim3 mesh_grid(MESH_LENGTH_NUM,MESH_DEEPTH_NUM);
    //define the seed for pair data structre
    dim3 pair_block(512);
    dim3 pair_grid((int)(sph.particle->total/16)+1);
    */

    // sph_avg_time(&sph);
    //for(sph.current_step;sph.current_step<sph.total_step;sph.current_step++)
    //{    
    /*---------------------------------------Predict Step---------------------------------------Predict Step---------------------------------------Predict Step---------------------------------------Predict Step---------------------------------------Predict Step---------------------------------------Predict Step*/
    /*
        //CUDA_CHECK(hipMemset(dev_mesh,0,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int)));
        sph_mesh_cuda<<<ptc_grid,ptc_block>>>(dev_x,dev_y,dev_accx,dev_accy,dev_drho,dev_type,dev_mesh,dev_count,sph.particle->total);
        CUDA_CHECK(hipDeviceSynchronize());
        //__global__ void sph_mesh_cuda(double *x,double *y,double *accx,double *accy,double *drho,int *type,int *mesh,int ptc_num)

        sph_nnps_cuda<<<mesh_grid,mesh_block>>>(dev_mesh,dev_x,dev_y,dev_type,dev_pair_i,dev_pair_j,dev_count);
        //__global__ void sph_nnps_cuda(int *mesh,double *x,double *y,int *type,int *pair_i,int *pair_j)
        CUDA_CHECK(hipDeviceSynchronize());

        sph_kernel_cuda<<<pair_grid,pair_block>>>(dev_x,dev_y,dev_kernel_w,dev_kernel_dwdx,dev_kernel_dwdy,dev_w,dev_pair_i,dev_pair_j,dev_count);
        CUDA_CHECK(hipDeviceSynchronize());
        //sph_kernel_cuda(double *x,double *y,double *w,double *dwdx,double *dwdy,doubel *ptc_w,int *pair_i,int *pair_j,int pair_num)

        sph_governing_cuda<<<pair_grid,pair_block>>>(dev_x,dev_y,dev_vx,dev_vy,dev_rho,dev_p,dev_type,dev_pair_i,dev_pair_j,dev_kernel_dwdx,dev_kernel_dwdy,dev_accx,dev_accy,dev_drho,dev_rigid,dev_count,sph.particle->total);
        CUDA_CHECK(hipDeviceSynchronize());
        //__global__ void sph_governing_cuda(double * x, double * y, double * vx, double * vy, double * rho, double * p, int * type, int * pair_i, int * pair_j, double * dwdx, double * dwdy, double * accx, double * accy, double * drho, double *rigid, int pair_num, int ptc_num)
        
        sph_predict_cuda<<<ptc_grid,ptc_block>>>(dev_x,dev_y,dev_temp_x,dev_temp_y,dev_vx,dev_vy,dev_temp_vx,dev_temp_vy,dev_accx,dev_accy,dev_rho,dev_temp_rho,dev_drho,dev_p,dev_type,sph.particle->total);
        CUDA_CHECK(hipDeviceSynchronize());
        //__global__ void sph_predict_cuda(double *x,double *y,double *temp_x,double *temp_y,double *vx,double *vy,double *temp_vx,double *temp_vy,double *accx,double *accy,double *rho,double *temp_rho,double *drho,int ptc_num)
    */
    /*---------------------------------------Correct Step---------------------------------------Correct Step---------------------------------------Correct Step---------------------------------------Correct Step---------------------------------------Correct Step---------------------------------------Correct Step*/
    /*
        sph_mesh_cuda<<<ptc_grid,ptc_block>>>(dev_x,dev_y,dev_accx,dev_accy,dev_drho,dev_type,dev_mesh,dev_count,sph.particle->total);
        CUDA_CHECK(hipDeviceSynchronize());

        sph_nnps_cuda<<<mesh_grid,mesh_block>>>(dev_mesh,dev_x,dev_y,dev_type,dev_pair_i,dev_pair_j,dev_count);
        //__global__ void sph_nnps_cuda(int *mesh,double *x,double *y,int *type,int *pair_i,int *pair_j)
        CUDA_CHECK(hipDeviceSynchronize());

        sph_kernel_cuda<<<pair_grid,pair_block>>>(dev_x,dev_y,dev_kernel_w,dev_kernel_dwdx,dev_kernel_dwdy,dev_w,dev_pair_i,dev_pair_j,dev_count);
        CUDA_CHECK(hipDeviceSynchronize());
        //sph_kernel_cuda(double *x,double *y,double *w,double *dwdx,double *dwdy,double *w,int *pair_i,int *pair_j,int pair_num)

        sph_governing_cuda<<<pair_grid,pair_block>>>(dev_x,dev_y,dev_vx,dev_vy,dev_rho,dev_p,dev_type,dev_pair_i,dev_pair_j,dev_kernel_dwdx,dev_kernel_dwdy,dev_accx,dev_accy,dev_drho,dev_rigid,dev_count,sph.particle->total);
        CUDA_CHECK(hipDeviceSynchronize());
        //__global__ void sph_governing_cuda(double * x, double * y, double * vx, double * vy, double * rho, double * p, int * type, int * pair_i, int * pair_j, double * dwdx, double * dwdy, double * accx, double * accy, double * drho, double *rigid, int pair_num, int ptc_num)
        
        sph_correct_cuda<<<ptc_grid,ptc_block>>>(dev_x,dev_y,dev_temp_x,dev_temp_y,dev_vx,dev_vy,dev_temp_vx,dev_temp_vy,dev_accx,dev_accy,dev_rho,dev_temp_rho,dev_drho,dev_p,dev_type,sph.particle->total);
        CUDA_CHECK(hipDeviceSynchronize());
        //__global__ void sph_predict_cuda(double *x,double *y,double *temp_x,double *temp_y,double *vx,double *vy,double *temp_vx,double *temp_vy,double *accx,double *accy,double *rho,double *temp_rho,double *drho,int ptc_num)
         
        
        sph_dummy_cuda<<<pair_grid,pair_block>>>(dev_x,dev_y,dev_vx,dev_vy,dev_p,dev_rho,dev_w,dev_kernel_w,dev_pair_i,dev_pair_j,dev_type,dev_rigid,dev_count);
        CUDA_CHECK(hipDeviceSynchronize());
        //__global__ void sph_dummy_cuda(double *vx,double *vy,double *p,double *rho,double *ptc_w,double *pair_w,int *pair_i,int *pair_j,int *type,double *rigid,int *pair_num)
        CUDA_CHECK(hipMemcpy(sph.particle->x,dev_x,sph.particle->total*sizeof(double),hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(sph.particle->y,dev_y,sph.particle->total*sizeof(double),hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(sph.particle->vx,dev_vy,sph.particle->total*sizeof(double),hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(sph.particle->vx,dev_vx,sph.particle->total*sizeof(double),hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(sph.particle->accx,dev_accx,sph.particle->total*sizeof(double),hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(sph.particle->accy,dev_accy,sph.particle->total*sizeof(double),hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(sph.particle->pressure,dev_p,sph.particle->total*sizeof(double),hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(sph.particle->density,dev_rho,sph.particle->total*sizeof(double),hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&host_count,dev_count,sizeof(int),hipMemcpyDeviceToHost));
        sph_save_single(&sph);
        
        printf("current step is:%d pair_num is:%d \n",sph.current_step,host_count);
    }
    */
    sph_free(&sph);
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_vx);
    hipFree(dev_vy);
    hipFree(dev_accx);
    hipFree(dev_accy);
    hipFree(dev_rho);
    hipFree(dev_drho);
    hipFree(dev_w);
    hipFree(dev_p);
    hipFree(dev_type);
    hipFree(dev_temp_x);
    hipFree(dev_temp_y);
    hipFree(dev_temp_vx);
    hipFree(dev_temp_vy);
    hipFree(dev_temp_rho);
    hipFree(dev_pair_i);
    hipFree(dev_pair_j);
    hipFree(dev_kernel_w);
    hipFree(dev_kernel_dwdx);
    hipFree(dev_kernel_dwdy);
    hipFree(dev_mesh);
    hipFree(dev_count);
    hipFree(dev_rigid);
    hipDeviceReset();
    return 0;
}

__global__ void sph_predict_cuda(double *x,double *y,double *temp_x,double *temp_y,double *vx,double *vy,double *temp_vx,double *temp_vy,double *accx,double *accy,double *rho,double *temp_rho,double *drho,double *p,int *type,int ptc_num)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= ptc_num )return;

    if(type[id] == 0)
    {
        temp_x[id] = x[id];
        temp_y[id] = y[id];
        temp_vx[id] = vx[id];
        temp_vy[id] = vy[id];
        temp_rho[id] = rho[id];

        x[id] += vx[id]*dev_dt*0.5;
        y[id] += vy[id]*dev_dt*0.5;
        vx[id] += accx[id]*dev_dt*0.5;
        vy[id] += accy[id]*dev_dt*0.5;
        rho[id] += drho[id]*dev_dt*0.5;
        if(rho[id] < REF_DENSITY) rho[id]=REF_DENSITY;
    }
    /*
    else
    {
        vx[id] = 0.0;
        vy[id] = 0.0;
        p[id] = 0.0;
    }
    */
}


__global__ void sph_correct_cuda(double *x,double *y,double *temp_x,double *temp_y,double *vx,double *vy,double *temp_vx,double *temp_vy,double *accx,double *accy,double *rho,double *temp_rho,double *drho,double *p,int *type,int ptc_num)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= ptc_num )return; 

    if(type[id] == 0)
    {
        x[id] = temp_x[id] + vx[id]*dev_dt;
        y[id] = temp_y[id] + vy[id]*dev_dt;
        vx[id] = temp_vx[id] + accx[id]*dev_dt;
        vy[id] = temp_vy[id] + accy[id]*dev_dt;
        rho[id] = temp_rho[id] + drho[id]*dev_dt;
        if(rho[id] < REF_DENSITY) rho[id]=REF_DENSITY;
    }
    else
    {
        vx[id] = 0.0;
        vy[id] = 0.0;
        p[id] = 0.0;
    }
}
/*
        CUDA_CHECK(hipMemcpy(sph.mesh,dev_mesh,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int),hipMemcpyDeviceToHost));
    string filename = "../data/postprocess/vtk/sph"; 
    filename += to_string(sph.current_step/PRINT_TIME_STEP);
    filename += ".vtk";

    ofstream vtkfile;
    vtkfile.open(filename.c_str());

    vtkfile << "# vtk DataFile Version 3.0" << endl;
    vtkfile << "sph data" << endl;
    vtkfile << "ASCII" << endl;
    vtkfile << "DATASET UNSTRUCTURED_GRID" << endl;
    vtkfile << "POINTS " << sph.particle->total << " " << "double" << endl;

    for(unsigned int i=0;i<MESH_DEEPTH_NUM;i++)
    {
        for(unsigned int j=0;j<MESH_LENGTH_NUM;j++)
        {
            temp = sph.mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH_NUM*(MESH_PTC_NUM-1)];
            for(unsigned int k=0;k<temp;k++)
            {
                temp_1 = sph.mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH_NUM*k];
                vtkfile << setiosflags(ios::scientific) << sph.particle->x[temp_1] << " " \
                << sph.particle->y[temp_1] << " " << 0.0 << endl;
            }
        }
    }
    vtkfile.close();*/