#include "hip/hip_runtime.h"
#include "SPH.cuh"

__global__ void check_ptc(SPH_CUDA *cuda,SPH_ARG *arg)
{
    const int id = threadIdx.x + threadIdx.y*blockDim.x;
    if(id >= arg->ptc_num)return;
    printf("%lf %lf\n",cuda->x[id],cuda->y[id]);
}

__global__ void check_pair(SPH_CUDA *cuda,SPH_ARG *arg)
{
    //double dx=0.0;
    //double dy=0.0;
    //double q=0.0;
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= arg->pair_num) return;

    /*
    dx = cuda->x[cuda->pair_i[id]] - cuda->x[cuda->pair_j[id]];
    dy = cuda->y[cuda->pair_i[id]] - cuda->y[cuda->pair_j[id]];
    q = sqrt(dx*dx+dy*dy)/arg->h;
    if(q > 2.0) printf("error !!!\n");*/

    
    //if(id == 0)printf("the pair num is:%d\n",arg->pair_num);
    for(int i=0;i<arg->pair_num;i++)
    {
        if(cuda->pair_i[id] == cuda->pair_i[i] && cuda->pair_j[id]==cuda->pair_j[i] && id!=i)
        {
            if(cuda->pair_i[id]!=0 && cuda->pair_j[id]!=0)
            {
                //printf("type1 index_1:%d index_2:%d pair_i:%d pair_j:%d\n",id,i,cuda->pair_i[id],cuda->pair_j[id]);
            }
            atomicAdd(&(arg->tmp),1);
        }
        else if(cuda->pair_i[id] == cuda->pair_j[i] && cuda->pair_j[id]==cuda->pair_i[i])
        {
            if(id == i)
            {
                //printf("type2 nnps error !!\n");
                if(cuda->pair_i[id]!=0 && cuda->pair_j[id]!=0)
                {
                    //printf("type2 index_1:%d index_2:%d pair_i:%d pair_j:%d\n",id,i,cuda->pair_i[id],cuda->pair_j[id]);
                }
            }
            else 
            {
                //printf("here is same pair\n");
                if(cuda->pair_i[id]!=0 && cuda->pair_j[id]!=0)
                {
                    //printf("type3 index_1:%d index_2:%d pair_i:%d pair_j:%d\n",id,i,cuda->pair_i[id],cuda->pair_j[id]);
                }
            }
            atomicAdd(&(arg->tmp),1);
        }
    }
}

__global__ void check_mesh(SPH_CUDA *cuda,SPH_ARG *arg)
{
    int mid = 0;
    int id = 0;
    const int mesh_id = blockIdx.x + blockIdx.y* gridDim.x;
    for(int i=0;i<cuda->mesh_count[mesh_id];i++)
    {
        id = cuda->mesh[i*arg->mesh_num + mesh_id];
        if(cuda->y[id] < arg->domain_y && cuda->y[id] >= 0.0)
        {
            mid = __double2int_rz(cuda->y[id]/arg->mesh_dx)*arg->mesh_xnum;
        }
        else
        {
            mid = (arg->mesh_ynum - 1)*arg->mesh_xnum;
        }
        if(cuda->x[id] < arg->domain_x && cuda->x[id] >= 0.0)
        {
            mid += __double2int_rz(cuda->x[id]/arg->mesh_dx);
            if(mid == mesh_id) atomicAdd(&(arg->tmp),1);
            //if(mid != mesh_id) printf("mid:%d mesh_id:%d id:%d x:%lf y:%lf\n",mid,mesh_id,id,cuda->x[id],cuda->y[id]);

        }
        else
        {
            mid += arg->mesh_xnum - 1;
            if(mid == mesh_id) atomicAdd(&(arg->tmp),1);
            //if(mid != mesh_id) printf("mid:%d mesh_id:%d id:%d x:%lf y:%lf\n",mid,mesh_id,id,cuda->x[id],cuda->y[id]);
        }
    }
    cuda->mesh_count[mesh_id] = 0;
    /*
    if(cuda->mesh_count[mesh_id]!=0)
    {
        printf("%d %d\n",mesh_id,cuda->mesh_count[mesh_id]);
    }*/
    /*
    if(cuda->mesh_count[mesh_id] != 0)
    {
        printf("mesh id is:%d ptc in mesh is:%d they are:",mesh_id,cuda->mesh_count[mesh_id]);
        for(int i=0;i<cuda->mesh_count[mesh_id];i++)
        {
            printf("%d",cuda->mesh[mesh_id+i*arg->mesh_num]);
        }
        printf("\n");
    }*/
}

int main(void)
{
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh;
    SPH_ARG arg;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.host_rigid = &wedge;
    sph.host_arg = &arg;
    sph.mesh = &mesh;

    hipSetDevice(0);
    sph_init(&sph); 

    //define the seed for ptc data structure
    dim3 ptc_block(256);
    dim3 ptc_grid((int)(sph.particle->total/256)+1);
    //define the seed for mesh data structure
    dim3 mesh_block(MESH_LENGTH_NUM);
    dim3 mesh_grid(MESH_DEEPTH_NUM);
    //define the seed for pair data structre
    dim3 pair_block(512);
    dim3 pair_grid((int)(sph.particle->total/16)+1);

    //int *host_mesh;
    //int *host_mesh_count;
    SPH_CUDA cuda;
    SPH_ARG tmp_arg;
    hipMemcpy(&cuda,sph.cuda,sizeof(SPH_CUDA),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for(int i=0;i<100;i++)
    {
        printf("current step is:%d\n",i);
        //check_ptc<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
        //hipDeviceSynchronize();
        sph_mesh_cuda<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
        hipDeviceSynchronize();
        //check_mesh<<<mesh_grid,1>>>(sph.cuda,sph.dev_arg);
        //hipDeviceSynchronize();
        sph_nnps_cuda<<<1,1>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();
        //check_pair<<<(int)(250000/1024)+1,1024>>>(sph.cuda,sph.dev_arg);
        //hipDeviceSynchronize();

        /*hipMemcpy(sph.mesh->ptc,cuda.mesh,sizeof(int)*sph.host_arg->mesh_num*sph.host_arg->mesh_volume,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(sph.mesh->count,cuda.mesh_count,sizeof(int)*sph.host_arg->mesh_num,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        for(int i=0;i<sph.host_arg->mesh_num;i++)
        {
            printf("%d\n",sph.mesh->count[i]);
        }*/
        //sph_nnps_cpu(&sph);

        /*
        hipMemcpy(sph.pair->i,cuda.pair_i,sizeof(int)*32*sph.particle->total,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(sph.pair->j,cuda.pair_j,sizeof(int)*32*sph.particle->total,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(&tmp_arg,sph.dev_arg,sizeof(SPH_ARG),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        for(int i=0;i<tmp_arg.pair_num;i++)
        {
            printf("id:%d i:%d j:%d\n",i,sph.pair->i[i],sph.pair->j[i]);
        }*/
        
        
        //printf("the total same pair num is:%d \n",tmp_arg.tmp);
        
        /*
        host_mesh = (int *)malloc(sizeof(int)*sph.host_arg->mesh_num*sph.host_arg->mesh_volume);
        host_mesh_count = (int *)malloc(sizeof(int)*sph.host_arg->mesh_num);

        hipMemcpy(host_mesh,cuda.mesh,sizeof(int)*sph.host_arg->mesh_num*sph.host_arg->mesh_volume,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(host_mesh_count,cuda.mesh_count,sizeof(int)*sph.host_arg->mesh_num,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        for(int j=0;j<sph.host_arg->mesh_num;j++)
        {
            //if(host_mesh_count[j]!=0) printf("error!!!!!!\n");
            printf("mesh id is:%d mesh num is:%d they are:",j,host_mesh_count[j]);
            for(int k=0;k<host_mesh_count[j];k++)
            {
                printf("%d,",host_mesh[j+k*sph.host_arg->mesh_num]);
            }
            printf("\n");
        }*/
        
    }

    /*
    for(int i=0;i<sph.host_arg->mesh_num;i++)
    {
        printf("mesh id is:%d mesh num is:%d\n",i,host_mesh_count[i]);
    }*/

    sph_free(&sph);
    hipDeviceReset();
    return 0;
}

