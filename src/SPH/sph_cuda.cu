#include "hip/hip_runtime.h"
#include "SPH.cuh"

__global__ void check_ptc(SPH_CUDA *cuda,SPH_ARG *arg)
{
    const int id = threadIdx.x + threadIdx.y*blockDim.x;
    if(id >= arg->ptc_num)return;
    printf("%lf %lf\n",cuda->x[id],cuda->y[id]);
}

__global__ void check_pair(SPH_CUDA *cuda,SPH_ARG *arg)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= arg->pair_num) return;
    if(id == 0)printf("the pair num is:%d\n",arg->pair_num);
    for(int i=0;i<arg->pair_num;i++)
    {
        if(cuda->pair_i[id] == cuda->pair_i[i] && cuda->pair_j[id]==cuda->pair_j[i] && id!=i)
        {
            if(cuda->pair_i[id]!=0 && cuda->pair_j[id]!=0)
            {
                printf("type1 index_1:%d index_2:%d pair_i:%d pair_j:%d\n",id,i,cuda->pair_i[id],cuda->pair_j[id]);
            }
            atomicAdd(&(arg->tmp),1);
        }
        else if(cuda->pair_i[id] == cuda->pair_j[i] && cuda->pair_j[id]==cuda->pair_i[i])
        {
            if(id == i)
            {
                //printf("type2 nnps error !!\n");
                if(cuda->pair_i[id]!=0 && cuda->pair_j[id]!=0)
                {
                    printf("type2 index_1:%d index_2:%d pair_i:%d pair_j:%d\n",id,i,cuda->pair_i[id],cuda->pair_j[id]);
                }
            }
            else 
            {
                //printf("here is same pair\n");
                if(cuda->pair_i[id]!=0 && cuda->pair_j[id]!=0)
                {
                    printf("type3 index_1:%d index_2:%d pair_i:%d pair_j:%d\n",id,i,cuda->pair_i[id],cuda->pair_j[id]);
                }
            }
            atomicAdd(&(arg->tmp),1);
        }
    }
}

__global__ void check_mesh(SPH_CUDA *cuda,SPH_ARG *arg)
{
    const int mesh_id = blockIdx.x + blockIdx.y* gridDim.x;
    if(cuda->mesh_count[mesh_id]!=0)
    {
        printf("%d %d\n",mesh_id,cuda->mesh_count[mesh_id]);
    }
    /*
    if(cuda->mesh_count[mesh_id] != 0)
    {
        printf("mesh id is:%d ptc in mesh is:%d they are:",mesh_id,cuda->mesh_count[mesh_id]);
        for(int i=0;i<cuda->mesh_count[mesh_id];i++)
        {
            printf("%d",cuda->mesh[mesh_id+i*arg->mesh_num]);
        }
        printf("\n");
    }*/
}

int main(void)
{
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh = NULL;
    SPH_ARG arg;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.host_rigid = &wedge;
    sph.host_arg = &arg;
    sph.mesh = mesh;

    hipSetDevice(0);
    sph_init(&sph); 

    //define the seed for ptc data structure
    dim3 ptc_block(256);
    dim3 ptc_grid((int)(sph.particle->total/256)+1);
    //define the seed for mesh data structure
    dim3 mesh_block(32,32);
    dim3 mesh_grid(MESH_LENGTH_NUM,MESH_DEEPTH_NUM);
    //define the seed for pair data structre
    dim3 pair_block(512);
    dim3 pair_grid((int)(sph.particle->total/16)+1);

    int *host_mesh;
    int *host_mesh_count;
    SPH_CUDA cuda;
    SPH_ARG tmp_arg;
    hipMemcpy(&cuda,sph.cuda,sizeof(SPH_CUDA),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for(int i=0;i<1;i++)
    {
        //printf("current step is:%d\n",i);
        //check_ptc<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
        //hipDeviceSynchronize();
        sph_mesh_cuda<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
        hipDeviceSynchronize();
        //check_mesh<<<mesh_grid,1>>>(sph.cuda,sph.dev_arg);
        //hipDeviceSynchronize();
        //sph_nnps_cuda<<<mesh_grid,mesh_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        //hipDeviceSynchronize();
        //check_pair<<<(int)(250000/1024)+1,1024>>>(sph.cuda,sph.dev_arg);
        //hipDeviceSynchronize();

        //hipMemcpy(&tmp_arg,sph.dev_arg,sizeof(SPH_ARG),hipMemcpyDeviceToHost);
        //printf("the total same pair num is:%d \n",tmp_arg.tmp);
        
        host_mesh = (int *)malloc(sizeof(int)*sph.host_arg->mesh_num*sph.host_arg->mesh_volume);
        host_mesh_count = (int *)malloc(sizeof(int)*sph.host_arg->mesh_num);

        hipMemcpy(host_mesh,cuda.mesh,sizeof(int)*sph.host_arg->mesh_num*sph.host_arg->mesh_volume,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(host_mesh_count,cuda.mesh_count,sizeof(int)*sph.host_arg->mesh_num,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        for(int j=0;j<sph.host_arg->mesh_num;j++)
        {
            //if(host_mesh_count[j]!=0) printf("error!!!!!!\n");
            printf("mesh id is:%d mesh num is:%d they are:",j,host_mesh_count[j]);
            for(int k=0;k<host_mesh_count[j];k++)
            {
                printf("%d,",host_mesh[j+k*sph.host_arg->mesh_num]);
            }
            printf("\n");
        }
    }

    /*
    for(int i=0;i<sph.host_arg->mesh_num;i++)
    {
        printf("mesh id is:%d mesh num is:%d\n",i,host_mesh_count[i]);
    }*/

    sph_free(&sph);
    hipDeviceReset();
    return 0;
}

