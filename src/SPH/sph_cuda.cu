#include "hip/hip_runtime.h"
#include "SPH.cuh"


__global__ void check_pair(SPH_ARG *arg)
{
    printf("the pair num is:%d\n",arg->pair_num);
}

int main(void)
{
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh = NULL;
    SPH_ARG arg;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.host_rigid = &wedge;
    sph.host_arg = &arg;
    sph.mesh = mesh;

    hipSetDevice(0);
    sph_init(&sph); 

    //define the seed for ptc data structure
    dim3 ptc_block(256);
    dim3 ptc_grid((int)(sph.particle->total/256)+1);
    //define the seed for mesh data structure
    dim3 mesh_block(32);
    dim3 mesh_grid(MESH_LENGTH_NUM,MESH_DEEPTH_NUM);
    //define the seed for pair data structre
    dim3 pair_block(512);
    dim3 pair_grid((int)(sph.particle->total/16)+1);

    sph_fuck_you<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
    hipDeviceSynchronize();
    sph_nnps_cuda<<<mesh_grid,mesh_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
    hipDeviceSynchronize();
    check_pair<<<1,1>>>(sph.dev_arg);
    hipDeviceSynchronize();

    sph_free(&sph);
    hipDeviceReset();
    return 0;
}

/*__global__ void sph_predict_cuda(double *x,double *y,double *temp_x,double *temp_y,double *vx,double *vy,double *temp_vx,double *temp_vy,double *accx,double *accy,double *rho,double *temp_rho,double *drho,double *p,int *type,int ptc_num)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= ptc_num )return;

    if(type[id] == 0)
    {
        temp_x[id] = x[id];
        temp_y[id] = y[id];
        temp_vx[id] = vx[id];
        temp_vy[id] = vy[id];
        temp_rho[id] = rho[id];

        x[id] += vx[id]*dev_dt*0.5;
        y[id] += vy[id]*dev_dt*0.5;
        vx[id] += accx[id]*dev_dt*0.5;
        vy[id] += accy[id]*dev_dt*0.5;
        rho[id] += drho[id]*dev_dt*0.5;
        if(rho[id] < REF_DENSITY) rho[id]=REF_DENSITY;
    }
 
    else
    {
        vx[id] = 0.0;
        vy[id] = 0.0;
        p[id] = 0.0;
    }

}


__global__ void sph_correct_cuda(double *x,double *y,double *temp_x,double *temp_y,double *vx,double *vy,double *temp_vx,double *temp_vy,double *accx,double *accy,double *rho,double *temp_rho,double *drho,double *p,int *type,int ptc_num)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= ptc_num )return; 

    if(type[id] == 0)
    {
        x[id] = temp_x[id] + vx[id]*dev_dt;
        y[id] = temp_y[id] + vy[id]*dev_dt;
        vx[id] = temp_vx[id] + accx[id]*dev_dt;
        vy[id] = temp_vy[id] + accy[id]*dev_dt;
        rho[id] = temp_rho[id] + drho[id]*dev_dt;
        if(rho[id] < REF_DENSITY) rho[id]=REF_DENSITY;
    }
    else
    {
        vx[id] = 0.0;
        vy[id] = 0.0;
        p[id] = 0.0;
    }
}*/
/*
        CUDA_CHECK(hipMemcpy(sph.mesh,dev_mesh,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int),hipMemcpyDeviceToHost));
    string filename = "../data/postprocess/vtk/sph"; 
    filename += to_string(sph.current_step/PRINT_TIME_STEP);
    filename += ".vtk";

    ofstream vtkfile;
    vtkfile.open(filename.c_str());

    vtkfile << "# vtk DataFile Version 3.0" << endl;
    vtkfile << "sph data" << endl;
    vtkfile << "ASCII" << endl;
    vtkfile << "DATASET UNSTRUCTURED_GRID" << endl;
    vtkfile << "POINTS " << sph.particle->total << " " << "double" << endl;

    for(unsigned int i=0;i<MESH_DEEPTH_NUM;i++)
    {
        for(unsigned int j=0;j<MESH_LENGTH_NUM;j++)
        {
            temp = sph.mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH_NUM*(MESH_PTC_NUM-1)];
            for(unsigned int k=0;k<temp;k++)
            {
                temp_1 = sph.mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH_NUM*k];
                vtkfile << setiosflags(ios::scientific) << sph.particle->x[temp_1] << " " \
                << sph.particle->y[temp_1] << " " << 0.0 << endl;
            }
        }
    }
    vtkfile.close();*/