#include "hip/hip_runtime.h"
#include "SPH.cuh"

__global__ void check_ptc(SPH_CUDA *cuda,SPH_ARG *arg)
{
    const int id = threadIdx.x + threadIdx.y*blockDim.x;
    if(id >= arg->ptc_num)return;
    printf("%lf %lf\n",cuda->x[id],cuda->y[id]);
}

__global__ void check_pair(SPH_ARG *arg)
{
    printf("the pair num is:%d\n",arg->pair_num);
}

__global__ void check_mesh(SPH_CUDA *cuda,SPH_ARG *arg)
{
    const int mesh_id = blockIdx.x + blockIdx.y* gridDim.x;
    if(cuda->mesh_count[mesh_id]!=0)
    {
        printf("%d %d\n",mesh_id,cuda->mesh_count[mesh_id]);
    }
    /*
    if(cuda->mesh_count[mesh_id] != 0)
    {
        printf("mesh id is:%d ptc in mesh is:%d they are:",mesh_id,cuda->mesh_count[mesh_id]);
        for(int i=0;i<cuda->mesh_count[mesh_id];i++)
        {
            printf("%d",cuda->mesh[mesh_id+i*arg->mesh_num]);
        }
        printf("\n");
    }*/
}

int main(void)
{
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh = NULL;
    SPH_ARG arg;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.host_rigid = &wedge;
    sph.host_arg = &arg;
    sph.mesh = mesh;

    hipSetDevice(0);
    sph_init(&sph); 

    //define the seed for ptc data structure
    dim3 ptc_block(256);
    dim3 ptc_grid((int)(sph.particle->total/256)+1);
    //define the seed for mesh data structure
    dim3 mesh_block(32,32);
    dim3 mesh_grid(MESH_LENGTH_NUM,MESH_DEEPTH_NUM);
    //define the seed for pair data structre
    dim3 pair_block(512);
    dim3 pair_grid((int)(sph.particle->total/16)+1);

    //check_ptc<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
    //hipDeviceSynchronize();
    sph_fuck_you<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
    hipDeviceSynchronize();
    check_mesh<<<mesh_grid,1>>>(sph.cuda,sph.dev_arg);
    hipDeviceSynchronize();
    //sph_nnps_cuda<<<mesh_grid,mesh_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
    //hipDeviceSynchronize();
    //check_pair<<<1,1>>>(sph.dev_arg);
    //hipDeviceSynchronize();

    sph_free(&sph);
    hipDeviceReset();
    return 0;
}

/*__global__ void sph_predict_cuda(double *x,double *y,double *temp_x,double *temp_y,double *vx,double *vy,double *temp_vx,double *temp_vy,double *accx,double *accy,double *rho,double *temp_rho,double *drho,double *p,int *type,int ptc_num)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= ptc_num )return;

    if(type[id] == 0)
    {
        temp_x[id] = x[id];
        temp_y[id] = y[id];
        temp_vx[id] = vx[id];
        temp_vy[id] = vy[id];
        temp_rho[id] = rho[id];

        x[id] += vx[id]*dev_dt*0.5;
        y[id] += vy[id]*dev_dt*0.5;
        vx[id] += accx[id]*dev_dt*0.5;
        vy[id] += accy[id]*dev_dt*0.5;
        rho[id] += drho[id]*dev_dt*0.5;
        if(rho[id] < REF_DENSITY) rho[id]=REF_DENSITY;
    }
 
    else
    {
        vx[id] = 0.0;
        vy[id] = 0.0;
        p[id] = 0.0;
    }

}


__global__ void sph_correct_cuda(double *x,double *y,double *temp_x,double *temp_y,double *vx,double *vy,double *temp_vx,double *temp_vy,double *accx,double *accy,double *rho,double *temp_rho,double *drho,double *p,int *type,int ptc_num)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= ptc_num )return; 

    if(type[id] == 0)
    {
        x[id] = temp_x[id] + vx[id]*dev_dt;
        y[id] = temp_y[id] + vy[id]*dev_dt;
        vx[id] = temp_vx[id] + accx[id]*dev_dt;
        vy[id] = temp_vy[id] + accy[id]*dev_dt;
        rho[id] = temp_rho[id] + drho[id]*dev_dt;
        if(rho[id] < REF_DENSITY) rho[id]=REF_DENSITY;
    }
    else
    {
        vx[id] = 0.0;
        vy[id] = 0.0;
        p[id] = 0.0;
    }
}*/
/*
        CUDA_CHECK(hipMemcpy(sph.mesh,dev_mesh,MESH_DEEPTH_NUM*MESH_LENGTH_NUM*MESH_PTC_NUM*sizeof(int),hipMemcpyDeviceToHost));
    string filename = "../data/postprocess/vtk/sph"; 
    filename += to_string(sph.current_step/PRINT_TIME_STEP);
    filename += ".vtk";

    ofstream vtkfile;
    vtkfile.open(filename.c_str());

    vtkfile << "# vtk DataFile Version 3.0" << endl;
    vtkfile << "sph data" << endl;
    vtkfile << "ASCII" << endl;
    vtkfile << "DATASET UNSTRUCTURED_GRID" << endl;
    vtkfile << "POINTS " << sph.particle->total << " " << "double" << endl;

    for(unsigned int i=0;i<MESH_DEEPTH_NUM;i++)
    {
        for(unsigned int j=0;j<MESH_LENGTH_NUM;j++)
        {
            temp = sph.mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH_NUM*(MESH_PTC_NUM-1)];
            for(unsigned int k=0;k<temp;k++)
            {
                temp_1 = sph.mesh[i*MESH_LENGTH_NUM+j+MESH_LENGTH_NUM*MESH_DEEPTH_NUM*k];
                vtkfile << setiosflags(ios::scientific) << sph.particle->x[temp_1] << " " \
                << sph.particle->y[temp_1] << " " << 0.0 << endl;
            }
        }
    }
    vtkfile.close();*/