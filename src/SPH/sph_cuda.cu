#include "hip/hip_runtime.h"
#include "SPH.cuh"

__global__ void check_ptc(SPH_CUDA *cuda,SPH_ARG *arg)
{
    const int id = threadIdx.x + threadIdx.y*blockDim.x;
    if(id >= arg->ptc_num)return;
    printf("%lf %lf\n",cuda->x[id],cuda->y[id]);
}

__global__ void check_pair(SPH_ARG *arg)
{
    printf("the pair num is:%d\n",arg->pair_num);
}

__global__ void check_mesh(SPH_CUDA *cuda,SPH_ARG *arg)
{
    const int mesh_id = blockIdx.x + blockIdx.y* gridDim.x;
    if(cuda->mesh_count[mesh_id]!=0)
    {
        printf("%d %d\n",mesh_id,cuda->mesh_count[mesh_id]);
    }
    /*
    if(cuda->mesh_count[mesh_id] != 0)
    {
        printf("mesh id is:%d ptc in mesh is:%d they are:",mesh_id,cuda->mesh_count[mesh_id]);
        for(int i=0;i<cuda->mesh_count[mesh_id];i++)
        {
            printf("%d",cuda->mesh[mesh_id+i*arg->mesh_num]);
        }
        printf("\n");
    }*/
}

int main(void)
{
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh = NULL;
    SPH_ARG arg;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.host_rigid = &wedge;
    sph.host_arg = &arg;
    sph.mesh = mesh;

    hipSetDevice(0);
    sph_init(&sph); 

    //define the seed for ptc data structure
    dim3 ptc_block(256);
    dim3 ptc_grid((int)(sph.particle->total/256)+1);
    //define the seed for mesh data structure
    dim3 mesh_block(32,32);
    dim3 mesh_grid(MESH_LENGTH_NUM,MESH_DEEPTH_NUM);
    //define the seed for pair data structre
    dim3 pair_block(512);
    dim3 pair_grid((int)(sph.particle->total/16)+1);

    //check_ptc<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
    //hipDeviceSynchronize();
    sph_mesh_cuda<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
    hipDeviceSynchronize();
    //check_mesh<<<mesh_grid,1>>>(sph.cuda,sph.dev_arg);
    //hipDeviceSynchronize();
    //sph_nnps_cuda<<<mesh_grid,mesh_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
    //hipDeviceSynchronize();
    //check_pair<<<1,1>>>(sph.dev_arg);
    //hipDeviceSynchronize();

    int *host_mesh;
    int *host_mesh_count;
    SPH_CUDA cuda;
    hipMemcpy(&cuda,sph.cuda,sizeof(SPH_CUDA),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    host_mesh = (int *)malloc(sizeof(int)*sph.host_arg->mesh_num*sph.host_arg->mesh_volume);
    host_mesh_count = (int *)mall0c(sizeof(int)*sph.host_arg->mesh_num);

    hipMemcpy(host_mesh,cuda.mesh,sizeof(int)*sph.host_arg->mesh_num*sph.host_arg->mesh_volume);
    hipDeviceSynchronize();
    hipMemcpy(host_mesh_count,cuda.mesh_count,sizeof(int)*sph.host->mesh_num);
    hipDeviceSynchronize();

    for(int i=0;i<sph.host_arg->mesh_num;i++)
    {
        printf("mesh id is:%d mesh num is:%d\n",i,host_mesh_count[i]);
    }

    sph_free(&sph);
    hipDeviceReset();
    return 0;
}

