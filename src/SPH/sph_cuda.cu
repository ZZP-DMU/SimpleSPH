#include "hip/hip_runtime.h"
#include "SPH.cuh"

__global__ void check_ptc(SPH_CUDA *cuda,SPH_ARG *arg)
{
    const int id = threadIdx.x + threadIdx.y*blockDim.x;
    if(id >= arg->ptc_num)return;
    printf("%lf %lf\n",cuda->x[id],cuda->y[id]);
}
__global__ void check_nnps(SPH_CUDA *cuda,SPH_ARG *arg,int *pair_i,int *pair_j,int p_num)
{
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    int id = 0;
    if( threadIdx.x >= cuda->pair_count[mesh_id]) return;
    else id = mesh_id*arg->pair_volume + threadIdx.x;
    int j=0;
    for(int i=0;i<p_num;i++)
    {   
        if(cuda->pair_i[id] == pair_i[i] && cuda->pair_j[id] == pair_j[i])
        {
            j++;
        }
        else if(cuda->pair_i[id] == pair_j[i] && cuda->pair_j[id] == pair_i[i])
        {
            j++;
        }
        else if(cuda->pair_i[id] == cuda->pair_j[id])
        {
            j++;
        }
    }
    if(j!=1)
        {
            printf("error the same pair num is:%d\n",j);
        }
}

__global__ void check_pair(SPH_CUDA *cuda,SPH_ARG *arg)
{
    //double dx=0.0;
    //double dy=0.0;
    //double q=0.0;
    //const int id = threadIdx.x + blockIdx.x * blockDim.x;
    //if(id >= arg->pair_num) return;
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    int id = 0;
    int tmp_id = 0;
    if(threadIdx.x >= cuda->pair_count[mesh_id]) return;
    else id = mesh_id*arg->pair_volume + threadIdx.x;
    for(int i=0;i<arg->mesh_num;i++)
    {
        for(int j=0;j<cuda->pair_count[i];j++)
        {
            tmp_id = i*arg->pair_volume + j;
            if(id != tmp_id)
            {
                if(cuda->pair_i[id] == cuda->pair_j[tmp_id] && cuda->pair_j[id] == cuda->pair_j[tmp_id])
                {
                    printf("type 1 errors\n");
                }
                else if(cuda->pair_i[id] == cuda->pair_j[tmp_id] && cuda->pair_j[id] == cuda->pair_i[tmp_id])
                {
                    printf("type 2 errors\n");
                }
            }
        }
    }

    /*
    dx = cuda->x[cuda->pair_i[id]] - cuda->x[cuda->pair_j[id]];
    dy = cuda->y[cuda->pair_i[id]] - cuda->y[cuda->pair_j[id]];
    q = sqrt(dx*dx+dy*dy)/arg->h;
    if(q > 2.0) printf("error !!!\n");*/

    
    //if(id == 0)printf("the pair num is:%d\n",arg->pair_num);
    /*for(int i=0;i<arg->pair_num;i++)
    {
        if(cuda->pair_i[id] == cuda->pair_i[i] && cuda->pair_j[id]==cuda->pair_j[i] && id!=i)
        {
            if(cuda->pair_i[id]!=0 && cuda->pair_j[id]!=0)
            {
                //printf("type1 index_1:%d index_2:%d pair_i:%d pair_j:%d\n",id,i,cuda->pair_i[id],cuda->pair_j[id]);
            }
            atomicAdd(&(arg->tmp),1);
        }
        else if(cuda->pair_i[id] == cuda->pair_j[i] && cuda->pair_j[id]==cuda->pair_i[i])
        {
            if(id == i)
            {
                //printf("type2 nnps error !!\n");
                if(cuda->pair_i[id]!=0 && cuda->pair_j[id]!=0)
                {
                    //printf("type2 index_1:%d index_2:%d pair_i:%d pair_j:%d\n",id,i,cuda->pair_i[id],cuda->pair_j[id]);
                }
            }
            else 
            {
                //printf("here is same pair\n");
                if(cuda->pair_i[id]!=0 && cuda->pair_j[id]!=0)
                {
                    //printf("type3 index_1:%d index_2:%d pair_i:%d pair_j:%d\n",id,i,cuda->pair_i[id],cuda->pair_j[id]);
                }
            }
            atomicAdd(&(arg->tmp),1);
        }
    }*/
}

__global__ void check_mesh(SPH_CUDA *cuda,SPH_ARG *arg)
{
    int mid = 0;
    int id = 0;
    const int mesh_id = blockIdx.x + blockIdx.y* gridDim.x;
    atomicAdd(&arg->tmp,cuda->pair_count[mesh_id]);
    cuda->pair_count[mesh_id]=0;
    /*
    for(int i=0;i<cuda->mesh_count[mesh_id];i++)
    {
        id = cuda->mesh[i*arg->mesh_num + mesh_id];
        if(cuda->y[id] < arg->domain_y && cuda->y[id] >= 0.0)
        {
            mid = __double2int_rz(cuda->y[id]/arg->mesh_dx)*arg->mesh_xnum;
        }
        else
        {
            mid = (arg->mesh_ynum - 1)*arg->mesh_xnum;
        }
        if(cuda->x[id] < arg->domain_x && cuda->x[id] >= 0.0)
        {
            mid += __double2int_rz(cuda->x[id]/arg->mesh_dx);
            if(mid == mesh_id) atomicAdd(&(arg->tmp),1);
            //if(mid != mesh_id) printf("mid:%d mesh_id:%d id:%d x:%lf y:%lf\n",mid,mesh_id,id,cuda->x[id],cuda->y[id]);

        }
        else
        {
            mid += arg->mesh_xnum - 1;
            if(mid == mesh_id) atomicAdd(&(arg->tmp),1);
            //if(mid != mesh_id) printf("mid:%d mesh_id:%d id:%d x:%lf y:%lf\n",mid,mesh_id,id,cuda->x[id],cuda->y[id]);
        }
    }
    cuda->mesh_count[mesh_id] = 0;*/
    /*
    if(cuda->mesh_count[mesh_id]!=0)
    {
        printf("%d %d\n",mesh_id,cuda->mesh_count[mesh_id]);
    }*/
    /*
    if(cuda->mesh_count[mesh_id] != 0)
    {
        printf("mesh id is:%d ptc in mesh is:%d they are:",mesh_id,cuda->mesh_count[mesh_id]);
        for(int i=0;i<cuda->mesh_count[mesh_id];i++)
        {
            printf("%d",cuda->mesh[mesh_id+i*arg->mesh_num]);
        }
        printf("\n");
    }*/
}

int main(void)
{
    SPH_PARTICLE particle;
    SPH_KERNEL kernel;
    SPH_PAIR pair;
    SPH_RIGID wedge;
    SPH_MESH mesh;
    SPH_ARG arg;
    SPH sph;
    sph.particle = &particle;
    sph.kernel = &kernel;
    sph.pair = &pair;
    sph.host_rigid = &wedge;
    sph.host_arg = &arg;
    sph.mesh = &mesh;

    hipSetDevice(0);
    sph_init(&sph); 

    //define the seed for ptc data structure
    dim3 ptc_block(256);
    dim3 ptc_grid((int)(sph.particle->total/256)+1);
    //define the seed for mesh data structure
    dim3 mesh_block(32,32);
    dim3 mesh_grid(MESH_LENGTH_NUM,MESH_DEEPTH_NUM);
    //define the seed for pair data structre
    dim3 pair_block(128);
    dim3 pair_grid(MESH_LENGTH_NUM,MESH_DEEPTH_NUM);


    SPH_CUDA cuda;
    SPH_ARG tmp_arg;
    hipMemcpy(&cuda,sph.cuda,sizeof(SPH_CUDA),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    //int *host_pair_count = (int *)calloc(sph.host_arg->mesh_num,sizeof(int));
    //int id = 0;
    int *cpu_pair_i;
    int *cpu_pair_j;
    hipMalloc(&cpu_pair_i,sizeof(int)*32*sph.particle->total);
    hipMalloc(&cpu_pair_j,sizeof(int)*32*sph.particle->total);

    for(int i=0;i<100;i++)
    {
        printf("current step is:%d\n",i);
        //check_ptc<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
        //hipDeviceSynchronize();
        sph_mesh_cuda<<<ptc_grid,ptc_block>>>(sph.cuda,sph.dev_arg);
        hipDeviceSynchronize();
        //check_mesh<<<mesh_grid,1>>>(sph.cuda,sph.dev_arg);
        //hipDeviceSynchronize();
        
        hipMemcpy(sph.mesh->ptc,cuda.mesh,sizeof(int)*sph.host_arg->mesh_num*sph.host_arg->mesh_volume,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(sph.mesh->count,cuda.mesh_count,sizeof(int)*sph.host_arg->mesh_num,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        sph_nnps_cpu(&sph);

        sph_nnps_cuda<<<mesh_grid,mesh_block>>>(sph.cuda,sph.dev_arg,sph.dev_rigid);
        hipDeviceSynchronize();

        
        /*
        for(int i=0;i<sph.host_arg->mesh_num;i++)
        {
            printf("%d\n",sph.mesh->count[i]);
        }*/
        
        hipMemcpy(cpu_pair_i,sph.pair->i,sizeof(int)*32*sph.particle->total,hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        hipMemcpy(cpu_pair_j,sph.pair->j,sizeof(int)*32*sph.particle->total,hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        check_nnps<<<pair_grid,pair_block>>>(sph.cuda,sph.dev_arg,cpu_pair_i,cpu_pair_j,sph.host_arg->pair_num);
        hipDeviceSynchronize();
        check_pair<<<pair_grid,pair_block>>>(sph.cuda,sph.dev_arg);
        hipDeviceSynchronize();
        check_mesh<<<mesh_grid,1>>>(sph.cuda,sph.dev_arg);
        hipDeviceSynchronize();


        hipError_t sph_error = hipGetLastError();
        printf("%s\n",hipGetErrorName(sph_error));
    /*
        hipMemcpy(sph.pair->i,cuda.pair_i,sizeof(int)*32*sph.particle->total,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(sph.pair->j,cuda.pair_j,sizeof(int)*32*sph.particle->total,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(&tmp_arg,sph.dev_arg,sizeof(SPH_ARG),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(host_pair_count,cuda.pair_count,sph.host_arg->mesh_num*sizeof(int),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        for(int i=0;i<sph.host_arg->mesh_num;i++)
        {
            for(int j=0;j<host_pair_count[i];j++)
            {
                id = i*sph.host_arg->pair_volume+j;
                printf("id:%d i:%d j:%d\n",id,sph.pair->i[id],sph.pair->j[id]); 
            }
        }
        */
        
        //printf("the total same pair num is:%d \n",tmp_arg.tmp);
        
        /*
        host_mesh = (int *)malloc(sizeof(int)*sph.host_arg->mesh_num*sph.host_arg->mesh_volume);
        host_mesh_count = (int *)malloc(sizeof(int)*sph.host_arg->mesh_num);

        hipMemcpy(host_mesh,cuda.mesh,sizeof(int)*sph.host_arg->mesh_num*sph.host_arg->mesh_volume,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipMemcpy(host_mesh_count,cuda.mesh_count,sizeof(int)*sph.host_arg->mesh_num,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        for(int j=0;j<sph.host_arg->mesh_num;j++)
        {
            //if(host_mesh_count[j]!=0) printf("error!!!!!!\n");
            printf("mesh id is:%d mesh num is:%d they are:",j,host_mesh_count[j]);
            for(int k=0;k<host_mesh_count[j];k++)
            {
                printf("%d,",host_mesh[j+k*sph.host_arg->mesh_num]);
            }
            printf("\n");
        }*/
        
    }

    /*
    for(int i=0;i<sph.host_arg->mesh_num;i++)
    {
        printf("mesh id is:%d mesh num is:%d\n",i,host_mesh_count[i]);
    }*/

    sph_free(&sph);
    hipDeviceReset();
    return 0;
}

