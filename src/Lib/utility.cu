#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_dummy_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    double dx = 0.0;
    double dy = 0.0;
    double tmp_accx = 0.0;
    double tmp_accy = 0.0;
    double tmp_prho = 0.0;
    double tmp_vx = 0.0;
    double tmp_vy = 0.0;
    int index_i = 0;
    int index_j = 0;
    int id = 0;
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    if(threadIdx < cuda->pair_count[mesh_id])
    {
        id = mesh_id * arg->pair_volume + threadIdx.x;
        index_i = cuda->pair_i[id];
        index_j = cuda->pair_j[id];

        if(cuda->type[index_j] != 0 cuda->ptc_w[index_j] != 0.0)
        {
            if(cuda->type[index_j] == -1)
            {
                tmp_accx = 0.0;
                tmp_accy = 0.0;
            }
            else if(cuda->type[index_j] == 1)
            {
                tmp_accx = rigid->accx - pow(rigid->omega,2)*(cuda->x[index_j]-rigid->cogx)- \
                              rigid->alpha*(cuda->y[index_j]-rigid->cogy);
                tmp_accy = rigid->accy - pow(rigid->omega,2)*(cuda->y[index_j]-rigid->cogy)+ \
                              rigid->alpha*(cuda->x[index_j]-rigid->cogx);
            }
            dx = cuda->x[index_i]-cuda->y[index_j];
            dy = cuda->y[index_i]-cuda->y[index_j];

            tmp_prho = (cuda->p[index_i]+cuda->rho[index_i]*(tmp_accx*dx+(tmp_accy+arg->g)*dy))*cuda->pair_w[id]/cuda->ptc_w[index_j];
            atomicAdd(&(cuda->p[index_j]),tmp_prho);
            
            tmp_prho /= arg->c*arg->c;
            atomicadd(&(cuda->rho[index_j]),tmp_prho);

            tmp_vx = cuda->vx[index_i]*cuda->pair_w[id]/cuda->ptc_w[index_j];
            atomicAdd(&(cuda->vx[index_j]),tmp_vx);

            tmp_vy = cuda->vy[index_i]*cuda->pair_w[id]/cuda->ptc_w[index_j];
            atomicAdd(&(cuda->vy[index_j]),tmp_vy);
        }   
    }
    __syncthreads();
    if( threadIdx.x == 0)cuda->pair_count[mesh_id] = 0;
}
