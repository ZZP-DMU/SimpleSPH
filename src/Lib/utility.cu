#include "hip/hip_runtime.h"
#include "Lib.cuh"

/*__global__ void sph_dummy_cuda(double *x,double *y,double *vx,double *vy,double *p,double *rho,double *ptc_w,double *pair_w,int *pair_i,int *pair_j,int *type,double *rigid,int *pair_num)
{
    double rigid_accx = 0.0;
    double rigid_accy = 0.0;
    double dx = 0.0;
    double dy = 0.0;
    double temp_p = 0.0;
    double temp_vx = 0.0;
    double temp_vy = 0.0;

    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= pair_num[0]) return;

    if(type[pair_j[id]] != 0 && ptc_w[pair_j[id]] != 0.0)
    {
        rigid_accx = 0.0;
        rigid_accy = 0.0;
        dx = x[pair_i[id]]-x[pair_j[id]];
        dy = y[pair_i[id]]-y[pair_j[id]];
        if(type[pair_j[id]] == -1)
        {
            rigid_accx = 0.0;
            rigid_accy = 0.0;
        }
        else if (type[pair_j[id]] == 1)
        {
            rigid_accx = rigid[ACCX] - pow(rigid[OMEGA],2)*(x[pair_j[id]]-rigid[COGX])- \
                              rigid[R_ALPHA]*(y[pair_j[id]]-rigid[COGY]);
            rigid_accy = rigid[ACCY] - pow(rigid[OMEGA],2)*(x[pair_j[id]]-rigid[COGY])+ \
                              rigid[R_ALPHA]*(x[pair_j[id]]-rigid[COGX]);
        }
        temp_p = (p[pair_i[id]]+rho[pair_i[id]]*(rigid_accx*dx+(rigid_accy+GRAVITY_ACC)*dy))*pair_w[id]/ptc_w[pair_j[id]];
        temp_vx = vx[pair_i[id]]*pair_w[id]/ptc_w[pair_j[id]];
        temp_vy = vy[pair_i[id]]*pair_w[id]/ptc_w[pair_j[id]];

        atomicAdd(&p[pair_j[id]],temp_p);
        atomicAdd(&vx[pair_j[id]],temp_vx);
        atomicAdd(&vy[pair_j[id]],temp_vy);
    }
}*/

