#include "hip/hip_runtime.h"
#include "Lib.cuh"

__device__ void sph_lock_cuda(SPH_ARG *arg)
{
    while (!atomicCAS(&(arg->lock),1,0))
    {
        continue;
    }
    
}

__device__ void sph_unlock_cuda(SPH_ARG *arg)
{
    atomicCAS(&(arg->lock),0,1);
}