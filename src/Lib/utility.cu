#include "hip/hip_runtime.h"
#include "Lib.cuh"

void ptc_density_correct(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    particle = sph->particle;
    pair= sph->pair;
    kernel = sph->kernel;

    double a = ALPHA;
    //double m = PTC_MASS;


    for(unsigned int i=0;i<particle->total;i++)
    {
        if(particle->type[i] == 0)
        {
            particle->w[i] = (a*2.0*particle->mass[i])/(3.0*particle->density[i]);
        }
    }

    for(unsigned int i=0;i<pair->total;i++)
    {
        particle->w[pair->i[i]] += kernel->w[i]*particle->mass[pair->j[i]]/particle->density[pair->j[i]];
        if(particle->type[pair->j[i]]==0)
        {
            particle->w[pair->j[i]] += kernel->w[i]*particle->mass[pair->i[i]]/particle->density[pair->i[i]];
        }
    }

    for(unsigned int i=0;i<particle->total;i++)
    {
        if(particle->type[i] == 0)
        {
            particle->density[i] = (a*2.0*particle->mass[i])/(3.0*particle->w[i]);
        }
    }

    for(unsigned int i=0;i<pair->total;i++)
    {
        particle->density[pair->i[i]] += particle->mass[pair->j[i]]*kernel->w[i]/particle->w[pair->i[i]];
        if(particle->type[pair->j[i]] == 0)
        {
            particle->density[pair->j[i]] += particle->mass[pair->i[i]]*kernel->w[i]/particle->w[pair->j[i]];
        }
    }
}

void ptc_dummy(SPH *sph)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_KERNEL *kernel;
    SPH_RIGID *wedge;
    particle = sph->particle;
    pair = sph->pair;
    kernel = sph->kernel;
    wedge = sph->rigid;


    //rigid body(wall & wedge)vx,vy,accx,accy,pressure init
    for(unsigned int i=0;i<particle->total;i++)
    {
        if(particle->type[i] != 0)
        {
            particle->w[i] = 0;
            particle->vx[i] = 0;
            particle->vy[i] = 0;
            particle->pressure[i] = 0;
            particle->density[i] = 0;
        }
    }
    
    //the not fluid weight term 
    for(unsigned int i=0;i<pair->total;i++)
    {
        if(particle->type[pair->j[i]] != 0) 
        {
            particle->w[pair->j[i]] += kernel->w[i];
        }
    }
    
    //rigid body(wall & wedege) pressure and velocity
    for(unsigned int i=0;i<pair->total;i++)
    {
        double dx = 0.0;
        double dy = 0.0;
        double rigid_acc_x = 0.0;
        double rigid_acc_y = 0.0;
        if(particle->type[pair->j[i]] != 0 && particle->w[pair->j[i]] != 0.0)
        {
            if(particle->type[pair->j[i]] == -1)
            {
                rigid_acc_x = 0.0;
                rigid_acc_y = 0.0;
            }
            else if (particle->type[pair->j[i]] == 1)
            {
                rigid_acc_x = wedge->accx - pow(wedge->omega,2)*(particle->x[pair->j[i]]-wedge->cogx)- \
                              wedge->alpha*(particle->y[pair->j[i]]-wedge->cogy);
                rigid_acc_y = wedge->accy - pow(wedge->omega,2)*(particle->y[pair->j[i]]-wedge->cogy)+ \
                              wedge->alpha*(particle->x[pair->j[i]]-wedge->cogx);
            }
            dx = particle->x[pair->i[i]] - particle->x[pair->j[i]];
            dy = particle->y[pair->i[i]] - particle->y[pair->j[i]];
            particle->pressure[pair->j[i]] += (particle->pressure[pair->i[i]]+particle->density[pair->i[i]]*\
                        (rigid_acc_x*dx+(rigid_acc_y+GRAVITY_ACC)*dy))*kernel->w[i]/particle->w[pair->j[i]];
            particle->vx[pair->j[i]] += particle->vx[pair->i[i]]*kernel->w[i]/particle->w[pair->j[i]];
            particle->vy[pair->j[i]] += particle->vy[pair->i[i]]*kernel->w[i]/particle->w[pair->j[i]];
        }
    }

    //rigid body(wall & wedege) densiy
    for(unsigned int i=0;i<particle->total;i++)
    {
        if(particle->type[i] != 0)
        {
            particle->density[i] = particle->pressure[i]/pow(sph->c,2)+REF_DENSITY;
        }
    }
}

void sph_avg_time(SPH *sph)
{
    static unsigned int step = 0;
    static double start;
    static double end;
    if(step == 0)
    {
        start = (double)time(nullptr);
    }
    else
    {
        end = (double)time(nullptr);
        sph->avg_time = (end-start)/(double)step;
    }
    step++;
}
