#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_sponge_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    //do not impliment
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    double r = 0.0;
    double tmp_0 = 0.0;
    double tmp_1 = 0.0;
    double tmp_2 = 0.0;
    if(id < arg->ptc_num)
    {
        if(cuda->type[id] == 0)
        {
            r = sqrt(pow((cuda->x[id]-arg->fluid_x/2.0),2)+pow((cuda->y[id]-arg->fluid_y),2)) - arg->fluid_x/2.0;
            if(r > 0.0)
            {
                tmp_0 = 50.0*r/arg->sponge_dx;
                tmp_1 = pow(0.3,tmp_0);
                tmp_2 = 1.0 - pow(100,-tmp_1);

                cuda->accx[id] *= tmp_2;
                cuda->accy[id] *= tmp_2;
                cuda->drho[id] *= tmp_2;
            }
        }
    }
}