#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_nnps_cuda(int *mesh,double *x,double *y,int *type,int *pair_i,int *pair_j)
{
    /*
    blockIdx.x ---> length direction
    blockIdx.y ---> deepth direction
    threadIdx.x ---> search the mesh
    threadIdx.y ---> search the near mesh
    */
    double q;
    int i,j;
    int count_temp;
    int mesh_ptc_num;
    int mesh_near_ptc_num;
    if( blockIdx.x >= MESH_LENGTH_NUM_CUDA || blockIdx.y >= MESH_DEEPTH_NUM_CUDA) return;
    mesh_ptc_num = mesh[ blockIdx.x + blockIdx.y*MESH_LENGTH_NUM_CUDA + MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*(MESH_PTC_NUM-1)];
    if( threadIdx.x >= mesh_ptc_num)return;
    i = blockIdx.x + blockIdx.y*MESH_LENGTH_NUM_CUDA + threadIdx.x*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
    

    //mesh[i,j]->mesh[i,j]
    mesh_near_ptc_num = mesh_ptc_num;
    if( threadIdx.y > threadIdx.x && threadIdx.y < mesh_near_ptc_num)
    {
        j = blockIdx.x + blockIdx.y*MESH_LENGTH_NUM_CUDA + threadIdx.y*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
        q = (x[mesh[i]]-x[mesh[j]])**(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])**(y[mesh[i]]-y[mesh[j]]);
        q = sqrt(q);
        if(q<2.0)
        {
            if(type[mesh[i]] == 0)
            {
                count_temp = atomicAdd(&count,1);
                pair_i[count_temp] = i;
                pair_j[count_temp] = j;
            }
            else if(type[mesh[j]] == 0)
            {
                count_temp = atomicAdd(&count,1);
                pair_i[count_temp] = j;
                pair_j[count_temp] = i;
            }
        }
    }

    //mesh[i,j]->mesh[i,j+1]
    if( blockIdx.x < (MESH_LENGTH_NUM_CUDA-1))
    {
        mesh_near_ptc_num = mesh[ (blockIdx.x+1) + blockIdx.y*MESH_LENGTH_NUM_CUDA + MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*(MESH_PTC_NUM-1)];
        if( threadIdx.y < mesh_near_ptc_num )
        {
            j = (blockIdx.x+1) + blockIdx.y*MESH_LENGTH_NUM_CUDA + threadIdx.y*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
            q = (x[mesh[i]]-x[mesh[j]])**(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])**(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q);
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&count,1);
                    pair_i[count_temp] = i;
                    pair_j[count_temp] = j;
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&count,1);
                    pair_i[count_temp] = j;
                    pair_j[count_temp] = i;
                }
            }
        }
    }

    //mesh[i,j]->mesh[i+1,j]
    if( blockIdx.y < (MESH_DEEPTH_NUM_CUDA-1))
    {
        mesh_near_ptc_num = mesh[ blockIdx.x + ( blockIdx.y+1)*MESH_LENGTH_NUM_CUDA + MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*(MESH_PTC_NUM-1)];
        if( threadIdx.y < mesh_near_ptc_num )
        {
            j = blockIdx.x +( blockIdx.y+1)*MESH_LENGTH_NUM_CUDA + threadIdx.y*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
            q = (x[mesh[i]]-x[mesh[j]])**(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])**(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q);
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&count,1);
                    pair_i[count_temp] = i;
                    pair_j[count_temp] = j;
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&count,1);
                    pair_i[count_temp] = j;
                    pair_j[count_temp] = i;
                }
            }
        }
    }

    //mesh[i,j]->mesh[i+1,j+1]
    if( blockIdx.x < (MESH_LENGTH_NUM_CUDA-1) && blockIdx.y < (MESH_DEEPTH_NUM_CUDA-1))
    {
        mesh_near_ptc_num = mesh[( blockIdx.x+1) + ( blockIdx.y+1)*MESH_LENGTH_NUM_CUDA + MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*(MESH_PTC_NUM-1)];
        if( threadIdx.y < mesh_near_ptc_num)
        {
            j = ( blockIdx.x+1) +( blockIdx.y+1)*MESH_LENGTH_NUM_CUDA + threadIdx.y*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
            q = (x[mesh[i]]-x[mesh[j]])**(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])**(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q);
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&count,1);
                    pair_i[count_temp] = i;
                    pair_j[count_temp] = j;
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&count,1);
                    pair_i[count_temp] = j;
                    pair_j[count_temp] = i;
                }
            }
        }
    }

    //mesh[i,j]->mesh[i-1,j+1]
    if( blockIdx.x < (MESH_LENGTH_NUM_CUDA-1) && blockIdx.y > 0)
    {
        mesh_near_ptc_num = mesh[( blockIdx.x+1) + ( blockIdx.y-1)*MESH_LENGTH_NUM_CUDA + MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*(MESH_PTC_NUM-1)];
        if( threadIdx.y < mesh_near_ptc_num)
        {
            j = ( blockIdx.x+1) +( blockIdx.y-1)*MESH_LENGTH_NUM_CUDA + threadIdx.y*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
            q = (x[mesh[i]]-x[mesh[j]])**(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])**(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q);
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&count,1);
                    pair_i[count_temp] = i;
                    pair_j[count_temp] = j;
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&count,1);
                    pair_i[count_temp] = j;
                    pair_j[count_temp] = i;
                }
            }
        }
    }
}

void ptc_nnps_mesh(SPH *sph)
//void ptc_nnps_mesh(SPH_PARTICLE *particle,SPH_PAIR *pair,unsigned int ***mesh)
{
    SPH_PARTICLE *particle;
    SPH_PAIR *pair;
    SPH_MESH mesh;
    particle = sph->particle;
    pair = sph->pair;
    mesh = sph->mesh;

    pair->total = 0;

    omp_lock_t lock;
    omp_init_lock(&lock);

    for(int j=0;j<MESH_LENGTH_NUM;j++)
    {
        for(int i=0;i<MESH_DEEPTH_NUM;i++)
        {
            for(unsigned int k=0;k<mesh[i][j][MESH_PTC_NUM-1];k++)
            {
                //mesh[i][j]-->mesh[i][j]
                for(unsigned int m=k+1;m<mesh[i][j][MESH_PTC_NUM-1];m++)
                {
                    if(PTC_DISTANCE(mesh[i][j][k],mesh[i][j][m])<=PTC_REGION_RADIUS)
                    {
                        if(particle->type[mesh[i][j][k]]==0)
			            {
                            pair->i[pair->total] = mesh[i][j][k];
                            pair->j[pair->total] = mesh[i][j][m];
                            pair->total++;
                        }
                        else if (particle->type[mesh[i][j][m]]==0)
                        {
                            pair->i[pair->total] = mesh[i][j][m];
                            pair->j[pair->total] = mesh[i][j][k];
                            pair->total++;
                        }
                    }
                }
                //mesh[i][j]-->mesh[i][j+1]
                if(j<(MESH_LENGTH_NUM-1))
                {
                    for(unsigned int m=0;m<mesh[i][j+1][MESH_PTC_NUM-1];m++)
                    {
                        if(PTC_DISTANCE(mesh[i][j][k],mesh[i][j+1][m])<=PTC_REGION_RADIUS)
                        {
                            if(particle->type[mesh[i][j][k]]==0)
			                {
                                pair->i[pair->total] = mesh[i][j][k];
                                pair->j[pair->total] = mesh[i][j+1][m];
                                pair->total++;
                            }
                            else if (particle->type[mesh[i][j+1][m]]==0)
                            {
                                pair->i[pair->total] = mesh[i][j+1][m];
                                pair->j[pair->total] = mesh[i][j][k];
                                pair->total++;
                            }
                        }
                    }
                }
                //mesh[i][j]-->mesh[i+1][j]
                if(i<(MESH_DEEPTH_NUM-1))
                {
                    for(unsigned int m=0;m<mesh[i+1][j][MESH_PTC_NUM-1];m++)
                    {
                        if(PTC_DISTANCE(mesh[i][j][k],mesh[i+1][j][m])<=PTC_REGION_RADIUS)
                        {
                            if(particle->type[mesh[i][j][k]]==0)
			                {
                                pair->i[pair->total] = mesh[i][j][k];
                                pair->j[pair->total] = mesh[i+1][j][m];
                                pair->total++;
                            }
                            else if (particle->type[mesh[i+1][j][m]]==0)
                            {
                                pair->i[pair->total] = mesh[i+1][j][m];
                                pair->j[pair->total] = mesh[i][j][k];
                                pair->total++;
                            }
                        }
                    }
                }
                //mesh[i][j]-->mesh[i+1][j+1]
                if(i < (MESH_DEEPTH_NUM-1) && j < (MESH_LENGTH_NUM-1))
                {
                    for(unsigned int m=0;m<mesh[i+1][j+1][MESH_PTC_NUM-1];m++)
                    {
                        if(PTC_DISTANCE(mesh[i][j][k],mesh[i+1][j+1][m])<=PTC_REGION_RADIUS)
                        {
                            if(particle->type[mesh[i][j][k]]==0)
			                {
                                pair->i[pair->total] = mesh[i][j][k];
                                pair->j[pair->total] = mesh[i+1][j+1][m];
                                pair->total++;
                            }
                            else if (particle->type[mesh[i+1][j+1][m]]==0)
                            {
                                pair->i[pair->total] = mesh[i+1][j+1][m];
                                pair->j[pair->total] = mesh[i][j][k];
                                pair->total++;
                            }
                        }
                    }

                }
                //mesh[i][j]-->mesh[i-1][j+1]
                if(i > 0 && j<(MESH_LENGTH_NUM-1))
                {
                    for(unsigned int m=0;m<mesh[i-1][j+1][MESH_PTC_NUM-1];m++)
                    {
                        if(PTC_DISTANCE(mesh[i][j][k],mesh[i-1][j+1][m])<=PTC_REGION_RADIUS)
                        {
                            if(particle->type[mesh[i][j][k]]==0)
			                {
                                pair->i[pair->total] = mesh[i][j][k];
                                pair->j[pair->total] = mesh[i-1][j+1][m];
                                pair->total++;
                            }
                            else if (particle->type[mesh[i-1][j+1][m]]==0)
                            {
                                pair->i[pair->total] = mesh[i-1][j+1][m];
                                pair->j[pair->total] = mesh[i][j][k];
                                pair->total++;
                            }
                        }
                    }
                }
            }
        }
    }
}