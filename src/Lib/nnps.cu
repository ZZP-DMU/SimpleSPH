#include "hip/hip_runtime.h"
#include "SPH.cuh"

void sph_nnps_cpu(SPH *sph)
{
    SPH_MESH *mesh;
    SPH_PARTICLE *particle;
    SPH_ARG *arg;
    //SPH_PAIR *pair;
    mesh = sph->mesh;
    particle = sph->particle;
    arg = sph->host_arg;
    //pair = sph->pair;

    double q=0.0;
    double dx=0.0;
    double dy=0.0;
    int mesh_id=0;
    int index_i=0;
    int index_j=0;
    arg->pair_num = 0;
    for(int i=0;i<arg->mesh_xnum;i++)
    {
        for(int j=0;j<arg->mesh_ynum;j++)
        {
            mesh_id = i+j*arg->mesh_xnum;
            for(int m=0;m<mesh->count[mesh_id];m++)
            {
                //[x,y]->[x,y]
                index_i = mesh_id + m*arg->mesh_num;
                for(int n=m+1;n<mesh->count[mesh_id];n++)
                {
                    index_j = mesh_id + n*arg->mesh_num;
                    dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                    dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                    q = sqrt(dx*dx+dy*dy)/arg->h;
                    if(q<2.0)
                    {
                        if(particle->type[mesh->ptc[index_i]]==0 || particle->type[mesh->ptc[index_j]]==0)
                        {
                            arg->pair_num ++;
                        }
                    }
                }
                //[x,y]->[x+1,y]
                if(i<arg->mesh_xnum-1)
                {
                    for(int n=0;n<mesh->count[mesh_id+1];n++)
                    {
                        index_j = mesh_id+1+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 || particle->type[mesh->ptc[index_j]]==0)
                            {
                                arg->pair_num ++;
                            }
                        } 
                    }
                }
                //[x,y]->[x,y+1]
                if(j<arg->mesh_ynum-1)
                {
                    for(int n=0;n<mesh->count[mesh_id+arg->mesh_xnum];n++)
                    {
                        index_j = mesh_id+arg->mesh_xnum+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 || particle->type[mesh->ptc[index_j]]==0)
                            {
                                arg->pair_num ++;
                            }
                        } 
                    }
                }
                //[x,y]->[x+1,y+1]
                if(i<(arg->mesh_xnum-1) && j<(arg->mesh_ynum-1))
                {
                    for(int n=0;n<mesh->count[mesh_id+1+arg->mesh_xnum];n++)
                    {
                        index_j = mesh_id+1+arg->mesh_xnum+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 || particle->type[mesh->ptc[index_j]]==0)
                            {
                                arg->pair_num ++;
                            }
                        } 
                    }
                }
                //[x,y]->[x-1,y+1]
                if(i>0 && j<(arg->mesh_ynum-1))
                {
                    for(int n=0;n<mesh->count[mesh_id-1+arg->mesh_xnum];n++)
                    {
                        index_j = mesh_id-1+arg->mesh_xnum+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 || particle->type[mesh->ptc[index_j]]==0)
                            {
                                arg->pair_num ++;
                            }
                        }
                    }
                }
            }
        }
    }
    printf("the pair num in cpu is :%d\n",arg->pair_num);
}


__global__ void sph_nnps_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{

    //blockIdx.x ---> length direction
    //blockIdx.y ---> deepth direction
    //threadIdx.x ---> search the mesh
    //threadIdx.y---> search the near mesh


    double q;
    int i,j;
    //int count_temp=0;
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    if( threadIdx.x >= cuda->mesh_count[mesh_id]) return;
    //if(threadIdx.y == 0)atomicAdd(&(arg->tmp),1);
    i = mesh_id + threadIdx.x*arg->mesh_num;
    
    //mesh[x,y]->mesh[x,y]
    if( threadIdx.y > threadIdx.x && threadIdx.y< cuda->mesh_count[mesh_id])
    {
        j = mesh_id + threadIdx.y*arg->mesh_num ;
        q = (cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])*(cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])+(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]])*(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]]);
        q = sqrt(q)/arg->h;
        if(q<2.0)
        {
            if(cuda->type[cuda->mesh[i]] == 0)
            {
                sph_lock_cuda(arg);
                cuda->pair_i[arg->pair_num] = cuda->mesh[i];
                cuda->pair_j[arg->pair_num] = cuda->mesh[j];
                arg->pair_num++;
                sph_unlock_cuda(arg);
                //count_temp = atomicAdd(&(arg->pair_num),1);
                //cuda->pair_i[count_temp] = cuda->mesh[i];
                //cuda->pair_j[count_temp] = cuda->mesh[j];
                if(cuda->mesh[i] == 0 || cuda->mesh[j] == 0)
                {
                    //printf("i:%d type:%d j:%d type:%d\n",i,cuda->type[cuda->mesh[i]],j,cuda->type[cuda->mesh[j]]);
                }
            }
            else if(cuda->type[cuda->mesh[j]] == 0)
            {
                sph_lock_cuda(arg);
                cuda->pair_i[arg->pair_num] = cuda->mesh[j];
                cuda->pair_j[arg->pair_num] = cuda->mesh[i];
                arg->pair_num++;
                sph_unlock_cuda(arg);
                //count_temp = atomicAdd(&(arg->pair_num),1);
                //cuda->pair_i[count_temp] = cuda->mesh[j];
                //cuda->pair_j[count_temp] = cuda->mesh[i];
                if(cuda->mesh[i] == 0 || cuda->mesh[j] == 0)
                {
                    //printf("i:%d type:%d j:%d type:%d\n",cuda->mesh[i],cuda->type[cuda->mesh[i]],cuda->mesh[j],cuda->type[cuda->mesh[j]]);
                    //printf("i:%d type:%d j:%d type:%d\n",i,cuda->type[cuda->mesh[i]],j,cuda->type[cuda->mesh[j]]);
                }
            }
        }
    }
    
    //mesh[x,y]->mesh[x+1,y]
    if( blockIdx.x < ( gridDim.x-1))
    {
        if( threadIdx.y< cuda->mesh_count[mesh_id+1] )
        {
            j = mesh_id + 1 + threadIdx.y*arg->mesh_num;
            q = (cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])*(cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])+(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]])*(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]]);
            q = sqrt(q)/arg->h;
            if(q<2.0)
            {
                if(cuda->type[cuda->mesh[i]] == 0)
                {
                    sph_lock_cuda(arg);
                    cuda->pair_i[arg->pair_num] = cuda->mesh[i];
                    cuda->pair_j[arg->pair_num] = cuda->mesh[j];
                    arg->pair_num++;
                    sph_unlock_cuda(arg);
                    //count_temp = atomicAdd(&(arg->pair_num),1);
                    //cuda->pair_i[count_temp] = cuda->mesh[i];
                    //cuda->pair_j[count_temp] = cuda->mesh[j];
                    if(cuda->mesh[i] == 0 || cuda->mesh[j] == 0)
                    {
                        //printf("i:%d type:%d j:%d type:%d\n",cuda->mesh[i],cuda->type[cuda->mesh[i]],cuda->mesh[j],cuda->type[cuda->mesh[j]]);
                        //printf("i:%d type:%d j:%d type:%d\n",i,cuda->type[cuda->mesh[i]],j,cuda->type[cuda->mesh[j]]);
                    }
                }
                else if(cuda->type[cuda->mesh[j]] == 0)
                {
                    sph_lock_cuda(arg);
                    cuda->pair_i[arg->pair_num] = cuda->mesh[j];
                    cuda->pair_j[arg->pair_num] = cuda->mesh[i];
                    arg->pair_num++;
                    sph_unlock_cuda(arg);
                    //count_temp = atomicAdd(&(arg->pair_num),1);
                    //cuda->pair_i[count_temp] = cuda->mesh[j];
                    //cuda->pair_j[count_temp] = cuda->mesh[i];
                    if(cuda->mesh[i] == 0 || cuda->mesh[j] == 0)
                    {
                        //printf("i:%d type:%d j:%d type:%d\n",cuda->mesh[i],cuda->type[cuda->mesh[i]],cuda->mesh[j],cuda->type[cuda->mesh[j]]);
                        //printf("i:%d type:%d j:%d type:%d\n",i,cuda->type[cuda->mesh[i]],j,cuda->type[cuda->mesh[j]]);
                    }
                }
            }
        }
    }

    //mesh[x,y]->mesh[x,y+1]
    if( blockIdx.y < ( gridDim.y-1))
    {
        if( threadIdx.y< cuda->mesh[ mesh_id + gridDim.x] )
        {
            j = mesh_id + gridDim.x + threadIdx.y*arg->mesh_num;
            q = (cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])*(cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])+(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]])*(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]]);
            q = sqrt(q)/arg->h;
            if(q<2.0)
            {
                if(cuda->type[cuda->mesh[i]] == 0)
                {
                    sph_lock_cuda(arg);
                    cuda->pair_i[arg->pair_num] = cuda->mesh[i];
                    cuda->pair_j[arg->pair_num] = cuda->mesh[j];
                    arg->pair_num++;
                    sph_unlock_cuda(arg);
                    //count_temp = atomicAdd(&(arg->pair_num),1);
                    //cuda->pair_i[count_temp] = cuda->mesh[i];
                    //cuda->pair_j[count_temp] = cuda->mesh[j];
                    if(cuda->mesh[i] == 0 || cuda->mesh[j] == 0)
                    {
                        //printf("i:%d type:%d j:%d type:%d\n",cuda->mesh[i],cuda->type[cuda->mesh[i]],cuda->mesh[j],cuda->type[cuda->mesh[j]]);
                        //printf("i:%d type:%d j:%d type:%d\n",i,cuda->type[cuda->mesh[i]],j,cuda->type[cuda->mesh[j]]);
                    }
                }
                else if(cuda->type[cuda->mesh[j]] == 0)
                {
                    sph_lock_cuda(arg);
                    cuda->pair_i[arg->pair_num] = cuda->mesh[j];
                    cuda->pair_j[arg->pair_num] = cuda->mesh[i];
                    arg->pair_num++;
                    sph_unlock_cuda(arg);
                    //count_temp = atomicAdd(&(arg->pair_num),1);
                    //cuda->pair_i[count_temp] = cuda->mesh[j];
                    //cuda->pair_j[count_temp] = cuda->mesh[i];
                    if(cuda->mesh[i] == 0 || cuda->mesh[j] == 0)
                    {
                        //printf("i:%d type:%d j:%d type:%d\n",cuda->mesh[i],cuda->type[cuda->mesh[i]],cuda->mesh[j],cuda->type[cuda->mesh[j]]);
                        //printf("i:%d type:%d j:%d type:%d\n",i,cuda->type[cuda->mesh[i]],j,cuda->type[cuda->mesh[j]]);
                    }
                }
            }
        }
    }

    //mesh[x,y]->mesh[x+1,y+1]
    if( blockIdx.x < ( gridDim.x-1) && blockIdx.y < ( gridDim.y-1))
    {
        if( threadIdx.y< cuda->mesh_count[mesh_id + 1 + gridDim.x])
        {
            j = mesh_id + 1 + gridDim.x + threadIdx.y*arg->mesh_num;
            q = (cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])*(cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])+(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]])*(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]]);
            q = sqrt(q)/arg->h;
            if(q<2.0)
            {
                if(cuda->type[cuda->mesh[i]] == 0)
                {
                    sph_lock_cuda(arg);
                    cuda->pair_i[arg->pair_num] = cuda->mesh[i];
                    cuda->pair_j[arg->pair_num] = cuda->mesh[j];
                    arg->pair_num++;
                    sph_unlock_cuda(arg);
                    //count_temp = atomicAdd(&(arg->pair_num),1);
                    //cuda->pair_i[count_temp] = cuda->mesh[i];
                    //cuda->pair_j[count_temp] = cuda->mesh[j];
                    if(cuda->mesh[i] == 0 || cuda->mesh[j] == 0)
                    {
                       // printf("i:%d type:%d j:%d type:%d\n",cuda->mesh[i],cuda->type[cuda->mesh[i]],cuda->mesh[j],cuda->type[cuda->mesh[j]]);
                       // printf("i:%d type:%d j:%d type:%d\n",i,cuda->type[cuda->mesh[i]],j,cuda->type[cuda->mesh[j]]);
                    }
                }
                else if(cuda->type[cuda->mesh[j]] == 0)
                {
                    sph_lock_cuda(arg);
                    cuda->pair_i[arg->pair_num] = cuda->mesh[j];
                    cuda->pair_j[arg->pair_num] = cuda->mesh[i];
                    arg->pair_num++;
                    sph_unlock_cuda(arg);
                    //count_temp = atomicAdd(&(arg->pair_num),1);
                    //cuda->pair_i[count_temp] = cuda->mesh[j];
                    //cuda->pair_j[count_temp] = cuda->mesh[i];
                    if(cuda->mesh[i] == 0 || cuda->mesh[j] == 0)
                    {
                        //printf("i:%d type:%d j:%d type:%d\n",cuda->mesh[i],cuda->type[cuda->mesh[i]],cuda->mesh[j],cuda->type[cuda->mesh[j]]);
                        //printf("i:%d type:%d j:%d type:%d\n",i,cuda->type[cuda->mesh[i]],j,cuda->type[cuda->mesh[j]]);
                    }
                }
            }
        }
    }

    //mesh[x,y]->mesh[x+1,y-1]
    if( blockIdx.x < ( gridDim.x-1) && blockIdx.y > 0)
    {
        if( threadIdx.y< cuda->mesh[mesh_id + 1 - gridDim.x])
        {
            j = mesh_id + 1 - gridDim.x + threadIdx.y*arg->mesh_num;
            q = (cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])*(cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])+(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]])*(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]]);
            q = sqrt(q)/arg->h;
            if(q<2.0)
            {
                if(cuda->type[cuda->mesh[i]] == 0)
                {
                    sph_lock_cuda(arg);
                    cuda->pair_i[arg->pair_num] = cuda->mesh[i];
                    cuda->pair_j[arg->pair_num] = cuda->mesh[j];
                    arg->pair_num++;
                    sph_unlock_cuda(arg);
                    //count_temp = atomicAdd(&(arg->pair_num),1);
                    //cuda->pair_i[count_temp] = cuda->mesh[i];
                    //cuda->pair_j[count_temp] = cuda->mesh[j];
                    if(cuda->mesh[i] == 0 || cuda->mesh[j] == 0)
                    {
                        //printf("i:%d type:%d j:%d type:%d\n",cuda->mesh[i],cuda->type[cuda->mesh[i]],cuda->mesh[j],cuda->type[cuda->mesh[j]]);
                        //printf("i:%d type:%d j:%d type:%d\n",i,cuda->type[cuda->mesh[i]],j,cuda->type[cuda->mesh[j]]);
                    }
                }
                else if(cuda->type[cuda->mesh[j]] == 0)
                {
                    sph_lock_cuda(arg);
                    cuda->pair_i[arg->pair_num] = cuda->mesh[j];
                    cuda->pair_j[arg->pair_num] = cuda->mesh[i];
                    arg->pair_num++;
                    sph_unlock_cuda(arg);
                    //count_temp = atomicAdd(&(arg->pair_num),1);
                    //cuda->pair_i[count_temp] = cuda->mesh[j];
                    //cuda->pair_j[count_temp] = cuda->mesh[i];
                    if(cuda->mesh[i] == 0 || cuda->mesh[j] == 0)
                    {
                        //printf("i:%d type:%d j:%d type:%d\n",cuda->mesh[i],cuda->type[cuda->mesh[i]],cuda->mesh[j],cuda->type[cuda->mesh[j]]);
                        //printf("i:%d type:%d j:%d type:%d\n",i,cuda->type[cuda->mesh[i]],j,cuda->type[cuda->mesh[j]]);
                    }
                }
            }
        }
    }
    cuda->mesh_count[mesh_id] = 0;
}
