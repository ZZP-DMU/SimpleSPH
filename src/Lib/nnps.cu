#include "hip/hip_runtime.h"
#include "SPH.cuh"

void sph_nnps_cpu(SPH *sph)
{
    SPH_MESH *mesh;
    SPH_PARTICLE *particle;
    SPH_ARG *arg;
    //SPH_PAIR *pair;
    mesh = sph->mesh;
    particle = sph->particle;
    arg = sph->host_arg;
    //pair = sph->pair;

    double q=0.0;
    double dx=0.0;
    double dy=0.0;
    int mesh_id=0;
    int index_i=0;
    int index_j=0;
    arg->pair_num = 0;
    for(int i=0;i<arg->mesh_xnum;i++)
    {
        for(int j=0;j<arg->mesh_ynum;j++)
        {
            mesh_id = i+j*arg->mesh_xnum;
            for(int m=0;m<mesh->count[mesh_id];m++)
            {
                //[x,y]->[x,y]
                index_i = mesh_id + m*arg->mesh_num;
                for(int n=m+1;n<mesh->count[mesh_id];n++)
                {
                    index_j = mesh_id + n*arg->mesh_num;
                    dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                    dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                    q = sqrt(dx*dx+dy*dy)/arg->h;
                    if(q<2.0)
                    {
                        if(particle->type[mesh->ptc[index_i]]==0 || particle->type[mesh->ptc[index_j]]==0)
                        {
                            arg->pair_num ++;
                        }
                    }
                }
                //[x,y]->[x+1,y]
                if(i<arg->mesh_xnum-1)
                {
                    for(int n=0;n<mesh->count[mesh_id+1];n++)
                    {
                        index_j = mesh_id+1+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 || particle->type[mesh->ptc[index_j]]==0)
                            {
                                arg->pair_num ++;
                            }
                        } 
                    }
                }
                //[x,y]->[x,y+1]
                if(j<arg->mesh_ynum-1)
                {
                    for(int n=0;n<mesh->count[mesh_id+arg->mesh_xnum];n++)
                    {
                        index_j = mesh_id+arg->mesh_xnum+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 || particle->type[mesh->ptc[index_j]]==0)
                            {
                                arg->pair_num ++;
                            }
                        } 
                    }
                }
                //[x,y]->[x+1,y+1]
                if(i<(arg->mesh_xnum-1) && j<(arg->mesh_ynum-1))
                {
                    for(int n=0;n<mesh->count[mesh_id+1+arg->mesh_xnum];n++)
                    {
                        index_j = mesh_id+1+arg->mesh_xnum+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 || particle->type[mesh->ptc[index_j]]==0)
                            {
                                arg->pair_num ++;
                            }
                        } 
                    }
                }
                //[x,y]->[x-1,y+1]
                if(i>0 && j<(arg->mesh_ynum-1))
                {
                    for(int n=0;n<mesh->count[mesh_id-1+arg->mesh_xnum];n++)
                    {
                        index_j = mesh_id-1+arg->mesh_xnum+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 || particle->type[mesh->ptc[index_j]]==0)
                            {
                                arg->pair_num ++;
                            }
                        }
                    }
                }
            }
        }
    }
    printf("the pair num in cpu is :%d\n",arg->pair_num);
}


__global__ void sph_nnps_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{

    //threadIdx.x---> (x)length direction
    //blockIdx.x ---> (y)depth direction

    double dx = 0.0;
    double dy = 0.0;
    double q = 0.0;
    int index_i = 0;
    int index_j = 0;
    int count_temp=0;
    if( blockIdx.x >= arg->mesh_ynum) return;
    if( threadIdx.x >= arg->mesh_xnum) return;
    const int mesh_id = threadIdx.x + blockIdx.x * blockDim.x;
    
    for(int i=0;i<cuda->mesh_count[mesh_id];i++)
    {
        index_i = mesh_id + i*arg->mesh_num;
        //(x,y)->(x,y)
        for(int j=i+1;j<cuda->mesh_count[mesh_id];j++)
        {
            index_j = mesh_id + j*arg->mesh_num;
            dx = cuda->x[cuda->mesh[index_i]] - cuda->x[cuda->mesh[index_j]];
            dy = cuda->y[cuda->mesh[index_i]] - cuda->y[cuda->mesh[index_j]];
            q = sqrt(dx*dx+dy*dy)/arg->h;
            if(q<2.0)
            {
                if(cuda->type[cuda->mesh[index_i]] == 0)
                {
                    count_temp = atomicAdd(&(arg->pair_num),1);
                    cuda->pair_i[count_temp] = cuda->mesh[index_i];
                    cuda->pair_j[count_temp] = cuda->mesh[index_j];
                }
                else if(cuda->type[cuda->mesh[index_j]] == 0)
                {
                    count_temp = atomicAdd(&(arg->pair_num),1);
                    cuda->pair_i[count_temp] = cuda->mesh[index_j];
                    cuda->pair_j[count_temp] = cuda->mesh[index_i];
                }
            }
        }
        //(x,y)->(x+1,y)
        if( threadIdx.x<(arg->mesh_xnum-1))
        {
            for(int j=0;j<cuda->mesh_count[mesh+1];j++)
            {
                index_j = mesh_id + 1 + j*arg->mesh_num;
                dx = cuda->x[cuda->mesh[index_i]] - cuda->x[cuda->mesh[index_j]];
                dy = cuda->y[cuda->mesh[index_i]] - cuda->y[cuda->mesh[index_j]];
                q = sqrt(dx*dx+dy*dy)/arg->h;
                if(q<2.0)
                {
                    if(cuda->type[cuda->mesh[index_i]] == 0)
                    {
                        count_temp = atomicAdd(&(arg->pair_num),1);
                        cuda->pair_i[count_temp] = cuda->mesh[index_i];
                        cuda->pair_j[count_temp] = cuda->mesh[index_j];
                    }
                    else if(cuda->type[cuda->mesh[index_j]] == 0)
                    {
                        count_temp = atomicAdd(&(arg->pair_num),1);
                        cuda->pair_i[count_temp] = cuda->mesh[index_j];
                        cuda->pair_j[count_temp] = cuda->mesh[index_i];
                    }
                }
            }   
        }

        //(x,y)->(x,y+1)
        if( blockIdx.x<(arg->mesh_ynum-1))
        {
            for(int j=0;j<cuda->mesh_count[mesh_id+arg->mesh_xnum];j++)
            {
                index_j = mesh_id + arg->mesh_xnum + j*arg->mesh_num;
                dx = cuda->x[cuda->mesh[index_i]] - cuda->x[cuda->mesh[index_j]];
                dy = cuda->y[cuda->mesh[index_i]] - cuda->y[cuda->mesh[index_j]];
                q = sqrt(dx*dx+dy*dy)/arg->h;
                if(q<2.0)
                {
                    if(cuda->type[cuda->mesh[index_i]] == 0)
                    {
                        count_temp = atomicAdd(&(arg->pair_num),1);
                        cuda->pair_i[count_temp] = cuda->mesh[index_i];
                        cuda->pair_j[count_temp] = cuda->mesh[index_j];
                    }
                    else if(cuda->type[cuda->mesh[index_j]] == 0)
                    {
                        count_temp = atomicAdd(&(arg->pair_num),1);
                        cuda->pair_i[count_temp] = cuda->mesh[index_j];
                        cuda->pair_j[count_temp] = cuda->mesh[index_i];
                    }
                }
            }
        }

        //(x,y)->(x+1,y+1)
        if( threadIdx.x<(arg->mesh_xnum-1) && blockIdx.x<(arg->mesh_ynum-1))
        {
            for(int j=0;j<cuda->mesh_count[mesh_id+1+arg->mesh_xnum];j++)
            {
                index_j = mesh_id + 1 + arg->mesh_xnum + j*arg->mesh_num;
                dx = cuda->x[cuda->mesh[index_i]] - cuda->x[cuda->mesh[index_j]];
                dy = cuda->y[cuda->mesh[index_i]] - cuda->y[cuda->mesh[index_j]];
                q = sqrt(dx*dx+dy*dy)/arg->h;
                if(q<2.0)
                {
                    if(cuda->type[cuda->mesh[index_i]] == 0)
                    {
                        count_temp = atomicAdd(&(arg->pair_num),1);
                        cuda->pair_i[count_temp] = cuda->mesh[index_i];
                        cuda->pair_j[count_temp] = cuda->mesh[index_j];
                    }
                    else if(cuda->type[cuda->mesh[index_j]] == 0)
                    {
                        count_temp = atomicAdd(&(arg->pair_num),1);
                        cuda->pair_i[count_temp] = cuda->mesh[index_j];
                        cuda->pair_j[count_temp] = cuda->mesh[index_i];
                    }
                }
            }
        }

        //(x,y)->(x-1,y+1)
        if( threadIdx.x>0 && blockIdx.x<(arg->mesh_ynum-1))
        {
            for(int j=0;j<cuda->mesh_count[mesh_id-1+arg->mesh_xnum];j++)
            {
                index_j = mesh_id - 1 + arg->mesh_xnum + j*arg->mesh_num;
                dx = cuda->x[cuda->mesh[index_i]] - cuda->x[cuda->mesh[index_j]];
                dy = cuda->y[cuda->mesh[index_i]] - cuda->y[cuda->mesh[index_j]];
                q = sqrt(dx*dx+dy*dy)/arg->h;
                if(q<2.0)
                {
                    if(cuda->type[cuda->mesh[index_i]] == 0)
                    {
                        count_temp = atomicAdd(&(arg->pair_num),1);
                        cuda->pair_i[count_temp] = cuda->mesh[index_i];
                        cuda->pair_j[count_temp] = cuda->mesh[index_j];
                    }
                    else if(cuda->type[cuda->mesh[index_j]] == 0)
                    {
                        count_temp = atomicAdd(&(arg->pair_num),1);
                        cuda->pair_i[count_temp] = cuda->mesh[index_j];
                        cuda->pair_j[count_temp] = cuda->mesh[index_i];
                    }
                }
            }
        }
    }
    cuda->mesh_count[mesh_id] = 0;
}
