#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_nnps_cuda(int *mesh,double *x,double *y,int *type,int *pair_i,int *pair_j)
{
    /*
    blockIdx.x ---> length direction
    blockIdx.y ---> deepth direction
    threadIdx.x ---> search the mesh
    threadIdx.y ---> search the near mesh
    */
    double q;
    int i,j;
    int count_temp;
    int mesh_ptc_num;
    int mesh_near_ptc_num;
    if( blockIdx.x >= MESH_LENGTH_NUM_CUDA || blockIdx.y >= MESH_DEEPTH_NUM_CUDA) return;
    mesh_ptc_num = mesh[ blockIdx.x + blockIdx.y*MESH_LENGTH_NUM_CUDA + MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*(MESH_PTC_NUM-1)];
    if( threadIdx.x >= mesh_ptc_num)return;
    i = blockIdx.x + blockIdx.y*MESH_LENGTH_NUM_CUDA + threadIdx.x*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
    

    //mesh[i,j]->mesh[i,j]
    mesh_near_ptc_num = mesh_ptc_num;
    if( threadIdx.y > threadIdx.x && threadIdx.y < mesh_near_ptc_num)
    {
        j = blockIdx.x + blockIdx.y*MESH_LENGTH_NUM_CUDA + threadIdx.y*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
        q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
        q = sqrt(q);
        if(q<2.0)
        {
            if(type[mesh[i]] == 0)
            {
                count_temp = atomicAdd(&dev_count,1);
                pair_i[count_temp] = i;
                pair_j[count_temp] = j;
            }
            else if(type[mesh[j]] == 0)
            {
                count_temp = atomicAdd(&dev_count,1);
                pair_i[count_temp] = j;
                pair_j[count_temp] = i;
            }
        }
    }

    //mesh[i,j]->mesh[i,j+1]
    if( blockIdx.x < (MESH_LENGTH_NUM_CUDA-1))
    {
        mesh_near_ptc_num = mesh[ (blockIdx.x+1) + blockIdx.y*MESH_LENGTH_NUM_CUDA + MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*(MESH_PTC_NUM-1)];
        if( threadIdx.y < mesh_near_ptc_num )
        {
            j = (blockIdx.x+1) + blockIdx.y*MESH_LENGTH_NUM_CUDA + threadIdx.y*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q);
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&dev_count,1);
                    pair_i[count_temp] = i;
                    pair_j[count_temp] = j;
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&dev_count,1);
                    pair_i[count_temp] = j;
                    pair_j[count_temp] = i;
                }
            }
        }
    }

    //mesh[i,j]->mesh[i+1,j]
    if( blockIdx.y < (MESH_DEEPTH_NUM_CUDA-1))
    {
        mesh_near_ptc_num = mesh[ blockIdx.x + ( blockIdx.y+1)*MESH_LENGTH_NUM_CUDA + MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*(MESH_PTC_NUM-1)];
        if( threadIdx.y < mesh_near_ptc_num )
        {
            j = blockIdx.x +( blockIdx.y+1)*MESH_LENGTH_NUM_CUDA + threadIdx.y*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q);
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&dev_count,1);
                    pair_i[count_temp] = i;
                    pair_j[count_temp] = j;
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&dev_count,1);
                    pair_i[count_temp] = j;
                    pair_j[count_temp] = i;
                }
            }
        }
    }

    //mesh[i,j]->mesh[i+1,j+1]
    if( blockIdx.x < (MESH_LENGTH_NUM_CUDA-1) && blockIdx.y < (MESH_DEEPTH_NUM_CUDA-1))
    {
        mesh_near_ptc_num = mesh[( blockIdx.x+1) + ( blockIdx.y+1)*MESH_LENGTH_NUM_CUDA + MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*(MESH_PTC_NUM-1)];
        if( threadIdx.y < mesh_near_ptc_num)
        {
            j = ( blockIdx.x+1) +( blockIdx.y+1)*MESH_LENGTH_NUM_CUDA + threadIdx.y*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q);
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&dev_count,1);
                    pair_i[count_temp] = i;
                    pair_j[count_temp] = j;
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&dev_count,1);
                    pair_i[count_temp] = j;
                    pair_j[count_temp] = i;
                }
            }
        }
    }

    //mesh[i,j]->mesh[i-1,j+1]
    if( blockIdx.x < (MESH_LENGTH_NUM_CUDA-1) && blockIdx.y > 0)
    {
        mesh_near_ptc_num = mesh[( blockIdx.x+1) + ( blockIdx.y-1)*MESH_LENGTH_NUM_CUDA + MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*(MESH_PTC_NUM-1)];
        if( threadIdx.y < mesh_near_ptc_num)
        {
            j = ( blockIdx.x+1) +( blockIdx.y-1)*MESH_LENGTH_NUM_CUDA + threadIdx.y*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q);
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&dev_count,1);
                    pair_i[count_temp] = i;
                    pair_j[count_temp] = j;
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&dev_count,1);
                    pair_i[count_temp] = j;
                    pair_j[count_temp] = i;
                }
            }
        }
    }
}
