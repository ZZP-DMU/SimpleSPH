#include "hip/hip_runtime.h"
#include "Lib.cuh"

void sph_nnps_cpu(SPH *sph)
{
    SPH_MESH *mesh;
    SPH_PARTICLE *particle;
    SPH_ARG *arg;
    SPH_PAIR *pair;
    mesh = sph->mesh;
    particle = sph->particle;
    arg = sph->host_arg;
    pair = sph->pair;

    double q=0.0;
    double dx=0.0;
    double dy=0.0;
    int mesh_id=0;
    int index_i=0;
    int index_j=0;
    arg->pair_num = 0;
    for(int i=0;i<arg->mesh_xnum;i++)
    {
        for(int j=0;j<arg->mesh_ynum;j++)
        {
            mesh_id = i+j*arg->mesh_xnum;
            for(int m=0;m<mesh->count[mesh_id];m++)
            {
                //[x,y]->[x,y]
                index_i = mesh_id + m*arg->mesh_num;
                for(int n=m+1;n<mesh->count[mesh_id];n++)
                {
                    index_j = mesh_id + n*arg->mesh_num;
                    dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                    dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                    q = sqrt(dx*dx+dy*dy)/arg->h;
                    if(q<2.0)
                    {
                        if(particle->type[mesh->ptc[index_i]]==0 )
                        {
                            pair->i[arg->pair_num] = mesh->ptc[index_i];
                            pair->j[arg->pair_num] = mesh->ptc[index_j];
                            arg->pair_num ++;
                        }
                        else if(particle->type[mesh->ptc[index_j]]==0)
                        {
                            pair->i[arg->pair_num] = mesh->ptc[index_j];
                            pair->j[arg->pair_num] = mesh->ptc[index_i];
                            arg->pair_num ++;
                        }
                    }
                }
                //[x,y]->[x+1,y]
                if(i<arg->mesh_xnum-1)
                {
                    for(int n=0;n<mesh->count[mesh_id+1];n++)
                    {
                        index_j = mesh_id+1+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 )
                            {   
                                pair->i[arg->pair_num] = mesh->ptc[index_i];
                                pair->j[arg->pair_num] = mesh->ptc[index_j];
                                arg->pair_num ++;
                            }
                            else if(particle->type[mesh->ptc[index_j]]==0)
                            {
                                pair->i[arg->pair_num] = mesh->ptc[index_j];
                                pair->j[arg->pair_num] = mesh->ptc[index_i];
                                arg->pair_num ++;
                            }
                        } 
                    }
                }
                //[x,y]->[x,y+1]
                if(j<arg->mesh_ynum-1)
                {
                    for(int n=0;n<mesh->count[mesh_id+arg->mesh_xnum];n++)
                    {
                        index_j = mesh_id+arg->mesh_xnum+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 )
                            {
                                pair->i[arg->pair_num] = mesh->ptc[index_i];
                                pair->j[arg->pair_num] = mesh->ptc[index_j];
                                arg->pair_num ++;
                            }
                            else if(particle->type[mesh->ptc[index_j]]==0)
                            {
                                pair->i[arg->pair_num] = mesh->ptc[index_j];
                                pair->j[arg->pair_num] = mesh->ptc[index_i];
                                arg->pair_num ++;
                            }
                        } 
                    }
                }
                //[x,y]->[x+1,y+1]
                if(i<(arg->mesh_xnum-1) && j<(arg->mesh_ynum-1))
                {
                    for(int n=0;n<mesh->count[mesh_id+1+arg->mesh_xnum];n++)
                    {
                        index_j = mesh_id+1+arg->mesh_xnum+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 )
                            {
                                pair->i[arg->pair_num] = mesh->ptc[index_i];
                                pair->j[arg->pair_num] = mesh->ptc[index_j];
                                arg->pair_num ++;
                            }
                            else if(particle->type[mesh->ptc[index_j]]==0)
                            {
                                pair->i[arg->pair_num] = mesh->ptc[index_j];
                                pair->j[arg->pair_num] = mesh->ptc[index_i];
                                arg->pair_num ++;
                            }
                        } 
                    }
                }
                //[x,y]->[x-1,y+1]
                if(i>0 && j<(arg->mesh_ynum-1))
                {
                    for(int n=0;n<mesh->count[mesh_id-1+arg->mesh_xnum];n++)
                    {
                        index_j = mesh_id-1+arg->mesh_xnum+n*arg->mesh_num;
                        dx = particle->x[mesh->ptc[index_i]] -particle->x[mesh->ptc[index_j]];
                        dy = particle->y[mesh->ptc[index_i]] -particle->y[mesh->ptc[index_j]];
                        q = sqrt(dx*dx+dy*dy)/arg->h;
                        if(q<2.0)
                        {
                            if(particle->type[mesh->ptc[index_i]]==0 )
                            {
                                pair->i[arg->pair_num] = mesh->ptc[index_i];
                                pair->j[arg->pair_num] = mesh->ptc[index_j];
                                arg->pair_num ++;
                            }
                            else if(particle->type[mesh->ptc[index_j]]==0)
                            {
                                pair->i[arg->pair_num] = mesh->ptc[index_j];
                                pair->j[arg->pair_num] = mesh->ptc[index_i];
                                arg->pair_num ++;
                            }
                        }
                    }
                }
            }
        }
    }
    printf("the pair num in cpu is :%d\n",arg->pair_num);
}


__global__ void sph_nnps_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    //blockIdx.x -> mesh x direction
    //blockIdx.y -> mesh y direction
    //threadIdx.x -> local mesh index
    //threadIdx.y -> near mesh index
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    int index_i = 0;
    int index_j = 0;
    double dx = 0.0;
    double dy = 0.0;
    double q = 0.0;
    int tmp_count;
    __shared__ int count;
    if(threadIdx.x == 0 && threadIdx.y == 0) count=0;
    __syncthreads();
    
    if(threadIdx.x < cuda->mesh_count[mesh_id])
    {
        index_i = cuda->mesh[mesh_id + threadIdx.x*arg->mesh_num];
        //(x,y)->(x,y)
        if( threadIdx.y> threadIdx.x && threadIdx.y<cuda->mesh_count[mesh_id])
        {
            index_j = cuda->mesh[mesh_id + threadIdx.y*arg->mesh_num];
            dx = cuda->x[index_i] - cuda->x[index_j];
            dy = cuda->y[index_i] - cuda->y[index_j];
            q = sqrt(dx*dx+dy*dy)/arg->h;
            if(q<2.0)
            {
                if(cuda->type[index_i] == 0)
                {
                    //atomicAdd(&(cuda->pair_count[mesh_id]),1);
                    tmp_count = atomicAdd(&count,1);
                    if(tmp_count >= arg->pair_volume) printf("Error in %s:%d---%d\n",__FILE__,__LINE__,tmp_count);
                    tmp_count += mesh_id*arg->pair_volume;
                    //cuda->pair_i[tmp_count] = index_i;
                    //cuda->pair_j[tmp_count] = index_j;
                    atomicExch(&(cuda->pair_i[tmp_count]),index_i);
                    atomicExch(&(cuda->pair_j[tmp_count]),index_j);
                }
                else if (cuda->type[index_j] == 0)
                {
                    tmp_count = atomicAdd(&count,1);
                    if(tmp_count >= arg->pair_volume) printf("Error in %s:%d---%d\n",__FILE__,__LINE__,tmp_count);
                    tmp_count += mesh_id*arg->pair_volume;
                    //cuda->pair_i[tmp_count] = index_j;
                    //cuda->pair_j[tmp_count] = index_i;
                    atomicExch(&(cuda->pair_i[tmp_count]),index_j);
                    atomicExch(&(cuda->pair_j[tmp_count]),index_i);
                }
            }
        }
    }
    __syncthreads();
    if(threadIdx.x < cuda->mesh_count[mesh_id])
    {
        //(x,y)->(x+1,y)
        if( blockIdx.x < ( gridDim.x-1))
        {
            if( threadIdx.y < cuda->mesh_count[mesh_id+1])
            {
                index_j = cuda->mesh[mesh_id + 1 + threadIdx.y*arg->mesh_num];
                dx = cuda->x[index_i] - cuda->x[index_j];
                dy = cuda->y[index_i] - cuda->y[index_j];
                q = sqrt(dx*dx+dy*dy)/arg->h;
                if(q<2.0)
                {
                    if(cuda->type[index_i] == 0)
                    {
                        //atomicAdd(&(cuda->pair_count[mesh_id]),1);
                        tmp_count = atomicAdd(&count,1);
                        if(tmp_count >= arg->pair_volume) printf("Error in %s:%d---%d\n",__FILE__,__LINE__,tmp_count);
                        tmp_count += mesh_id*arg->pair_volume;
                        //cuda->pair_i[tmp_count] = index_i;
                        //cuda->pair_j[tmp_count] = index_j;
                        atomicExch(&(cuda->pair_i[tmp_count]),index_i);
                        atomicExch(&(cuda->pair_j[tmp_count]),index_j);
                    }
                    else if (cuda->type[index_j] == 0)
                    {
                        tmp_count = atomicAdd(&count,1);
                        if(tmp_count >= arg->pair_volume) printf("Error in %s:%d---%d\n",__FILE__,__LINE__,tmp_count);
                        tmp_count += mesh_id*arg->pair_volume;
                        //cuda->pair_i[tmp_count] = index_j;
                        //cuda->pair_j[tmp_count] = index_i;
                        atomicExch(&(cuda->pair_i[tmp_count]),index_j);
                        atomicExch(&(cuda->pair_j[tmp_count]),index_i);
                    }
                }
            }
        }
    }
    __syncthreads();
    if(threadIdx.x < cuda->mesh_count[mesh_id])
    {
        //(x,y)->(x,y+1)
        if( blockIdx.y < ( gridDim.y -1))
        {
            if( threadIdx.y < cuda->mesh_count[mesh_id+ gridDim.x])
            {
                index_j = cuda->mesh[mesh_id + gridDim.x + threadIdx.y*arg->mesh_num];
                dx = cuda->x[index_i] - cuda->x[index_j];
                dy = cuda->y[index_i] - cuda->y[index_j];
                q = sqrt(dx*dx+dy*dy)/arg->h;
                if(q<2.0)
                {
                    if(cuda->type[index_i] == 0)
                    {
                        //atomicAdd(&(cuda->pair_count[mesh_id]),1);
                        tmp_count = atomicAdd(&count,1);
                        if(tmp_count >= arg->pair_volume) printf("Error in %s:%d---%d\n",__FILE__,__LINE__,tmp_count);
                        tmp_count += mesh_id*arg->pair_volume;
                        //cuda->pair_i[tmp_count] = index_i;
                        //cuda->pair_j[tmp_count] = index_j;
                        atomicExch(&(cuda->pair_i[tmp_count]),index_i);
                        atomicExch(&(cuda->pair_j[tmp_count]),index_j);
                    }
                    else if (cuda->type[index_j] == 0)
                    {
                        tmp_count = atomicAdd(&count,1);
                        if(tmp_count >= arg->pair_volume) printf("Error in %s:%d---%d\n",__FILE__,__LINE__,tmp_count);
                        tmp_count += mesh_id*arg->pair_volume;
                        //cuda->pair_i[tmp_count] = index_j;
                        //cuda->pair_j[tmp_count] = index_i;
                        atomicExch(&(cuda->pair_i[tmp_count]),index_j);
                        atomicExch(&(cuda->pair_j[tmp_count]),index_i);
                    }
                }
            }
        }
    }
    __syncthreads();
    if(threadIdx.x < cuda->mesh_count[mesh_id])
    {
        //(x,y)->(x+1,y+1)
        if( blockIdx.x<( gridDim.x-1) && blockIdx.y<( gridDim.y-1))
        {
            if(threadIdx.y < cuda->mesh_count[mesh_id+ 1+ gridDim.x])
            {
               index_j = cuda->mesh[mesh_id + 1+  gridDim.x + threadIdx.y*arg->mesh_num];
                dx = cuda->x[index_i] - cuda->x[index_j];
                dy = cuda->y[index_i] - cuda->y[index_j];
                q = sqrt(dx*dx+dy*dy)/arg->h;
                if(q<2.0)
                {
                    if(cuda->type[index_i] == 0)
                    {
                        //atomicAdd(&(cuda->pair_count[mesh_id]),1);
                        tmp_count = atomicAdd(&count,1);
                        if(tmp_count >= arg->pair_volume) printf("Error in %s:%d---%d\n",__FILE__,__LINE__,tmp_count);
                        tmp_count += mesh_id*arg->pair_volume;
                        //cuda->pair_i[tmp_count] = index_i;
                        //cuda->pair_j[tmp_count] = index_j;
                        atomicExch(&(cuda->pair_i[tmp_count]),index_i);
                        atomicExch(&(cuda->pair_j[tmp_count]),index_j);
                    }
                    else if (cuda->type[index_j] == 0)
                    {
                        tmp_count = atomicAdd(&count,1);
                        if(tmp_count >= arg->pair_volume) printf("Error in %s:%d---%d\n",__FILE__,__LINE__,tmp_count);
                        tmp_count += mesh_id*arg->pair_volume;
                        //cuda->pair_i[tmp_count] = index_j;
                        //cuda->pair_j[tmp_count] = index_i;
                        atomicExch(&(cuda->pair_i[tmp_count]),index_j);
                        atomicExch(&(cuda->pair_j[tmp_count]),index_i);
                    }
                } 
            }
        }
    }
    __syncthreads();
    if(threadIdx.x < cuda->mesh_count[mesh_id])
    {
        //(x,y)->(x-1,y+1)
        if( blockIdx.x>0 && blockIdx.y<( gridDim.y-1))
        {
            if(threadIdx.y < cuda->mesh_count[mesh_id- 1+ gridDim.x])
            {
                index_j = cuda->mesh[mesh_id - 1+  gridDim.x + threadIdx.y*arg->mesh_num];
                dx = cuda->x[index_i] - cuda->x[index_j];
                dy = cuda->y[index_i] - cuda->y[index_j];
                q = sqrt(dx*dx+dy*dy)/arg->h;
                if(q<2.0)
                {
                    if(cuda->type[index_i] == 0)
                    {
                        //atomicAdd(&(cuda->pair_count[mesh_id]),1);
                        tmp_count = atomicAdd(&count,1);
                        if(tmp_count >= arg->pair_volume) printf("Error in %s:%d---%d\n",__FILE__,__LINE__,tmp_count);
                        tmp_count += mesh_id*arg->pair_volume;
                        //cuda->pair_i[tmp_count] = index_i;
                        //cuda->pair_j[tmp_count] = index_j;
                        atomicExch(&(cuda->pair_i[tmp_count]),index_i);
                        atomicExch(&(cuda->pair_j[tmp_count]),index_j);
                    }
                    else if (cuda->type[index_j] == 0)
                    {
                        tmp_count = atomicAdd(&count,1);
                        if(tmp_count >= arg->pair_volume) printf("Error in %s:%d---%d\n",__FILE__,__LINE__,tmp_count);
                        tmp_count += mesh_id*arg->pair_volume;
                        //cuda->pair_i[tmp_count] = index_j;
                        //cuda->pair_j[tmp_count] = index_i;
                        atomicExch(&(cuda->pair_i[tmp_count]),index_j);
                        atomicExch(&(cuda->pair_j[tmp_count]),index_i);
                    }
                } 
            }
        }
    }
    __syncthreads();
    if( threadIdx.x == 0 && threadIdx.y == 0)
    {
        atomicAdd(&(arg->pair_num),count);
        cuda->pair_count[mesh_id]=count;
        cuda->mesh_count[mesh_id]=0;
    }
    __syncthreads();
}