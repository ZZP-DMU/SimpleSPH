#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_nnps_cuda(int *mesh,double *x,double *y,int *type,int *pair_i,int *pair_j,int *count)
{
    /*
    blockIdx.x ---> length direction
    blockIdx.y ---> deepth direction
    threadIdx.x ---> search the mesh
    blockIdx.z---> search the near mesh
    */
    double q;
    int i,j;
    int count_temp;
    int mesh_ptc_num;
    int mesh_near_ptc_num;
    if( blockIdx.x >= dev_mesh_lnum || blockIdx.y >= dev_mesh_dnum ) return;
    mesh_ptc_num = mesh[ blockIdx.x + blockIdx.y*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
    mesh[ blockIdx.x + blockIdx.y*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)] = 0;
    if( threadIdx.x >= mesh_ptc_num)return;
    i = blockIdx.x + blockIdx.y*dev_mesh_lnum + threadIdx.x*dev_mesh_tnum;
    

    //mesh[i,j]->mesh[i,j]
    mesh_near_ptc_num = mesh_ptc_num;
    if( blockIdx.z> threadIdx.x && blockIdx.z< mesh_near_ptc_num)
    {
        j = blockIdx.x + blockIdx.y*dev_mesh_lnum + blockIdx.z*dev_mesh_tnum;
        q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
        q = sqrt(q)/PTC_SML;
        if(q<2.0)
        {
            if(type[mesh[i]] == 0)
            {
                count_temp = atomicAdd(count,1);
                pair_i[count_temp] = mesh[i];
                pair_j[count_temp] = mesh[j];
            }
            else if(type[mesh[j]] == 0)
            {
                count_temp = atomicAdd(count,1);
                pair_i[count_temp] = mesh[j];
                pair_j[count_temp] = mesh[i];
            }
        }
    }

    //mesh[i,j]->mesh[i,j+1]
    if( blockIdx.x < (dev_mesh_lnum-1))
    {
        mesh_near_ptc_num = mesh[ (blockIdx.x+1) + blockIdx.y*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( blockIdx.z< mesh_near_ptc_num )
        {
            j = ( blockIdx.x +1) + blockIdx.y*dev_mesh_lnum + blockIdx.z*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/PTC_SML;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[i,j]->mesh[i+1,j]
    if( blockIdx.y < (dev_mesh_dnum-1))
    {
        mesh_near_ptc_num = mesh[ blockIdx.x + ( blockIdx.y+1)*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( blockIdx.z< mesh_near_ptc_num )
        {
            j = blockIdx.x +( blockIdx.y+1)*dev_mesh_lnum + blockIdx.z*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/PTC_SML;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[i,j]->mesh[i+1,j+1]
    if( blockIdx.x < (dev_mesh_lnum-1) && blockIdx.y < (dev_mesh_dnum-1))
    {
        mesh_near_ptc_num = mesh[( blockIdx.x+1) + ( blockIdx.y+1)*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( blockIdx.z< mesh_near_ptc_num)
        {
            j = ( blockIdx.x+1) +( blockIdx.y+1)*dev_mesh_lnum + blockIdx.z*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/PTC_SML;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[i,j]->mesh[i-1,j+1]
    if( blockIdx.x < (dev_mesh_lnum-1) && blockIdx.y > 0)
    {
        mesh_near_ptc_num = mesh[( blockIdx.x+1) + ( blockIdx.y-1)*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( blockIdx.z< mesh_near_ptc_num)
        {
            j = ( blockIdx.x+1) +( blockIdx.y-1)*dev_mesh_lnum + blockIdx.z*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/PTC_SML;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }
}
