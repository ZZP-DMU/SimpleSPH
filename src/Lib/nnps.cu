#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_nnps_cuda(int *mesh,double *x,double *y,int *type,int *pair_i,int *pair_j,int *count)
{
    /*
    blockIdx.x ---> length direction
    blockIdx.y ---> deepth direction
    blockIdx.z ---> search the mesh
    threadIdx.x---> search the near mesh
    */
    double q;
    int i,j;
    int count_temp=0;
    int mesh_ptc_num;
    int mesh_near_ptc_num;
    if( blockIdx.x >= dev_mesh_lnum || blockIdx.y >= dev_mesh_dnum ) return;
    mesh_ptc_num = mesh[ blockIdx.x + blockIdx.y*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
    mesh[ blockIdx.x + blockIdx.y*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)] = 0;
    if( blockIdx.z >= mesh_ptc_num)return;
    i = blockIdx.x + blockIdx.y*dev_mesh_lnum + blockIdx.z*dev_mesh_tnum;
    

    //mesh[i,j]->mesh[i,j]
    mesh_near_ptc_num = mesh_ptc_num;
    if( threadIdx.x > blockIdx.z && threadIdx.x< mesh_near_ptc_num)
    {
        j = blockIdx.x + blockIdx.y*dev_mesh_lnum + threadIdx.x*dev_mesh_tnum;
        q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
        q = sqrt(q)/dev_h;
        if(q<2.0)
        {
            if(type[mesh[i]] == 0)
            {
                count_temp = atomicAdd(count,1);
                pair_i[count_temp] = mesh[i];
                pair_j[count_temp] = mesh[j];
            }
            else if(type[mesh[j]] == 0)
            {
                count_temp = atomicAdd(count,1);
                pair_i[count_temp] = mesh[j];
                pair_j[count_temp] = mesh[i];
            }
        }
    }

    //mesh[i,j]->mesh[i,j+1]
    if( blockIdx.x < (dev_mesh_lnum-1))
    {
        mesh_near_ptc_num = mesh[ (blockIdx.x+1) + blockIdx.y*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( threadIdx.x< mesh_near_ptc_num )
        {
            j = ( blockIdx.x +1) + blockIdx.y*dev_mesh_lnum + threadIdx.x*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[i,j]->mesh[i+1,j]
    if( blockIdx.y < (dev_mesh_dnum-1))
    {
        mesh_near_ptc_num = mesh[ blockIdx.x + ( blockIdx.y+1)*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( threadIdx.x< mesh_near_ptc_num )
        {
            j = blockIdx.x +( blockIdx.y+1)*dev_mesh_lnum + threadIdx.x*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[i,j]->mesh[i+1,j+1]
    if( blockIdx.x < (dev_mesh_lnum-1) && blockIdx.y < (dev_mesh_dnum-1))
    {
        mesh_near_ptc_num = mesh[( blockIdx.x+1) + ( blockIdx.y+1)*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( threadIdx.x< mesh_near_ptc_num)
        {
            j = ( blockIdx.x+1) +( blockIdx.y+1)*dev_mesh_lnum + threadIdx.x*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[i,j]->mesh[i-1,j+1]
    if( blockIdx.x < (dev_mesh_lnum-1) && blockIdx.y > 0)
    {
        mesh_near_ptc_num = mesh[( blockIdx.x+1) + ( blockIdx.y-1)*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( threadIdx.x< mesh_near_ptc_num)
        {
            j = ( blockIdx.x+1) +( blockIdx.y-1)*dev_mesh_lnum + threadIdx.x*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }
}
