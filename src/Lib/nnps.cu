
#include <hip/hip_runtime.h>
/*#include "Lib.cuh"

__global__ void sph_nnps_cuda(int *mesh,double *x,double *y,int *type,int *pair_i,int *pair_j,int *count)
{

    //blockIdx.x ---> length direction
    //blockIdx.y ---> deepth direction
    //threadIdx.x ---> search the mesh
    //threadIdx.y---> search the near mesh
    k ---> search the near mesh


    double q;
    int i,j;
    int count_temp=0;
    int mesh_local_num;
    int mesh_near_num;
    //if( blockIdx.x >= dev_mesh_lnum || blockIdx.y >= dev_mesh_dnum ) return;
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;


    mesh_local_num = mesh[ mesh_id + dev_mesh_tnum*(MESH_PTC_NUM-2)];
    mesh[ mesh_id + dev_mesh_tnum*(MESH_PTC_NUM-2)] = 0;
    if( threadIdx.x >= mesh_local_num)return;
    i = mesh_id + threadIdx.x*dev_mesh_tnum;
    
    //mesh[x,y]-->mesh[x,y]
    mesh_near_num = mesh_local_num;
    for(int k=blockIdx.x+1;k<mesh_near_num;k++)
    {
        j = mesh_id + k * dev_mesh_tnum;
        q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
        q = sqrt(q)/dev_h;
        if(q<2.0)
        {
            if(type[mesh[i]] == 0)
            {
                count_temp = atomicAdd(count,1);
                pair_i[count_temp] = mesh[i];
                pair_j[count_temp] = mesh[j];
            }
            else if(type[mesh[j]] == 0)
            {
                count_temp = atomicAdd(count,1);
                pair_i[count_temp] = mesh[j];
                pair_j[count_temp] = mesh[i];
            }
        } 
    }

    //mesh[x,y]-->mesh[x+1,y]
    if( blockIdx.x < ( gridDim.x-1))
    {
        mesh_near_num = mesh[mesh_id + 1 + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        for(int k=0;k<mesh_near_num;k++)
        {
            j = mesh_id + 1 +dev_mesh_tnum*k;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[x,y]-->mesh[x+1,y+1]
    if( blockIdx.x<( gridDim.x-1) && blockIdx.y<( gridDim.y-1))
    {
        mesh_near_num = mesh[mesh_id + 1 + gridDim.x + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        for(int k=0;k<mesh_near_num;k++)
        {
            j = mesh_id + 1 + gridDim.x + dev_mesh_tnum*k;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[x,y]-->mesh[x,y+1]
    if( blockIdx.y<( gridDim.y-1))
    {
        mesh_near_num = mesh[mesh_id + gridDim.x + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        for(int k=0;k<mesh_near_num;k++)
        {
            j = mesh_id + gridDim.x + dev_mesh_tnum*k;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[x,y]-->mesh[x-1,y+1]
    if( blockIdx.x>0 && blockIdx.y<( gridDim.y-1))
    {
        mesh_near_num = mesh[mesh_id - 1 + gridDim.x + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        for(int k=0;k<mesh_near_num;k++)
        {
            j = mesh_id - 1 + gridDim.x + dev_mesh_tnum*k;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }
*/


    /*
    //mesh[i,j]->mesh[i,j]
    mesh_near_num = mesh_local_num;
    if( threadIdx.y > threadIdx.x && threadIdx.y< mesh_near_num)
    {
        j = blockIdx.x + blockIdx.y*dev_mesh_lnum + threadIdx.y*dev_mesh_tnum;
        q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
        q = sqrt(q)/dev_h;
        if(q<2.0)
        {
            if(type[mesh[i]] == 0)
            {
                count_temp = atomicAdd(count,1);
                pair_i[count_temp] = mesh[i];
                pair_j[count_temp] = mesh[j];
            }
            else if(type[mesh[j]] == 0)
            {
                count_temp = atomicAdd(count,1);
                pair_i[count_temp] = mesh[j];
                pair_j[count_temp] = mesh[i];
            }
        }
    }

    //mesh[i,j]->mesh[i,j+1]
    if( blockIdx.x < (dev_mesh_lnum-1))
    {
        mesh_near_num = mesh[ (blockIdx.x+1) + blockIdx.y*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( threadIdx.y< mesh_near_num )
        {
            j = ( blockIdx.x +1) + blockIdx.y*dev_mesh_lnum + threadIdx.y*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[i,j]->mesh[i+1,j]
    if( blockIdx.y < (dev_mesh_dnum-1))
    {
        mesh_near_num = mesh[ blockIdx.x + ( blockIdx.y+1)*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( threadIdx.y< mesh_near_num )
        {
            j = blockIdx.x +( blockIdx.y+1)*dev_mesh_lnum + threadIdx.y*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[i,j]->mesh[i+1,j+1]
    if( blockIdx.x < (dev_mesh_lnum-1) && blockIdx.y < (dev_mesh_dnum-1))
    {
        mesh_near_num = mesh[( blockIdx.x+1) + ( blockIdx.y+1)*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( threadIdx.y< mesh_near_num)
        {
            j = ( blockIdx.x+1) +( blockIdx.y+1)*dev_mesh_lnum + threadIdx.y*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

    //mesh[i,j]->mesh[i-1,j+1]
    if( blockIdx.x < (dev_mesh_lnum-1) && blockIdx.y > 0)
    {
        mesh_near_num = mesh[( blockIdx.x+1) + ( blockIdx.y-1)*dev_mesh_lnum + dev_mesh_tnum*(MESH_PTC_NUM-2)];
        if( threadIdx.y< mesh_near_num)
        {
            j = ( blockIdx.x+1) +( blockIdx.y-1)*dev_mesh_lnum + threadIdx.y*dev_mesh_tnum;
            q = (x[mesh[i]]-x[mesh[j]])*(x[mesh[i]]-x[mesh[j]])+(y[mesh[i]]-y[mesh[j]])*(y[mesh[i]]-y[mesh[j]]);
            q = sqrt(q)/dev_h;
            if(q<2.0)
            {
                if(type[mesh[i]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[i];
                    pair_j[count_temp] = mesh[j];
                }
                else if(type[mesh[j]] == 0)
                {
                    count_temp = atomicAdd(count,1);
                    pair_i[count_temp] = mesh[j];
                    pair_j[count_temp] = mesh[i];
                }
            }
        }
    }

}*/
