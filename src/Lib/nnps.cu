#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_nnps_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{

    //blockIdx.x ---> length direction
    //blockIdx.y ---> deepth direction
    //threadIdx.x ---> search the mesh
    //threadIdx.y---> search the near mesh


    double q;
    int i,j;
    int count_temp=0;
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    if( threadIdx.x >= cuda->mesh_count[mesh_id]) return;
    i = mesh_id + threadIdx.x*arg->mesh_num;
    
    //mesh[x,y]->mesh[x,y]
    if( threadIdx.y > threadIdx.x && threadIdx.y< cuda->mesh_count[mesh_id])
    {
        j = mesh_id + threadIdx.y*arg->mesh_num ;
        q = (cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])*(cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])+(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]])*(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]]);
        q = sqrt(q)/arg->h;
        if(q<2.0)
        {
            if(cuda->type[cuda->mesh[i]] == 0)
            {
                count_temp = atomicAdd(&(arg->pair_num),1);
                cuda->pair_i[count_temp] = cuda->mesh[i];
                cuda->pair_j[count_temp] = cuda->mesh[j];
            }
            else if(cuda->type[cuda->mesh[j]] == 0)
            {
                count_temp = atomicAdd(&(arg->pair_num),1);
                cuda->pair_i[count_temp] = cuda->mesh[j];
                cuda->pair_j[count_temp] = cuda->mesh[i];
            }
        }
    }

    //mesh[x,y]->mesh[x+1,y]
    if( blockIdx.x < ( gridDim.x-1))
    {
        if( threadIdx.y< cuda->mesh_count[mesh_id+1] )
        {
            j = mesh_id + 1 + threadIdx.y*arg->mesh_num;
            q = (cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])*(cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])+(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]])*(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]]);
            q = sqrt(q)/arg->h;
            if(q<2.0)
            {
                if(cuda->type[cuda->mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&(arg->pair_num),1);
                    cuda->pair_i[count_temp] = cuda->mesh[i];
                    cuda->pair_j[count_temp] = cuda->mesh[j];
                }
                else if(cuda->type[cuda->mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&(arg->pair_num),1);
                    cuda->pair_i[count_temp] = cuda->mesh[j];
                    cuda->pair_j[count_temp] = cuda->mesh[i];
                }
            }
        }
    }

    //mesh[x,y]->mesh[x,y+1]
    if( blockIdx.y < ( gridDim.y-1))
    {
        mesh_near_num = mesh[ mesh_id + gridDim.x];
        if( threadIdx.y< mesh_near_num )
        {
            j = mesh_id + gridDim.x + threadIdx.y*arg->mesh_num;
            q = (cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])*(cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])+(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]])*(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]]);
            q = sqrt(q)/arg->h;
            if(q<2.0)
            {
                if(cuda->type[cuda->mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&(arg->pair_num),1);
                    cuda->pair_i[count_temp] = cuda->mesh[i];
                    cuda->pair_j[count_temp] = cuda->mesh[j];
                }
                else if(cuda->type[cuda->mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&(arg->pair_num),1);
                    cuda->pair_i[count_temp] = cuda->mesh[j];
                    cuda->pair_j[count_temp] = cuda->mesh[i];
                }
            }
        }
    }

    //mesh[x,y]->mesh[x+1,y+1]
    if( blockIdx.x < ( gridDim.x-1) && blockIdx.y < ( gridDim.y-1))
    {
        
        if( threadIdx.y< cuda->mesh_count[mesh_id + 1 + gridDim.x])
        {
            j = mesh_id + 1 + gridDim.x + threadIdx.y*arg->mesh_num;
            q = (cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])*(cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])+(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]])*(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]]);
            q = sqrt(q)/arg->h;
            if(q<2.0)
            {
                if(cuda->type[cuda->mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&(arg->pair_num),1);
                    cuda->pair_i[count_temp] = cuda->mesh[i];
                    cuda->pair_j[count_temp] = cuda->mesh[j];
                }
                else if(cuda->type[cuda->mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&(arg->pair_num),1);
                    cuda->pair_i[count_temp] = cuda->mesh[j];
                    cuda->pair_j[count_temp] = cuda->mesh[i];
                }
            }
        }
    }

    //mesh[x,y]->mesh[x+1,y-1]
    if( blockIdx.x < ( gridDim.x-1) && blockIdx.y > 0)
    {
        if( threadIdx.y< cuda->mesh[mesh_id + 1 - gridDim.x])
        {
            j = mesh_id + 1 - gridDim.x + threadIdx.y*arg->mesh_num;
            q = (cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])*(cuda->x[cuda->mesh[i]]-cuda->x[cuda->mesh[j]])+(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]])*(cuda->y[cuda->mesh[i]]-cuda->y[cuda->mesh[j]]);
            q = sqrt(q)/arg->h;
            if(q<2.0)
            {
                if(cuda->type[cuda->cuda->mesh[i]] == 0)
                {
                    count_temp = atomicAdd(&(arg->pair_num),1);
                    cuda->pair_i[count_temp] = cuda->mesh[i];
                    cuda->pair_j[count_temp] = cuda->mesh[j];
                }
                else if(cuda->type[cuda->mesh[j]] == 0)
                {
                    count_temp = atomicAdd(&(arg->pair_num),1);
                    cuda->pair_i[count_temp] = cuda->mesh[j];
                    cuda->pair_j[count_temp] = cuda->mesh[i];
                }
            }
        }
    }
    cuda->mesh[mesh_id] = 0;
}
