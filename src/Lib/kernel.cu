#include "hip/hip_runtime.h"
#include "Lib.cuh"

/*
__global__ void sph_kernel_cuda(double *x,double *y,double *w,double *dwdx,double *dwdy,double *ptc_w,int *pair_i,int *pair_j,int* pair_num)
{
    double dx,dy,q;
    const int id = threadIdx.x + blockIdx.x* blockDim.x;
    if(id >= pair_num[0]) return;

    dx = x[pair_i[id]]-x[pair_j[id]];
    dy = y[pair_i[id]]-y[pair_j[id]];
    q = sqrt(dx*dx+dy*dy)/dev_h;

    if(q<1.0)
    {
        w[id] = dev_a*(2.0/3.0-q*q+0.5*q*q*q);
        dwdx[id] = dev_a*((-2.0+1.5*q)*dx)/pow(dev_h,2);
        dwdy[id] = dev_a*((-2.0+1.5*q)*dy)/pow(dev_h,2);
        atomicAdd(&ptc_w[pair_i[id]],w[id]);
        atomicAdd(&ptc_w[pair_j[id]],w[id]);
        /*
            kernel->w[i] = a*(2.0/3.0-q*q+0.5*q*q*q);
            //each pair's differential kernel value in x and y direction
            kernel->dwdx[i] = a*((-2.0+1.5*q)*dx/pow(dev_h,2));
            kernel->dwdy[i] = a*((-2.0+1.5*q)*dy/pow(dev_h,2));
        */
    }
    else if(q<2.0)
    {
        w[id] = dev_a*((2.0-q)*(2.0-q)*(2.0-q))/6.0;
        dwdx[id] = -dev_a*0.5*((2.0-q)*(2.0-q)*dx)/(dev_h*dev_h*q);
        dwdy[id] = -dev_a*0.5*((2.0-q)*(2.0-q)*dy)/(dev_h*dev_h*q);
        atomicAdd(&ptc_w[pair_i[id]],w[id]);
        atomicAdd(&ptc_w[pair_j[id]],w[id]);
        /*
            //each pair's kernel value
            kernel->w[i] = a*((2.0-q)*(2.0-q)*(2.0-q)/6.0); 
            //each pair's differential kernel value in x and y direction
            kernel->dwdx[i] = -a*0.5*((2.0-q)*(2.0-q)*dx/(dev_h*r));
            kernel->dwdy[i] = -a*0.5*((2.0-q)*(2.0-q)*dy/(dev_h*r));
        */
    }
    else
    {
        w[id] = dwdx[id] = dwdy[id] = 0.0;
    }
}*/