#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_kernel_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    double dx = 0.0;
    double dy = 0.0;
    double r = 0.0;
    double q = 0.0;
    int index_i = 0;
    int index_j = 0;
    int id = 0;
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    if( threadIdx.x < cuda->pair_count[mesh_id])
    {
        id = mesh_id * arg->pair_volume + threadIdx.x;
        index_i = cuda->pair_i[id];
        index_j = cuda->pair_j[id];

        dx = cuda->x[index_i] - cuda->x[index_j];
        dy = cuda->y[index_i] - cuda->y[index_j];
        r = sqrt(dx*dx+dy*dy);
        q = r/arg->h;

        if(q <= 2.0)
        {
            cuda->pair_w[id] = arg->alpha*((1.0+2.0*q)*pow((1.0-0.5*q),4));
            cuda->dwdx[id] = arg->alpha*(2.0*pow((1.0-0.5*q),4)-2.0*(1.0+2.0*q)*pow((1.0-0.5*q),3))*dx/(r*arg->h);
            cuda->dwdy[id] = arg->alpha*(2.0*pow((1.0-0.5*q),4)-2.0*(1.0+2.0*q)*pow((1.0-0.5*q),3))*dy/(r*arg->h);
        }
        else
        {
            cuda->pair_w[id] = 0;
            cuda->dwdx[id] = 0;
            cuda->dwdy[id] = 0;
        }
    }
    __syncthreads();
}

__global__ void sph_sum_w(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    int index_i = 0;
    int index_j = 0;
    int id = 0;
    double ptc_w_i = 0.0;
    double ptc_w_j = 0.0;
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    if( threadIdx.x < cuda->pair_count[mesh_id])
    {
        id = mesh_id * arg->pair_volume + threadIdx.x;
        index_i = cuda->pair_i[id];
        index_j = cuda->pair_j[id];

        ptc_w_i = arg->m*cuda->pair_w[id]/cuda->rho[index_j];
        if(cuda->type[index_j] == 0) ptc_w_j = arg->m*cuda->pair_w[id]/cuda->rho[index_i];
        else ptc_w_j = cuda->pair_w[id];

        atomicAdd(&(cuda->ptc_w[index_i]),ptc_w_i);
        atomicAdd(&(cuda->ptc_w[index_j]),ptc_w_j);
    }
    __syncthreads();
}
/*if(q<=1.0)
    {
        cuda->pair_w[id] = arg->alpha*(2.0/3.0-q*q+0.5*q*q*q);
        cuda->dwdx[id] = arg->alpha*((-2.0+1.5*q)*dx/pow(arg->h,2));
        cuda->dwdy[id] = arg->alpha*((-2.0+1.5*q)*dy/pow(arg->h,2));
    }
    else if(1.0 <q && q < 2.0)
    {
        cuda->pair_w[id] = arg->alpha*((2.0-q)*(2.0-q)*(2.0-q)/6.0);
        cuda->dwdx[id] = -arg->alpha*0.5*((2.0-q)*(2.0-q)*dx/(arg->h*r));
        cuda->dwdy[id] = -arg->alpha*0.5*((2.0-q)*(2.0-q)*dy/(arg->h*r));
    }
    else
    {
        cuda->pair_w[id] = 0;
        cuda->dwdx[id] = 0;
        cuda->dwdy[id] = 0;
    }*/


//sum ptc w here,but get wrong result
/*atomicAdd(&(cuda->ptc_w[index_i]),cuda->pair_w[id]*arg->m/cuda->rho[index_j]);
    if(cuda->type[index_j] != 0)
    {
        atomicAdd(&(cuda->ptc_w[index_j]),cuda->pair_w[id]);
    }
    else
    {
        atomicAdd(&(cuda->ptc_w[index_j]),cuda->pair_w[id]*arg->m/cuda->rho[index_i]);
    }*/