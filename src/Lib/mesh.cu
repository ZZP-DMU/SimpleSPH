#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_mesh_cuda(double *x,double *y,int *mesh,int ptc_num)
{
    //const int bid = blockIdx.x;
    //const int tid = threadIdx.x;
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= ptc_num) return;

    int mid;

    if(y[id] < TOL_DOMAIN_DEEPTH && y[id] >= 0)
    {
        mid = __double2int_rz(y[id]/MESH_SPACING)*MESH_LENGTH_NUM_CUDA;
    }
    else
    {
        mid = (MESH_DEEPTH_NUM_CUDA - 1)*MESH_LENGTH_NUM_CUDA;
    }
    if(x[id] < TOL_DOMAIN_LENGTH && x[id] >= 0)
    {
        mid += __double2int_rz(x[id]/MESH_SPACING);
    }
    else
    {
        mid += MESH_LENGTH_NUM_CUDA - 1;
    }
    mid += MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA*atomicAdd(&mesh[mid+(MESH_PTC_NUM-1)*MESH_DEEPTH_NUM_CUDA*MESH_LENGTH_NUM_CUDA],1);
    mesh[mid] = id;
}

