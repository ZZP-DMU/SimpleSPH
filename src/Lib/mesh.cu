#include "hip/hip_runtime.h"
#include "Lib.cuh"
using namespace std;

__global__ void ptc_mesh_cuda(double *x,double *y,int *mesh,int ptc_num)
{
    //const int bid = blockIdx.x;
    //const int tid = threadIdx.x;
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id > ptc_num) return;

    int mid;

    if(y[id] < TOL_DOMAIN_DEEPTH && y[id] >= 0)
    {
        mid = __double2int_rz(y[id]/MESH_SPACING)*MESH_LENGTH_NUM;
    }
    else if(y[id] >= TOL_DOMAIN_DEEPTH)
    {
        mid = (MESH_DEEPTH_NUM - 1)*MESH_LENGTH_NUM;
    }
    if(x[id] < TOL_DOMAIN_LENGTH && x[id] >= 0)
    {
        mid += __double2int_rz(x[id]/MESH_SPACING);
    }
    else if(x[id] >= TOL_DOMAIN_LENGTH)
    {
        mid += MESH_LENGTH_NUM - 1;
    }
    mid += MESH_DEEPTH_NUM*MESH_LENGTH_NUM*(&mesh[mid+MESH_PTC_NUM],1);
    mesh[mid] = id;
    /*
    head = mesh[j][k][MESH_PTC_NUM-1];
    if(head<MESH_PTC_NUM-1)
    {
        mesh[j][k][head] = i;        
        mesh[j][k][MESH_PTC_NUM-1]++;
    }*/
    
    

}