#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_mesh_cuda(SPH_CUDA *cuda,SPH_ARG *arg)
{
    //const int bid = blockIdx.x;
    //const int tid = threadIdx.x;
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= arg->ptc_num) return;

    /*这里需要进行加速度和密度变化的初始化*/

    /*这里需要对pair_num进行初始化*/
    if(id == 0) arg->pair_num = 0;

    int mid = 0;

    if(cuda->y[id] < arg->domain_y && cuda->y[id] >= 0.0)
    {
        mid = __double2int_rz(cuda->y[id]/arg->mesh_dx)*arg->mesh_xnum;
    }
    else
    {
        mid = (arg->mesh_ynum - 1)*arg->mesh_xnum;
    }
    if(cuda->x[id] < arg->domain_x && cuda->x[id] >= 0.0)
    {
        mid += __double2int_rz(cuda->x[id]/arg->mesh_dx);
    }
    else
    {
        mid += arg->mesh_xnum - 1;
    }
    sph_cuda_lock(arg);

    cuda->mesh[mid + arg->mesh_num*cuda->mesh_count[mid]] = id;
    cuda->mesh_count[mid] += 1;

    sph_cuda_unlock(arg);
}

