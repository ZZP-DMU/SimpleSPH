#include "hip/hip_runtime.h"
#include "SPH.cuh"

__global__ void sph_fuck_you(SPH_CUDA *cuda,SPH_ARG *arg)
{
    //const int bid = blockIdx.x;
    //const int tid = threadIdx.x;
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= arg->ptc_num) return;

    /*这里需要进行加速度和密度变化的初始化*/

    /*这里需要对pair_num进行初始化*/
    if(id == 0) arg->pair_num = 0;

    int mid = 0;

    if(cuda->y[id] < arg->domain_y && cuda->y[id] >= 0.0)
    {
        mid = __double2int_rz(cuda->y[id]/arg->mesh_dx)*arg->mesh_xnum;
    }
    else
    {
        mid = (arg->mesh_ynum - 1)*arg->mesh_xnum;
    }
    if(cuda->x[id] < arg->domain_x && cuda->x[id] >= 0.0)
    {
        mid += __double2int_rz(cuda->x[id]/arg->mesh_dx);
    }
    else
    {
        mid += arg->mesh_xnum - 1;
    }

    while(!atomicCAS(&arg->lock,1,0))
    {
        continue;
    } 
    cuda->mesh[mid + arg->mesh_num*cuda->mesh_count[mid]] = id;
    cuda->mesh_count[mid] += 1; 
    atomicCAS(&arg->lock,0,1);
}

