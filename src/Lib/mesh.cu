#include "hip/hip_runtime.h"
#include "SPH.cuh"

__global__ void sph_mesh_cuda(SPH_CUDA *cuda,SPH_ARG *arg)
{
    //const int bid = blockIdx.x;
    //const int tid = threadIdx.x;
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= arg->ptc_num) return;

    /*这里需要进行加速度和密度变化的初始化*/

    /*这里需要对pair_num进行初始化*/
    if(id == 0) 
    {
        printf("the arg tmp is:%d the pair num is:%d \n",arg->tmp,arg->pair_num);
        //printf("the pair num is:%d \n",arg->pair_num);
        arg->tmp = 0;
        arg->pair_num = 0;
    }

    int mid = 0;
    int mesh_index = 0;

    if(cuda->y[id] < arg->domain_y && cuda->y[id] >= 0.0)
    {
        mid = __double2int_rz(cuda->y[id]/arg->mesh_dx)*arg->mesh_xnum;
    }
    else
    {
        mid = (arg->mesh_ynum - 1)*arg->mesh_xnum;
    }
    if(cuda->x[id] < arg->domain_x && cuda->x[id] >= 0.0)
    {
        mid += __double2int_rz(cuda->x[id]/arg->mesh_dx);
    }
    else
    {
        mid += arg->mesh_xnum - 1;
    }
    //printf("mid is:%lf\n",mid);
    //printf("xnum is:%d,ynum is:%d\n",__double2int_rz(cuda->x[id]/arg->mesh_dx),__double2int_rz(cuda->y[id]/arg->mesh_dx));
    //printf("x is:%lf,y is:%lf\n",cuda->x[id],cuda->y[id]);
    mesh_index = atomicAdd(&cuda->mesh_count[mid],1);
    mesh_index = mesh_index*arg->mesh_num + mid;
    cuda->mesh[mesh_index] = id;
}

