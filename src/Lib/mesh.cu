#include "hip/hip_runtime.h"
#include "Lib.cuh"

__global__ void sph_mesh_cuda(double *x,double *y,double *accx,double *accy,double *drho,int *type,int *mesh,int ptc_num)
{
    //const int bid = blockIdx.x;
    //const int tid = threadIdx.x;
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= ptc_num) return;
    accx[id] = 0.0;
    drho[id] = 0.0;
    if(type[id] == 0) accy[id] = -GRAVITY_ACC;
    else accy[id] = 0.0;

    int mid;

    if(y[id] < TOL_DOMAIN_DEEPTH && y[id] >= 0)
    {
        mid = __double2int_rz(y[id]/dev_mesh_spacing)*dev_mesh_lnum;
    }
    else
    {
        mid = (dev_mesh_dnum - 1)*dev_mesh_lnum;
    }
    if(x[id] < TOL_DOMAIN_LENGTH && x[id] >= 0)
    {
        mid += __double2int_rz(x[id]/dev_mesh_spacing);
    }
    else
    {
        mid += dev_mesh_lnum - 1;
    }
    mid += dev_mesh_tnum*atomicAdd(&mesh[mid+(MESH_PTC_NUM-1)*dev_mesh_tnum],1);
    mesh[mid] = id;
}

