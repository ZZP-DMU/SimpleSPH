#include "hip/hip_runtime.h"
#include "Equations.cuh"

__global__ void sph_L_init(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id < arg->ptc_num)
    {
        cuda->Lxx[id] = 0.0;
        cuda->Lxy[id] = 0.0;
        cuda->Lyx[id] = 0.0;
        cuda->Lyy[id] = 0.0;

        cuda->Lrho_x[id] = 0.0;
        cuda->Lrho_y[id] = 0.0;
    }
}

__global__ void sph_L_sum(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    int id = 0;
    int index_i = 0;
    int index_j = 0;
    double tmp_Lxx = 0.0;
    double tmp_Lxy = 0.0;
    double tmp_Lyx = 0.0;
    double tmp_Lyy = 0.0;

    if( threadIdx.x < cuda->pair_count[mesh_id])
    {
        id = mesh_id * arg->pair_volume + threadIdx.x;
        index_i = cuda->pair_i[id];
        index_j = cuda->pair_j[id];

        //dx = cuda->x[index_i] - cuda->x[index_j];
        //dy = cuda->y[index_i] - cuda->y[index_j];
        tmp_Lxx = (cuda->x[index_i] - cuda->x[index_j])*cuda->dwdx[id]*arg->m;
        tmp_Lxy = (cuda->x[index_i] - cuda->x[index_j])*cuda->dwdy[id]*arg->m;
        tmp_Lyx = (cuda->y[index_i] - cuda->y[index_j])*cuda->dwdx[id]*arg->m;
        tmp_Lyy = (cuda->y[index_i] - cuda->y[index_j])*cuda->dwdy[id]*arg->m;

        atomicAdd(&(cuda->Lxx[index_i]),tmp_Lxx/cuda->rho[index_j]);
        atomicAdd(&(cuda->Lxy[index_i]),tmp_Lxy/cuda->rho[index_j]);
        atomicAdd(&(cuda->Lyx[index_i]),tmp_Lyx/cuda->rho[index_j]);
        atomicAdd(&(cuda->Lyy[index_i]),tmp_Lyy/cuda->rho[index_j]);
        if(cuda->type[index_j] == 0)
        {
            atomicAdd(&(cuda->Lxx[index_j]),tmp_Lxx/cuda->rho[index_i]);
            atomicAdd(&(cuda->Lxy[index_j]),tmp_Lxy/cuda->rho[index_i]);
            atomicAdd(&(cuda->Lyx[index_j]),tmp_Lyx/cuda->rho[index_i]);
            atomicAdd(&(cuda->Lyy[index_j]),tmp_Lyy/cuda->rho[index_i]);
        }
    }
}

__global__ void sph_L_inver(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    double det = 0.0;
    double tmp_Lxx = 0.0;
    double tmp_Lyy = 0.0;

    if(id < arg->ptc_num)
    {
        if(cuda->type[id] == 0)
        {
            det = cuda->Lxx[id]*cuda->Lyy[id] - cuda->Lxy[id]*cuda->Lyx[id];
            if(det != 0.0)
            {
                tmp_Lxx = cuda->Lyy[id]/det;
                tmp_Lyy = cuda->Lxx[id]/det;

                cuda->Lxx[id] = tmp_Lxx;
                cuda->Lyy[id] = tmp_Lyy;
                cuda->Lxy[id] /= -det;
                cuda->Lyx[id] /= -det;
            }
        }
    }
}

__global__ void sph_L_rho(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    int id = 0;
    int index_i = 0;
    int index_j = 0;
    double tmp_Lrho_x_i = 0.0;
    double tmp_Lrho_y_i = 0.0;
    double tmp_Lrho_x_j = 0.0;
    double tmp_Lrho_y_j = 0.0;
    if( threadIdx.x < cuda->pair_count[mesh_id])
    {
        id = mesh_id * arg->pair_volume + threadIdx.x;
        index_i = cuda->pair_i[id];
        index_j = cuda->pair_j[id];

        tmp_Lrho_x_i = (cuda->rho[index_i]-cuda->rho[index_j])*(cuda->Lxx[index_i]*cuda->dwdx[id] + cuda->Lxy[index_i]*cuda->dwdy[id])*arg->m/cuda->rho[index_j];
        tmp_Lrho_y_i = (cuda->rho[index_i]-cuda->rho[index_j])*(cuda->Lyx[index_i]*cuda->dwdx[id] + cuda->Lyy[index_i]*cuda->dwdy[id])*arg->m/cuda->rho[index_j];
        atomicAdd(&(cuda->Lrho_x[index_i]),tmp_Lrho_x_i);
        atomicAdd(&(cuda->Lrho_y[index_i]),tmp_Lrho_y_i);

        tmp_Lrho_x_j = (cuda->rho[index_i]-cuda->rho[index_j])*(cuda->Lxx[index_j]*cuda->dwdx[id] + cuda->Lxy[index_j]*cuda->dwdy[id])*arg->m/cuda->rho[index_i];
        tmp_Lrho_y_j = (cuda->rho[index_i]-cuda->rho[index_j])*(cuda->Lyx[index_j]*cuda->dwdx[id] + cuda->Lyy[index_j]*cuda->dwdy[id])*arg->m/cuda->rho[index_i];
        atomicAdd(&(cuda->Lrho_x[index_j]),tmp_Lrho_x_j);
        atomicAdd(&(cuda->Lrho_y[index_j]),tmp_Lrho_y_j);
    }
}

__global__ void sph_delta_term(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    int id = 0;
    int index_i = 0;
    int index_j = 0;
    double drho = 0.0;
    double dx = 0.0;
    double dy = 0.0;
    if( threadIdx.x < cuda->pair_count[mesh_id])
    {
        id =mesh_id * arg->pair_volume + threadIdx.x;
        index_i = cuda->pair_i[id];
        index_j = cuda->pair_j[id];
        dx = cuda->x[index_i] - cuda->x[index_j];
        dy = cuda->y[index_i] - cuda->y[index_j];
        
        drho = 2.0*(cuda->rho[index_i] - cuda->rho[index_j]);
        drho += (cuda->Lrho_x[index_i] + cuda->Lrho_x[index_j])*dx + (cuda->Lrho_y[index_i]+cuda->Lrho_y[index_j])*dy;
        drho *= 0.01*arg->c*arg->h*(dx*cuda->dwdx[id] + dy*cuda->dwdy[id])*arg->m/(dx*dx+dy*dy);

        atomicAdd(&(cuda->drho[index_i]),-drho/cuda->rho[index_j]);
        atomicAdd(&(cuda->drho[index_j]),drho/cuda->rho[index_i]);
    }
    __syncthreads();
    if( threadIdx.x == 0) cuda->pair_count[mesh_id] = 0;
}

void sph_delta_cuda(SPH *sph)
{
    dim3 pair_block(sph->host_arg->pair_volume);
    dim3 pair_grid(sph->host_arg->mesh_xnum, sph->host_arg->mesh_ynum);
    dim3 ptc_block(256);
    dim3 ptc_grid((int)(sph->host_arg->ptc_num / 256) + 1);

    sph_L_init<<<ptc_grid,ptc_block>>>(sph->cuda,sph->dev_arg,sph->dev_rigid);
    hipDeviceSynchronize();
    sph_L_sum<<<pair_grid,pair_block>>>(sph->cuda,sph->dev_arg,sph->dev_rigid);
    hipDeviceSynchronize();
    sph_L_inver<<<ptc_grid,ptc_block>>>(sph->cuda,sph->dev_arg,sph->dev_rigid);
    hipDeviceSynchronize();
    sph_L_rho<<<pair_grid,pair_block>>>(sph->cuda,sph->dev_arg,sph->dev_rigid);
    hipDeviceSynchronize();
    sph_delta_term<<<pair_grid,pair_block>>>(sph->cuda,sph->dev_arg,sph->dev_rigid);
    hipDeviceSynchronize();
}