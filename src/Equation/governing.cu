#include "hip/hip_runtime.h"
#include "Equations.cuh"

__global__ void sph_governing_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    double accx_i = 0.0;
    double accx_j = 0.0;

    double accy_i = 0.0;
    double accy_j = 0.0;

    double drho = 0.0;
    double tmp_acc_p = 0.0;
    double tmp_acc_v = 0.0;
    int index_i = 0.0;
    int index_j = 0.0;
    double dx = 0.0;
    double dy = 0.0;
    double dvx = 0.0;
    double dvy = 0.0;
    const int mesh_id =  blockIdx.x + blockIdx.y * gridDim.x;
    int id = 0;
    if( threadIdx.x < cuda->pair_count[mesh_id]) 
    {
        id = mesh_id*arg->pair_volume + threadIdx.x;
        index_i = cuda->pair_i[id];
        index_j = cuda->pair_j[id];

        //tmp_acc_p = cuda->p[index_i]/pow(cuda->rho[index_i],2) + cuda->p[index_j]/pow(cuda->rho[index_j],2);
        tmp_acc_p = (cuda->p[index_i] + cuda->p[index_j])/(cuda->rho[index_i] * cuda->rho[index_j]);
        dx = cuda->x[index_i] - cuda->x[index_j];
        dy = cuda->y[index_i] - cuda->y[index_j];
        if(cuda->type[index_j] == 0)
        {
            dvx = cuda->vx[index_i] - cuda->vx[index_j];
            dvy = cuda->vy[index_i] - cuda->vy[index_j];
            drho = (cuda->vx[index_i]-cuda->vx[index_j])*cuda->dwdx[id]+(cuda->vy[index_i]-cuda->vy[index_j])*cuda->dwdy[id];
            drho *= arg->m;
        }
        else if(cuda->type[index_j] == -1)
        {
            //dvx = cuda->vx[index_i] - (0.0 - cuda->vx[index_j]);
            //dvy = cuda->vy[index_i] - (0.0 - cuda->vy[index_j]);
            dvx = cuda->vx[index_i];
            dvy = cuda->vy[index_i];
            drho = cuda->vx[index_i]*cuda->dwdx[id]+cuda->vy[index_i]*cuda->dwdy[id];
            drho *= arg->m;
        }
        else if(cuda->type[index_j] == 1 || cuda->type[index_j] == 2)
        {
            //dvx = cuda->vx[index_i] - (2.0*(rigid->vx - rigid->omega*(cuda->y[index_j]-rigid->cogy)) - cuda->vx[index_j]);
            //dvy = cuda->vy[index_i] - (2.0*(rigid->vy + rigid->omega*(cuda->x[index_j]-rigid->cogx)) - cuda->vy[index_j]);
            dvx = cuda->vx[index_i] - (rigid->vx - rigid->omega*(cuda->y[index_j]-rigid->cogy));
            dvy = cuda->vy[index_j] - (rigid->vy + rigid->omega*(cuda->x[index_j]-rigid->cogx));
            drho = (cuda->vx[index_i] - (rigid->vx - rigid->omega*(cuda->y[index_j]-rigid->cogy)))*cuda->dwdx[id]+\
                      (cuda->vy[index_i] - (rigid->vy + rigid->omega*(cuda->x[index_j]-rigid->cogx)))*cuda->dwdy[id];
            drho *= arg->m;
        }

        tmp_acc_v = dx*dvx+dy*dvy;
        if(tmp_acc_v > 0.0) tmp_acc_v = 0.0;
        tmp_acc_v = (tmp_acc_v*0.05*arg->h*arg->c)/((dx*dx+dy*dy+0.01*arg->h)*0.5*(cuda->rho[index_i]+cuda->rho[index_j]));

        //accx = arg->m * ( tmp_acc_v - tmp_acc_p) *cuda->dwdx[id];
        //accy = arg->m * ( tmp_acc_v - tmp_acc_p) *cuda->dwdy[id];
        accx_i = arg->m * ( tmp_acc_v - tmp_acc_p) *cuda->dwdx[id];
        accy_i = arg->m * ( tmp_acc_v - tmp_acc_p) *cuda->dwdy[id];

        if(cuda->type[index_j == 0])
        {
            accx_j = -accx_i;
            accy_j = -accy_i;
        }
        else if (cuda->type[index_j] == 2)
        {
            accx_j = arg->m * tmp_acc_p * cuda->dwdx[id];
            accy_j = arg->m * tmp_acc_p * cuda->dwdy[id];
        }
        

        
        atomicAdd(&(cuda->accx[index_i]),accx_i);
        atomicAdd(&(cuda->accx[index_j]),accx_j);
        atomicAdd(&(cuda->accy[index_i]),accy_i);
        atomicAdd(&(cuda->accy[index_j]),accy_j);
        atomicAdd(&(cuda->drho[index_i]),drho);
        atomicAdd(&(cuda->drho[index_j]),drho);
    }
    __syncthreads();
    //if( threadIdx.x == 0)cuda->pair_count[mesh_id] = 0;
}