#include "hip/hip_runtime.h"
#include "Equations.cuh"

__global__ void sph_governing_cuda(SPH_CUDA *cuda,SPH_ARG *arg,SPH_RIGID *rigid)
{
    double accx = 0.0;
    double accy = 0.0;
    double drho_i = 0.0;
    double drho_j = 0.0;
    double tmp_acc_p = 0.0;
    double tmp_acc_v = 0.0;
    int index_i = 0.0;
    int index_j = 0.0;
    double dx = 0.0;
    double dy = 0.0;
    double dvx = 0.0;
    double dvy = 0.0;
    const int mesh_id = blockIdx.x + blockIdx.y * gridDim.x;
    int id = 0;
    if( threadIdx.x < cuda->pair_count[mesh_id]) 
    {
        id = mesh_id*arg->pair_volume + threadIdx.x;
        index_i = cuda->pair_i[id];
        index_j = cuda->pair_j[id];

        tmp_acc_p = cuda->p[index_i]/pow(cuda->rho[index_i],2) + cuda->p[index_j]/pow(cuda->rho[index_j],2);
        dx = cuda->x[index_i] - cuda->x[index_j];
        dy = cuda->y[index_i] - cuda->y[index_j];
        if(cuda->type[index_j] == 0)
        {
            dvx = cuda->vx[index_i] - cuda->vx[index_j];
            dvy = cuda->vy[index_i] - cuda->vy[index_j];
            drho_i = (cuda->vx[index_i]-cuda->vx[index_j])*cuda->dwdx[id]+(cuda->vy[index_i]-cuda->vy[index_j])*cuda->dwdy[id];
            drho_i *= arg->m;
        }
        else if(cuda->type[index_j] == -1)
        {
            dvx = cuda->vx[index_i] - (0.0 - cuda->vx[index_j]);
            dvy = cuda->vy[index_i] - (0.0 - cuda->vy[index_j]);
            drho_i = cuda->vx[index_i]*cuda->dwdx[id]+cuda->vy[index_i]*cuda->dwdy[id];
            drho_i *= arg->m;
        }
        else if(cuda->type[index_j] == 1)
        {
            dvx = cuda->vx[index_i] - (2.0*(rigid->vx - rigid->omega*(cuda->y[index_j]-rigid->cogy)) - cuda->vx[index_j]);
            dvy = cuda->vy[index_i] - (2.0*(rigid->vy + rigid->omega*(cuda->x[index_j]-rigid->cogx)) - cuda->vy[index_j]);
            drho_i = (cuda->vx[index_i] - (rigid->vx - rigid->omega*(cuda->y[index_j]-rigid->cogy)))*cuda->dwdx[id]+\
                      (cuda->vy[index_i] - (rigid->vy + rigid->omega*(cuda->x[index_j]-rigid->cogx)))*cuda->dwdy[id];
            drho_i *= arg->m;
        }

        tmp_acc_v = dx*dvx+dy*dvy;
        if(tmp_acc_v > 0.0) tmp_acc_v = 0.0;
        tmp_acc_v = (tmp_acc_v*0.01*arg->h*arg->c)/((dx*dx+dy*dy+0.01*arg->h)*0.5*(cuda->rho[index_i]+cuda->rho[index_j]));

        accx = arg->m * ( tmp_acc_v - tmp_acc_p) *cuda->dwdx[id];
        accy = arg->m * ( tmp_acc_v - tmp_acc_p) *cuda->dwdy[id];

        drho_j = drho_i;
        drho_i += 0.01*arg->h*arg->c*2*(cuda->rho[index_i]/cuda->rho[index_j]-1)*arg->m*(dx*cuda->dwdx[id]+dy*cuda->dwdy[id])/(dx*dx+dy*dy);
        drho_j += 0.01*arg->h*arg->c*2*(cuda->rho[index_j]/cuda->rho[index_i]-1)*arg->m*(dx*cuda->dwdx[id]+dy*cuda->dwdy[id])/(dx*dx+dy*dy); 
        
        atomicAdd(&(cuda->accx[index_i]),accx);
        atomicAdd(&(cuda->accx[index_j]),-accx);
        atomicAdd(&(cuda->accy[index_i]),accy);
        atomicAdd(&(cuda->accy[index_j]),-accy);
        atomicAdd(&(cuda->drho[index_i]),drho_i);
        atomicAdd(&(cuda->drho[index_j]),drho_j);
    }
    __syncthreads();
    if( threadIdx.x == 0)cuda->pair_count[mesh_id] = 0;
}