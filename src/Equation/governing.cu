#include "hip/hip_runtime.h"
#include "Equations.cuh"

__global__ void sph_governing_cuda(double *x,double *y,double *vx,double *vy,\
double *rho,double *p,int *type,int *pair_i,int *pair_j,double *dwdx,\
double *dwdy,double *accx,double *accy,double *drho,double *rigid,int* pair_num,int ptc_num)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= pair_num[0] )return;

    double dx;
    double dy;
    double dvx;
    double dvy;
    double rho_temp=0.0;
    double accx_temp=0.0;
    double accy_temp=0.0;
    double temp=0.0;

    dx = x[pair_i[id]]-x[pair_j[id]];
    dy = x[pair_i[id]]-y[pair_j[id]];

    accx_temp = (-dev_m*p[pair_i[id]]*dwdx[id])/pow(rho[pair_i[id]],2)+(-dev_m*p[pair_j[id]]*dwdx[id])/pow(rho[pair_j[id]],2);
    accy_temp = (-dev_m*p[pair_i[id]]*dwdy[id])/pow(rho[pair_i[id]],2)+(-dev_m*p[pair_j[id]]*dwdy[id])/pow(rho[pair_j[id]],2);
    //accx_temp = -dev_m*(p[pair_i[id]]/(rho[pair_i[id]]*rho[pair_i[id]])+p[pair_j[id]]/(rho[pair_j[id]]*rho[pair_j[id]]));

    //accx[id] = acc_temp*dwdx[id];
    //accy[id] = acc_temp*dwdy[id];

    if(type[pair_j[id]] == 0)
    {
        dvx = vx[pair_i[id]]-vx[pair_j[id]];
        dvy = vy[pair_i[id]]-vy[pair_j[id]];
        rho_temp = dvx*dwdx[id]+dvy*dwdy[id];
        rho_temp *= dev_m;
    }
    else if(type[pair_j[id]] == 1)
    {
        dvx = vx[pair_i[id]] - (2.0*(rigid[VX] - rigid[OMEGA]*(y[pair_j[id]]-rigid[COGY])) - vx[pair_j[id]]);
        dvy = vy[pair_i[id]] - (2.0*(rigid[VY] + rigid[OMEGA]*(x[pair_j[id]]-rigid[COGX])) - vy[pair_j[id]]);
        rho_temp = (vx[pair_i[id]]-(rigid[VX] - rigid[OMEGA]*(y[pair_j[id]]-rigid[COGY])))*dwdx[id]+\
                   (vy[pair_i[id]]-(rigid[VY] + rigid[OMEGA]*(x[pair_j[id]]-rigid[COGX])))*dwdy[id];
        rho_temp *= dev_m;
    }
    else if(type[pair_j[id]] == -1)
    {
        dvx = vx[pair_i[id]] - (0.0 - vx[pair_j[id]]); 
        dvy = vy[pair_i[id]] - (0.0 - vy[pair_j[id]]);
        rho_temp = vx[pair_i[id]]*dwdx[id]+vy[pair_j[id]]*dwdy[id];
        rho_temp *= dev_m;
    }
    /*
    accy_temp = dx*dvx+dy*dvy;
    if(accy_temp < 0.0) accy_temp = 0.0;
    
    accx_temp += accy_temp*dev_m*0.01*dev_h*dev_c/((dx*dx+dy*dy+0.01*dev_h*dev_h)*0.5*(rho[pair_i[id]]+rho[pair_j[id]]));
    accy_temp = accx_temp*dwdy[id];
    accx_temp *= dwdx[id];
    */

    atomicAdd(&accx[pair_i[id]], accx_temp);
    atomicAdd(&accx[pair_j[id]],-accx_temp);
    atomicAdd(&accy[pair_i[id]], accy_temp);
    atomicAdd(&accy[pair_j[id]],-accy_temp);
    atomicAdd(&drho[pair_i[id]],rho_temp);
    atomicAdd(&drho[pair_j[id]],rho_temp);
}