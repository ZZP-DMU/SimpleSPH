#include "hip/hip_runtime.h"
#include "Equations.cuh"

/*
__global__ void sph_governing_cuda(double *x,double *y,double *vx,double *vy,\
double *rho,double *p,int *type,int *pair_i,int *pair_j,double *dwdx,\
double *dwdy,double *accx,double *accy,double *drho,double *rigid,int* pair_num,int ptc_num)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= pair_num[0] )return;

    if(rho[pair_i[id]] == 0.0 || rho[pair_j[id]] == 0.0)
    {
        printf("id:%d pair_i:%d pair_j:%d pair_num:%d ptc_num:%d\n",id,pair_i[id],pair_j[id],*pair_num,ptc_num);
    }

    double dx;
    double dy;
    double dvx;
    double dvy;
    double rho_temp=0.0;
    double accx_temp=0.0;
    double accy_temp=0.0;
    double temp=0.0;

    dx = x[pair_i[id]]-x[pair_j[id]];
    dy = x[pair_i[id]]-y[pair_j[id]];
    if(rho[pair_i[id]] == 0.0 || rho[pair_j[id]] == 0.0)
    {
        printf("id:%d pair_i:%d i_type:%d pair_j:%d j_type:%d ptc_num:%d pair_num:%d\n",id,pair_i[id],type[pair_i[id]],pair_j[id],type[pair_j[id]],ptc_num,*pair_num);
    }

    //accx_temp = (-dev_m*p[pair_i[id]]*dwdx[id])/pow(rho[pair_i[id]],2)+(-dev_m*p[pair_j[id]]*dwdx[id])/pow(rho[pair_j[id]],2);
    //accy_temp = (-dev_m*p[pair_i[id]]*dwdy[id])/pow(rho[pair_i[id]],2)+(-dev_m*p[pair_j[id]]*dwdy[id])/pow(rho[pair_j[id]],2);
    accx_temp = -dev_m*(p[pair_i[id]]/(rho[pair_i[id]]*rho[pair_i[id]])+p[pair_j[id]]/(rho[pair_j[id]]*rho[pair_j[id]]));

    //accx[id] = acc_temp*dwdx[id];
    //accy[id] = acc_temp*dwdy[id];

    if(type[pair_j[id]] == 0)
    {
        dvx = vx[pair_i[id]]-vx[pair_j[id]];
        dvy = vy[pair_i[id]]-vy[pair_j[id]];
        rho_temp = dvx*dwdx[id]+dvy*dwdy[id];
        rho_temp *= dev_m;
    }
    else if(type[pair_j[id]] == 1)
    {
        dvx = vx[pair_i[id]] - (2.0*(rigid[VX] - rigid[OMEGA]*(y[pair_j[id]]-rigid[COGY])) - vx[pair_j[id]]);
        dvy = vy[pair_i[id]] - (2.0*(rigid[VY] + rigid[OMEGA]*(x[pair_j[id]]-rigid[COGX])) - vy[pair_j[id]]);
        rho_temp = (vx[pair_i[id]]-(rigid[VX] - rigid[OMEGA]*(y[pair_j[id]]-rigid[COGY])))*dwdx[id]+\
                   (vy[pair_i[id]]-(rigid[VY] + rigid[OMEGA]*(x[pair_j[id]]-rigid[COGX])))*dwdy[id];
        rho_temp *= dev_m;
    }
    else if(type[pair_j[id]] == -1)
    {
        dvx = vx[pair_i[id]] - (0.0 - vx[pair_j[id]]); 
        dvy = vy[pair_i[id]] - (0.0 - vy[pair_j[id]]);
        rho_temp = vx[pair_i[id]]*dwdx[id]+vy[pair_j[id]]*dwdy[id];
        rho_temp *= dev_m;
    }

    accy_temp = dx*dvx+dy*dvy;
    if(accy_temp < 0.0) accy_temp = 0.0;
    
    accx_temp += accy_temp*dev_m*0.01*dev_h*dev_c/((dx*dx+dy*dy+0.01*dev_h*dev_h)*0.5*(rho[pair_i[id]]+rho[pair_j[id]]));
    accy_temp = accx_temp*dwdy[id];
    accx_temp *= dwdx[id];

    atomicAdd(&accx[pair_i[id]], accx_temp);
    atomicAdd(&accx[pair_j[id]],-accx_temp);
    atomicAdd(&accy[pair_i[id]], accy_temp);
    atomicAdd(&accy[pair_j[id]],-accy_temp);
    atomicAdd(&drho[pair_i[id]],rho_temp);
    atomicAdd(&drho[pair_j[id]],rho_temp);
}*/